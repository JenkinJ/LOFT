#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "datastructs.cu"
#include "macros.cpp"
#include "interp.cu"
#ifndef VORT_CU
#define VORT_CU

/* Compute the nondimensional pressure */
__device__ void calc_pi(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    // this is actually the pressure
    // perturbation, not the full pressure
    float *buf0 = data->prespert;
    float p = BUF4D(i, j, k, t) + grid->p0[k]; 
    buf0 = data->pi;
    BUF4D(i, j, k, t) = powf( p / 100000., 0.28571426);
}

/* Compute the x component of vorticity. After this is called by the calvort kernel, you must also run 
   the kernel for applying the lower boundary condition and then the kernel for averaging to the
   scalar grid. */
__device__ void calc_xvort(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *vstag = data->vstag;
    float *wstag = data->wstag;
    float *dum0 = data->tem1;
    float dy = yf(j) - yf(j-1);
    float dz = zf(k) - zf(k-1);

    float dwdy = ( ( WA4D(i, j, k, t) - WA4D(i, j-1, k, t) )/dy );
    float dvdz = ( ( VA4D(i, j, k, t) - VA4D(i, j, k-1, t) )/dz );
    TEM4D(i, j, k, t) = dwdy - dvdz; 
    if (k == 1) {
        TEM4D(i, j, 0, t) = dwdy - dvdz; 
    }
}

/* Compute the y component of vorticity. After this is called by the calvort kernel, you must also run 
   the kernel for applying the lower boundary condition and then the kernel for averaging to the
   scalar grid. */
__device__ void calc_yvort(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->ustag;
    float *wstag = data->wstag;
    float *dum0 = data->tem2;
    float dx = xf(i) - xf(i-1);
    float dz = zf(k) - zf(k-1);

    float dwdx = ( ( WA4D(i, j, k, t) - WA4D(i-1, j, k, t) )/dx );
    float dudz = ( ( UA4D(i, j, k, t) - UA4D(i, j, k-1, t) )/dz );
    TEM4D(i, j, k, t) = dudz - dwdx;
    if (k == 1) {
        TEM4D(i, j, 0, t) = dudz - dwdx;
    }
}

/* Compute the z component of vorticity. After this is called by the calvort kernel, you must also run 
   the kernel for applying the lower boundary condition and then the kernel for averaging to the
   scalar grid. */
__device__ void calc_zvort(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->ustag;
    float *vstag = data->vstag;
    float *dum0 = data->tem3;
    float dx = xf(i) - xf(i-1);
    float dy = yf(j) - yf(j-1);

    float dvdx = ( ( VA4D(i, j, k, t) - VA4D(i-1, j, k, t) )/dx);
    float dudy = ( ( UA4D(i, j, k, t) - UA4D(i, j-1, k, t) )/dy);
    TEM4D(i, j, k, t) = dvdx - dudy;
}

/* Compute the X component of vorticity tendency due
   to tilting Y and Z components into the X direction */
__device__ void calc_xvort_tilt(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->ustag;
    float *dum0;
    float dy = yf(j) - yf(j-1);
    float dz = zf(k) - zf(k-1);

    if (k >= 0) {
        // dudy in tem1
        dum0 = data->tem1;
        TEM4D(i, j, k, t) = ( ( UA4D(i, j, k, t) - UA4D(i, j-1, k, t) ) / dy );
    }

    if (k >= 1) {
        // dudz in tem2
        dum0 = data->tem2;
        TEM4D(i, j, k, t) = ( ( UA4D(i, j, k, t) - UA4D(i, j, k-1, t) ) / dz );
    }
    // This is the equivalent of our zero strain lower boundary
    if (k == 1) {
        // dudz in tem2
        dum0 = data->tem2;
        TEM4D(i, j, 0, t) = ( ( UA4D(i, j, 1, t) - UA4D(i, j, 0, t) ) / dz );
    }    
}

/* Compute the Y component of vorticity tendency due
   to tilting X and Z components into the X direction */
__device__ void calc_yvort_tilt(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *vstag = data->vstag;
    float *dum0;
    float dx = xf(i) - xf(i-1);
    float dz = zf(k) - zf(k-1);
    
    if (k >=0) {
        // dvdx in tem1
        dum0 = data->tem1;
        TEM4D(i, j, k, t) = ( ( VA4D(i, j, k, t) - VA4D(i-1, j, k, t) ) / dx );
    }

    if (k >= 1) {
        // dvdz in tem2
        dum0 = data->tem2;
        TEM4D(i, j, k, t) = ( ( VA4D(i, j, k, t) - VA4D(i, j, k-1, t) ) / dz );
    }
    // This is the equivalent of our zero strain lower boundary
    if (k == 1) {
        // dvdz in tem2
        dum0 = data->tem2;
        TEM4D(i, j, 0, t) = ( ( VA4D(i, j, 1, t) - VA4D(i, j, 0, t) ) / dz );
    }
}

/* Compute the Z component of vorticity tendency due
   to tilting X and Y components into the X direction */
__device__ void calc_zvort_tilt(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *wstag = data->wstag;
    float dx = xf(i) - xf(i-1);
    float dy = yf(j) - yf(j-1);

    // Compute dw/dx and put it in the tem1 array. The derivatives
    // land on weird places so we have to average each derivative back
    // to the scalar grid, resulting in this clunky approach
    if (k >= 0) {
        float *dum0 = data->tem1;
        TEM4D(i, j, k, t) = ( ( WA4D(i, j, k, t) - WA4D(i-1, j, k, t) ) / dx );

        // put dw/dy in tem2
        dum0 = data->tem2;
        TEM4D(i, j, k, t) = ( ( WA4D(i, j, k, t) - WA4D(i, j-1, k, t) ) / dy );
    }
}

/* Compute the X component of vorticity tendency due
   to stretching of the vorticity along the X axis. */
__device__ void calc_xvort_stretch(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];
    

    float *wstag = data->wstag;
    float *vstag = data->vstag;
    float *xvort = data->xvort;
    float *xvort_stretch = data->xvstretch;
    float dy = yf(j+1) - yf(j);
    float dz = zf(k+1) - zf(k);

    float rrv = grid->rho0[k];
    float rrw1, rrw2;
    if ( k == 0 ) {
        rrw1 = 1.75*grid->rho0[1] - grid->rho0[2] + 0.25*grid->rho0[3];
        rrw2 = 0.5*grid->rho0[2] + 0.5*grid->rho0[3];
    }
    else {
        rrw1 = 0.5*grid->rho0[k-1] + 0.5*grid->rho0[k  ];
        rrw2 = 0.5*grid->rho0[k  ] + 0.5*grid->rho0[k+1];
    }

    // this stencil conveniently lands itself on the scalar grid,
    // so we won't have to worry about doing any averaging. I think.
    float *buf0 = xvort;
    float xv = BUF4D(i, j, k, t);
    float dvdy, dwdz;
    dvdy = rrv * ( ( VA4D(i, j+1, k, t) - VA4D(i, j, k, t) )/dy);
    dwdz = ( (  rrw2*WA4D(i, j, k+1, t) - rrw1*WA4D(i, j, k, t) )/dz);

    buf0 = xvort_stretch;
    BUF4D(i, j, k, t) = -xv*( (dvdy + dwdz) / rrv);

}

/* Compute the Y component of vorticity tendency due
   to stretching of the vorticity along the Y axis. */
__device__ void calc_yvort_stretch(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->ustag;
    float *wstag = data->wstag;
    float *yvort = data->yvort;
    float *yvort_stretch = data->yvstretch;
    float dx = xf(i+1) - xf(i);
    float dz = zf(k+1) - zf(k);

    float rru = grid->rho0[k];
    float rrw1, rrw2;
    if ( k == 0 ) {
        rrw1 = 1.75*grid->rho0[1] - grid->rho0[2] + 0.25*grid->rho0[3];
        rrw2 = 0.5*grid->rho0[2] + 0.5*grid->rho0[3];
    }
    else {
        rrw1 = 0.5*grid->rho0[k-1] + 0.5*grid->rho0[k  ];
        rrw2 = 0.5*grid->rho0[k  ] + 0.5*grid->rho0[k+1];
    }

    // this stencil conveniently lands itself on the scalar grid,
    // so we won't have to worry about doing any averaging. I think.
    float *buf0 = yvort;
    float yv = BUF4D(i, j, k, t);
    float dudx, dwdz;
    dudx = ( rru*( UA4D(i+1, j, k, t) - UA4D(i, j, k, t) )/dx);
    dwdz = ( ( rrw2*WA4D(i, j, k+1, t) - rrw1*WA4D(i, j, k, t) )/dz);

    buf0 = yvort_stretch;
    BUF4D(i, j, k, t) = -yv*( (dudx + dwdz) / rru);
}

/* Compute the Z component of vorticity tendency due
   to stretching of the vorticity along the Z axis. */
__device__ void calc_zvort_stretch(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->ustag;
    float *vstag = data->vstag;
    float *wstag = data->wstag;
    float *zvort = data->zvort;
    float *zvort_stretch = data->zvstretch;
    float dx = xf(i+1) - xf(i);
    float dy = yf(j+1) - yf(j);

    // this stencil conveniently lands itself on the scalar grid,
    // so we won't have to worry about doing any averaging. I think.
    float *buf0 = zvort;
    float zv = BUF4D(i, j, k, t);
    float dudx = ( ( UA4D(i+1, j, k, t) - UA4D(i, j, k, t) )/dx);
    float dvdy = ( ( VA4D(i, j+1, k, t) - VA4D(i, j, k, t) )/dy);

    buf0 = zvort_stretch;
    BUF4D(i, j, k, t) = -zv*( dudx + dvdy);
}

/* Compute the X vorticity tendency due to the turbulence closure scheme */
__device__ void calc_xvortturb_ten(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *vstag = data->turbv;
    float *wstag = data->turbw;
    float *dum0 = data->tem1;
    float dy = yf(j) - yf(j-1);
    float dz = zf(k) - zf(k-1);

    float dwdy = ( ( WA4D(i, j, k, t) - WA4D(i, j-1, k, t) )/dy );
    float dvdz = ( ( VA4D(i, j, k, t) - VA4D(i, j, k-1, t) )/dz );
    TEM4D(i, j, k, t) = dwdy - dvdz; 
    if (k == 1) {
        TEM4D(i, j, 0, t) = dwdy - dvdz; 
    }
}

/* Compute the Y vorticity tendency due to the turbulence closure scheme */
__device__ void calc_yvortturb_ten(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->turbu;
    float *wstag = data->turbw;
    float *dum0 = data->tem2;
    float dx = xf(i) - xf(i-1);
    float dz = zf(k) - zf(k-1);

    float dwdx = ( ( WA4D(i, j, k, t) - WA4D(i-1, j, k, t) )/dx );
    float dudz = ( ( UA4D(i, j, k, t) - UA4D(i, j, k-1, t) )/dz );
    TEM4D(i, j, k, t) = dudz - dwdx;
    if (k == 1) {
        TEM4D(i, j, 0, t) = dudz - dwdx;
    }
}

/* Compute the Z vorticity tendency due to the turbulence closure scheme */
__device__ void calc_zvortturb_ten(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->turbu;
    float *vstag = data->turbv;
    float *dum0 = data->tem3;
    float dx = xf(i) - xf(i-1);
    float dy = yf(j) - yf(j-1);

    float dvdx = ( ( VA4D(i, j, k, t) - VA4D(i-1, j, k, t) )/dx);
    float dudy = ( ( UA4D(i, j, k, t) - UA4D(i, j-1, k, t) )/dy);
    TEM4D(i, j, k, t) = dvdx - dudy;
}


/* Compute the X vorticity tendency due to the 6th order numerical diffusion */
__device__ void calc_xvortdiff_ten(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *vstag = data->diffv;
    float *wstag = data->diffw;
    float *dum0 = data->tem1;
    float dy = yf(j) - yf(j-1);
    float dz = zf(k) - zf(k-1);

    float dwdy = ( ( WA4D(i, j, k, t) - WA4D(i, j-1, k, t) )/dy );
    float dvdz = ( ( VA4D(i, j, k, t) - VA4D(i, j, k-1, t) )/dz );
    TEM4D(i, j, k, t) = dwdy - dvdz; 
    if (k == 1) {
        TEM4D(i, j, 0, t) = dwdy - dvdz; 
    }
}

/* Compute the Y vorticity tendency due to the 6th order numerical diffusion */
__device__ void calc_yvortdiff_ten(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->diffu;
    float *wstag = data->diffw;
    float *dum0 = data->tem2;
    float dx = xf(i) - xf(i-1);
    float dz = zf(k) - zf(k-1);

    float dwdx = ( ( WA4D(i, j, k, t) - WA4D(i-1, j, k, t) )/dx );
    float dudz = ( ( UA4D(i, j, k, t) - UA4D(i, j, k-1, t) )/dz );
    TEM4D(i, j, k, t) = dudz - dwdx;
    if (k == 1) {
        TEM4D(i, j, 0, t) = dudz - dwdx;
    }
}

/* Compute the Z vorticity tendency due to the 6th order numerical diffusion */
__device__ void calc_zvortdiff_ten(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->diffu;
    float *vstag = data->diffv;
    float *dum0 = data->tem3;
    float dx = xf(i) - xf(i-1);
    float dy = yf(j) - yf(j-1);

    float dvdx = ( ( VA4D(i, j, k, t) - VA4D(i-1, j, k, t) )/dx);
    float dudy = ( ( UA4D(i, j, k, t) - UA4D(i, j-1, k, t) )/dy);
    TEM4D(i, j, k, t) = dvdx - dudy;
}

__device__ void calc_xvort_solenoid(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float cp = 1005.7;
    const float reps = 461.5 / 287.04;

    float dy = yf(j) - yf(j-1);
    float dz = zf(k) - zf(k-1);

    float *buf0 = data->pi;
    // dPi/dz
    float pi_upper = BUF4D(i, j, k+1, t);
    float pi_lower = BUF4D(i, j, k-1, t);
    float dpidz = ( (pi_upper - pi_lower) / ( 2*dz ) );
    // dPi/dy
    float dpidy = ( (BUF4D(i, j+1, k, t) - BUF4D(i, j-1, k, t)) / ( 2*dy ) );

    buf0 = data->thrhopert;
    float qvbar1 = grid->qv0[k+1];
    float qvbar2 = grid->qv0[k-1];
    float thbar1 = grid->th0[k+1]*(1.0+reps*qvbar1)/(1.0+qvbar1); 
    float thbar2 = grid->th0[k-1]*(1.0+reps*qvbar2)/(1.0+qvbar2); 
    // dthrho/dy
    float dthdy = ( (BUF4D(i, j+1, k, t) - BUF4D(i, j-1, k, t)) / ( 2*dy ) );

    // dthrho/dz
    float dthdz = ( ((BUF4D(i, j, k+1, t) + thbar1) - (BUF4D(i, j, k-1, t) + thbar2)) / ( 2*dz ) );

    // compute and save to the array
    buf0 = data->xvort_solenoid; 
    BUF4D(i, j, k, t) = -cp*(dthdy*dpidz - dthdz*dpidy); 
    if (k == 1) {
        // the d/dy terms are defined at k = 1,
        // go get those
        dpidy = ( ( BUF4D(i, j+1, 0, t) - BUF4D(i, j-1, 0, t) ) / (2*dy) );
        dthdy = ( (BUF4D(i, j+1, 0, t) - BUF4D(i, j-1, 0, t)) / ( 2*dy ) );
        BUF4D(i, j, 0, t) = -cp*(dthdy*dpidz - dthdz*dpidy); 
    }
}

__device__ void calc_yvort_solenoid(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float cp = 1005.7;
    const float reps = 461.5 / 287.04;
    float dx = xf(i) - xf(i-1);
    float dz = zf(k) - zf(k-1);

    float *buf0 = data->pi;
    // dPi/dz
    float pi_upper = BUF4D(i, j, k+1, t);
    float pi_lower = BUF4D(i, j, k-1, t);
    float dpidz = ( (pi_upper - pi_lower) / ( 2*dz ) );
    // dPi/dx
    float dpidx = ( (BUF4D(i+1, j, k, t) - BUF4D(i-1, j, k, t)) / ( 2*dx ) );

    buf0 = data->thrhopert;
    float qvbar1 = grid->qv0[k+1];
    float qvbar2 = grid->qv0[k-1];
    float thbar1 = grid->th0[k+1]*(1.0+reps*qvbar1)/(1.0+qvbar1); 
    float thbar2 = grid->th0[k-1]*(1.0+reps*qvbar2)/(1.0+qvbar2); 
    // dthrho/dx
    float dthdx = ( (BUF4D(i+1, j, k, t) - BUF4D(i-1, j, k, t)) / ( 2*dx ) );

    // dthrho/dz
    float dthdz = ( ((BUF4D(i, j, k+1, t) + thbar1) - (BUF4D(i, j, k-1, t) + thbar2)) / ( 2*dz ) );

    // compute and save to the array
    buf0 = data->yvort_solenoid; 
    BUF4D(i, j, k, t) = -cp*(dthdz*dpidx - dthdx*dpidz); 
    if (k == 1) {
        // the d/dx terms are defined at k = 1,
        // go get those
        dpidx = ( (BUF4D(i+1, j, 0, t) - BUF4D(i-1, j, 0, t)) / ( 2*dx ) );
        dthdx = ( (BUF4D(i+1, j, 0, t) - BUF4D(i-1, j, 0, t)) / ( 2*dx ) );
        BUF4D(i, j, 0, t) = -cp*(dthdz*dpidx - dthdx*dpidz); 
    }
}

__device__ void calc_zvort_solenoid(datagrid *grid, model_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float cp = 1005.7;
    float dx = xf(i) - xf(i-1);
    float dy = yf(j) - yf(j-1);

    float *buf0 = data->pi;
    // dPi/dx
    float dpidx = ( (BUF4D(i+1, j, k, t) - BUF4D(i-1, j, k, t)) / ( 2*dx ) );
    // dPi/dy
    float dpidy = ( (BUF4D(i, j+1, k, t) - BUF4D(i, j-1, k, t)) / ( 2*dy ) );

    buf0 = data->thrhopert;
    // dthrho/dx
    float dthdx = ( (BUF4D(i+1, j, k, t) - BUF4D(i-1, j, k, t)) / ( 2*dx ) );

    // dthrho/dy
    float dthdy = ( (BUF4D(i, j+1, k, t) - BUF4D(i, j-1, k, t)) / ( 2*dy ) );

    // compute and save to the array
    buf0 = data->yvort_solenoid; 
    BUF4D(i, j, k, t) = -cp*(dthdx*dpidy - dthdy*dpidx); 
}

/* When doing the parcel trajectory integration, George Bryan does
   some fun stuff with the lower boundaries/ghost zones of the arrays, presumably
   to prevent the parcels from exiting out the bottom of the domain
   or experience artificial values. This sets the ghost zone values. */
__global__ void applyMomentumBC(float *ustag, float *vstag, float *wstag, int NX, int NY, int NZ, int tStart, int tEnd) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    // this is done for easy comparison to CM1 code
    int ni = NX; int nj = NY;

    // this is a lower boundary condition, so only when k is 0
    // also this is on the u staggered mesh
    if (( j < nj+1) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the u stagger macro to handle the
            // proper indexing
            UA4D(i, j, 0, tidx) = UA4D(i, j, 1, tidx);
        }
    }
    
    // do the same but now on the v staggered grid
    if (( j < nj+1) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the v stagger macro to handle the
            // proper indexing
            VA4D(i, j, 0, tidx) = VA4D(i, j, 1, tidx);
        }
    }

    // do the same but now on the w staggered grid
    if (( j < nj+1) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the w stagger macro to handle the
            // proper indexing
            WA4D(i, j, 0, tidx) = -1*WA4D(i, j, 2, tidx);
        }
    }
}


__global__ void doTurbVort(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    // KELTON. FOR THE LOVE OF ALL THAT IS GOOD.
    // STOP CHANGING THE INDEX CHECK CONDITIONS. 
    // YOU'VE DONE THIS LIKE 5 TIMES NOW AND
    // CAUSE SEG FAULTS EVERY TIME. LEARN YOUR 
    // LESSON ALREADY. THIS WORKS. DON'T BREAK.
    // BAD.

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY+1) && (k > 0) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvortturb_ten(grid, data, idx_4D, NX, NY, NZ);
        }
    }

    if ((i < NX+1) && (j < NY) && (k > 0) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_yvortturb_ten(grid, data, idx_4D, NX, NY, NZ);
        }
    }
    if ((i < NX+1) && (j < NY+1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvortturb_ten(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

__global__ void doDiffVort(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    // KELTON. FOR THE LOVE OF ALL THAT IS GOOD.
    // STOP CHANGING THE INDEX CHECK CONDITIONS. 
    // YOU'VE DONE THIS LIKE 5 TIMES NOW AND
    // CAUSE SEG FAULTS EVERY TIME. LEARN YOUR 
    // LESSON ALREADY. THIS WORKS. DON'T BREAK.
    // BAD.

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY+1) && (k > 0) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvortdiff_ten(grid, data, idx_4D, NX, NY, NZ);
        }
    }

    if ((i < NX+1) && (j < NY) && (k > 0) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_yvortdiff_ten(grid, data, idx_4D, NX, NY, NZ);
        }
    }
    if ((i < NX+1) && (j < NY+1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvortdiff_ten(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

__global__ void calcpi(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX+1) && (j < NY+1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_pi(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

__global__ void calcvort(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    // KELTON. FOR THE LOVE OF ALL THAT IS GOOD.
    // STOP CHANGING THE INDEX CHECK CONDITIONS. 
    // YOU'VE DONE THIS LIKE 5 TIMES NOW AND
    // CAUSE SEG FAULTS EVERY TIME. LEARN YOUR 
    // LESSON ALREADY. THIS WORKS. DON'T BREAK.
    // BAD.

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY+1) && (k > 0) && (k < NZ)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort(grid, data, idx_4D, NX, NY, NZ);
        }
    }

    if ((i < NX+1) && (j < NY) && (k > 0) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_yvort(grid, data, idx_4D, NX, NY, NZ);
        }
    }
    if ((i < NX+1) && (j < NY+1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvort(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the Vorticity Equation */
__global__ void calcvortstretch(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY) && (k < NZ)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort_stretch(grid, data, idx_4D, NX, NY, NZ);
        }
    }

    if ((i < NX) && (j < NY) && (k < NZ)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_yvort_stretch(grid, data, idx_4D, NX, NY, NZ);
        }
    }
    if ((i < NX) && (j < NY) && (k < NZ)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvort_stretch(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the Vorticity Equation */
__global__ void calcxvorttilt(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY) && (k > 0) && (k < NZ)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort_tilt(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the Vorticity Equation */
__global__ void calcyvorttilt(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY) && (k > 0) && (k < NZ)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_yvort_tilt(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the Vorticity Equation */
__global__ void calczvorttilt(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY) && (k > 0) && (k < NZ)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvort_tilt(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the pressure-volume solenoid term */
__global__ void calcvortsolenoid(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    // Even though there are NZ points, it's a center difference
    // and we reach out NZ+1 points to get the derivatives
    if ((i < NX-1) && (j < NY-1) && (k < NZ) && ( i > 0 ) && (j > 0)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvort_solenoid(grid, data, idx_4D, NX, NY, NZ);
        }
    }
    if ((i < NX-1) && (j < NY-1) && (k < NZ) && ( i > 0 ) && (j > 0) && (k > 0)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort_solenoid(grid, data, idx_4D, NX, NY, NZ);
            calc_yvort_solenoid(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Zero out the temporary arrays */
__global__ void zeroTemArrays(datagrid *grid, model_data *data, int tStart, int tEnd) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *dum0;
    if (( i < NX+1) && ( j < NY+1) && ( k < NZ+1)) {
        dum0 = data->tem1;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem2;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem3;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem4;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem5;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem6;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
    }
}


/* Average our vorticity values back to the scalar grid for interpolation
   to the parcel paths. We're able to do this in parallel by making use of
   the three temporary arrays allocated on our grid, which means that the
   xvort/yvort/zvort arrays will be averaged into tem1/tem2/tem3. After
   calling this kernel, you MUST set the new pointers appropriately. */
__global__ void doVortAvg(datagrid *grid, model_data *data, int tStart, int tEnd) {

    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;

    if ((i < NX) && (j < NY) && (k < NZ)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // average the temporary arrays into the result arrays
            dum0 = data->tem1;
            buf0 = data->xvort;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) +\
                                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem2;
            buf0 = data->yvort;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) +\
                                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem3;
            buf0 = data->zvort;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) +\
                                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );
        }
    }
}

__global__ void doTurbVortAvg(datagrid *grid, model_data *data, int tStart, int tEnd) {

    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;

    if ((i < NX) && (j < NY) && (k < NZ)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // average the temporary arrays into the result arrays
            dum0 = data->tem1;
            buf0 = data->turbxvort;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) +\
                                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem2;
            buf0 = data->turbyvort;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) +\
                                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem3;
            buf0 = data->turbzvort;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) +\
                                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );
        }
    }
}


__global__ void doDiffVortAvg(datagrid *grid, model_data *data, int tStart, int tEnd) {

    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;

    if ((i < NX) && (j < NY) && (k < NZ)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // average the temporary arrays into the result arrays
            dum0 = data->tem1;
            buf0 = data->diffxvort;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) +\
                                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem2;
            buf0 = data->diffyvort;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) +\
                                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem3;
            buf0 = data->diffzvort;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) +\
                                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );
        }
    }
}

/* Average the derivatives within the temporary arrays used to compute
   the tilting rate and then combine the terms into the final xvtilt
   array. It is assumed that the derivatives have been precomputed into
   the temporary arrays. */
__global__ void doXVortTiltAvg(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;
    float dudy,dudz;

    // We do the average for each array at a given point
    // and then finish the computation for the zvort tilt
    if ((i < NX) && (j < NY) && (k < NZ)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            dum0 = data->tem1;
            //dudy = TEM4D(i, j, k, tidx);
            dudy = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );

            dum0 = data->tem2;
            //dudz = TEM4D(i, j, k, tidx);
            dudz = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            buf0 = data->zvort;
            float zvort = BUF4D(i, j, k, tidx);
            buf0 = data->yvort;
            float yvort = BUF4D(i, j, k, tidx);

            buf0 = data->xvtilt;
            BUF4D(i, j, k, tidx) = zvort * dudz + yvort * dudy; 
        }
    }
}

/* Average the derivatives within the temporary arrays used to compute
   the tilting rate and then combine the terms into the final yvtilt
   array. It is assumed that the derivatives have been precomputed into
   the temporary arrays. */
__global__ void doYVortTiltAvg(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;
    float dvdx, dvdz;

    // We do the average for each array at a given point
    // and then finish the computation for the zvort tilt
    if ((i < NX) && (j < NY) && (k < NZ)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            dum0 = data->tem1;
            //dvdx = TEM4D(i, j, k, tidx);
            dvdx = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );

            dum0 = data->tem2;
            //dvdz = TEM4D(i, j, k, tidx);
            dvdz = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            buf0 = data->xvort;
            float xvort = BUF4D(i, j, k, tidx);
            buf0 = data->zvort;
            float zvort = BUF4D(i, j, k, tidx);

            buf0 = data->yvtilt;
            BUF4D(i, j, k, tidx) = xvort * dvdx + zvort * dvdz; 
        }
    }
}

/* Average the derivatives within the temporary arrays used to compute
   the tilting rate and then combine the terms into the final zvtilt
   array. It is assumed that the derivatives have been precomputed into
   the temporary arrays. */
__global__ void doZVortTiltAvg(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;
    float dwdx, dwdy;

    // We do the average for each array at a given point
    // and then finish the computation for the zvort tilt
    if ((i < NX) && (j < NY) && (k < NZ)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            dum0 = data->tem1;
            //dwdx = TEM4D(i, j, k, tidx);
            dwdx = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem2;
            dwdy = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );
            //dwdy = TEM4D(i, j, k, tidx);
            buf0 = data->xvort;
            float xvort = BUF4D(i, j, k, tidx);
            buf0 = data->yvort;
            float yvort = BUF4D(i, j, k, tidx);
            
            buf0 = data->zvtilt;
            BUF4D(i, j, k, tidx) = xvort * dwdx + yvort * dwdy; 
        }
    }
}
#endif
