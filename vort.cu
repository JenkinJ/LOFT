#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "datastructs.cu"
#include "macros.cpp"
#include "interp.cu"
#ifndef VORT_CU
#define VORT_CU

/* Compute the Exner function / nondimensionalized pressure */
__device__ void calc_pi(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    // This is p'
    float *buf0 = data->pres_4d_chunk;
    float pi = powf( BUF4D(i, j, k, t) / 1000., 0.28571426);
    buf0 = data->pi_4d_chunk;
    BUF4D(i, j, k, t) = pi;
}

/* Compute the x component of vorticity. After this is called by the calvort kernel, you must also run 
   the kernel for applying the lower boundary condition and then the kernel for averaging to the
   scalar grid. */
__device__ void calc_xvort(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *vstag = data->v_4d_chunk;
    float *wstag = data->w_4d_chunk;
    float *dum0 = data->tem1_4d_chunk;

    float dwdy = ( ( WA4D(i, j, k, t) - WA4D(i, j-1, k, t) )/grid->dy ) * VF(j);
    float dvdz = ( ( VA4D(i, j, k, t) - VA4D(i, j, k-1, t) )/grid->dz ) * MF(k);
    TEM4D(i, j, k, t) = dwdy - dvdz; 
}

/* Compute the y component of vorticity. After this is called by the calvort kernel, you must also run 
   the kernel for applying the lower boundary condition and then the kernel for averaging to the
   scalar grid. */
__device__ void calc_yvort(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *wstag = data->w_4d_chunk;
    float *dum0 = data->tem2_4d_chunk;

    float dwdx = ( ( WA4D(i, j, k, t) - WA4D(i-1, j, k, t) )/grid->dx ) * UF(i);
    float dudz = ( ( UA4D(i, j, k, t) - UA4D(i, j, k-1, t) )/grid->dz ) * MF(k);
    TEM4D(i, j, k, t) = dudz - dwdx;
}

/* Compute the z component of vorticity. After this is called by the calvort kernel, you must also run 
   the kernel for applying the lower boundary condition and then the kernel for averaging to the
   scalar grid. */
__device__ void calc_zvort(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *vstag = data->v_4d_chunk;
    float *dum0 = data->tem3_4d_chunk;

    float dvdx = ( ( VA4D(i, j, k, t) - VA4D(i-1, j, k, t) )/grid->dx) * UF(i);
    float dudy = ( ( UA4D(i, j, k, t) - UA4D(i, j-1, k, t) )/grid->dy) * VF(j);
    TEM4D(i, j, k, t) = dvdx - dudy;
}

/* Compute the X component of vorticity tendency due
   to tilting Y and Z components into the X direction */
__device__ void calc_xvort_tilt(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *vstag = data->v_4d_chunk;
    float *wstag = data->w_4d_chunk;

    // dudy in tem1
    float *dum0 = data->tem1_4d_chunk;
    TEM4D(i, j, k, t) = ( ( UA4D(i, j, k, t) - UA4D(i, j-1, k, t) ) / grid->dy ) * VF(j);

    // dwdx in tem2
    dum0 = data->tem2_4d_chunk;
    TEM4D(i, j, k, t) = ( ( WA4D(i, j, k, t) - WA4D(i-1, j, k, t) ) / grid->dx ) * UF(i);

    // dudz in tem3
    dum0 = data->tem3_4d_chunk;
    TEM4D(i, j, k, t) = ( ( UA4D(i, j, k, t) - UA4D(i, j, k-1, t) ) / grid->dz ) * MF(k);

    // dvdx in tem4
    dum0 = data->tem4_4d_chunk;
    TEM4D(i, j, k, t) = ( ( VA4D(i, j, k, t) - VA4D(i-1, j, k, t) ) / grid->dx ) * UF(i);
}

/* Compute the Y component of vorticity tendency due
   to tilting X and Z components into the X direction */
__device__ void calc_yvort_tilt(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *vstag = data->v_4d_chunk;
    float *wstag = data->w_4d_chunk;
    
    // dvdx in tem1
    float *dum0 = data->tem1_4d_chunk;
    TEM4D(i, j, k, t) = ( ( VA4D(i, j, k, t) - VA4D(i-1, j, k, t) ) / grid->dx ) * UF(i);

    // dwdy in tem2
    dum0 = data->tem2_4d_chunk;
    TEM4D(i, j, k, t) = ( ( WA4D(i, j, k, t) - WA4D(i, j-1, k, t) ) / grid->dy ) * VF(j);

    // dvdz in tem3
    dum0 = data->tem3_4d_chunk;
    TEM4D(i, j, k, t) = ( ( VA4D(i, j, k, t) - VA4D(i, j, k-1, t) ) / grid->dz ) * MF(k);

    // dudy in tem4
    dum0 = data->tem4_4d_chunk;
    TEM4D(i, j, k, t) = ( ( UA4D(i, j, k, t) - UA4D(i, j-1, k, t) ) / grid->dy ) * VF(j);
}

/* Compute the Z component of vorticity tendency due
   to tilting X and Y components into the X direction */
__device__ void calc_zvort_tilt(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *vstag = data->v_4d_chunk;
    float *wstag = data->w_4d_chunk;

    // Compute dw/dx and put it in the tem1 array. The derivatives
    // land on weird places so we have to average each derivative back
    // to the scalar grid, resulting in this clunky approach
    float *dum0 = data->tem1_4d_chunk;
    TEM4D(i, j, k, t) = ( ( WA4D(i, j, k, t) - WA4D(i-1, j, k, t) ) / grid->dx ) * UF(i);

    // put dv/dz in tem2
    dum0 = data->tem2_4d_chunk;
    TEM4D(i, j, k, t) = ( ( VA4D(i, j, k, t) - VA4D(i, j, k-1, t) ) / grid->dz ) * MF(k);

    // put dw/dy in tem3
    dum0 = data->tem3_4d_chunk;
    TEM4D(i, j, k, t) = ( ( WA4D(i, j, k, t) - WA4D(i, j-1, k, t) ) / grid->dy ) * VF(j);

    // put du/dz in tem4
    dum0 = data->tem4_4d_chunk;
    TEM4D(i, j, k, t) = ( ( UA4D(i, j, k, t) - UA4D(i, j, k-1, t) ) / grid->dz ) * MF(k);
}

/* Compute the X component of vorticity tendency due
   to stretching of the vorticity along the X axis. */
__device__ void calc_xvort_stretch(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *vstag = data->v_4d_chunk;
    float *wstag = data->w_4d_chunk;
    float *xvort = data->xvort_4d_chunk;
    float *xvort_stretch = data->xvstretch_4d_chunk;

    // this stencil conveniently lands itself on the scalar grid,
    // so we won't have to worry about doing any averaging. I think.
    float *buf0 = xvort;
    float xv = BUF4D(i, j, k, t);
    float dvdy = ( ( VA4D(i, j, k, t) - VA4D(i, j-1, k, t) )/grid->dy) * VF(j);
    float dwdz = ( ( WA4D(i, j, k, t) - WA4D(i, j, k-1, t) )/grid->dz) * MF(k);

    buf0 = xvort_stretch;
    BUF4D(i, j, k, t) = -1.0*xv*( dvdy + dwdz);

}

/* Compute the Y component of vorticity tendency due
   to stretching of the vorticity along the Y axis. */
__device__ void calc_yvort_stretch(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *wstag = data->w_4d_chunk;
    float *yvort = data->yvort_4d_chunk;
    float *yvort_stretch = data->yvstretch_4d_chunk;

    // this stencil conveniently lands itself on the scalar grid,
    // so we won't have to worry about doing any averaging. I think.
    float *buf0 = yvort;
    float yv = BUF4D(i, j, k, t);
    float dudx = ( ( UA4D(i, j, k, t) - UA4D(i-1, j, k, t) )/grid->dx) * UF(i);
    float dwdz = ( ( WA4D(i, j, k, t) - WA4D(i, j, k-1, t) )/grid->dz) * MF(k);

    buf0 = yvort_stretch;
    BUF4D(i, j, k, t) = -1.0*yv*( dudx + dwdz);
}

/* Compute the Z component of vorticity tendency due
   to stretching of the vorticity along the Z axis. */
__device__ void calc_zvort_stretch(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *vstag = data->v_4d_chunk;
    float *zvort = data->zvort_4d_chunk;
    float *zvort_stretch = data->zvstretch_4d_chunk;

    // this stencil conveniently lands itself on the scalar grid,
    // so we won't have to worry about doing any averaging. I think.
    float *buf0 = zvort;
    float zv = BUF4D(i, j, k, t);
    float dudx = ( ( UA4D(i, j, k, t) - UA4D(i-1, j, k, t) )/grid->dx) * UF(i);
    float dvdy = ( ( VA4D(i, j, k, t) - VA4D(i, j-1, k, t) )/grid->dy) * VF(j);

    buf0 = zvort_stretch;
    BUF4D(i, j, k, t) = -1.0*zv*( dudx + dvdy);
}

/* Compute the X vorticity tendency due to the turbulence closure scheme */
__device__ void calc_xvortturb_ten(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *vstag = data->turbv_4d_chunk;
    float *wstag = data->turbw_4d_chunk;
    float *dum0 = data->tem1_4d_chunk;

    float dwdy = ( ( WA4D(i, j, k, t) - WA4D(i, j-1, k, t) )/grid->dy ) * VF(j);
    float dvdz = ( ( VA4D(i, j, k, t) - VA4D(i, j, k-1, t) )/grid->dz ) * MF(k);
    TEM4D(i, j, k, t) = dwdy - dvdz; 
}

/* Compute the Y vorticity tendency due to the turbulence closure scheme */
__device__ void calc_yvortturb_ten(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->turbu_4d_chunk;
    float *wstag = data->turbw_4d_chunk;
    float *dum0 = data->tem2_4d_chunk;

    float dwdx = ( ( WA4D(i, j, k, t) - WA4D(i-1, j, k, t) )/grid->dx ) * UF(i);
    float dudz = ( ( UA4D(i, j, k, t) - UA4D(i, j, k-1, t) )/grid->dz ) * MF(k);
    TEM4D(i, j, k, t) = dudz - dwdx;
}

/* Compute the Z vorticity tendency due to the turbulence closure scheme */
__device__ void calc_zvortturb_ten(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->turbu_4d_chunk;
    float *vstag = data->turbv_4d_chunk;
    float *dum0 = data->tem3_4d_chunk;

    float dvdx = ( ( VA4D(i, j, k, t) - VA4D(i-1, j, k, t) )/grid->dx) * UF(i);
    float dudy = ( ( UA4D(i, j, k, t) - UA4D(i, j-1, k, t) )/grid->dy) * VF(j);
    TEM4D(i, j, k, t) = dvdx - dudy;
}

__device__ void calc_zvort_solenoid(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    // We can use p' here with no problems
    float *dum0 = data->pres_4d_chunk;
    // dP/dx
    float dpdx = ( (TEM4D(i+1, j, k, t) - TEM4D(i-1, j, k, t)) / ( 2*grid->dx ) ) * UH(i);
    // dP/dy
    float dpdy = ( (TEM4D(i, j+1, k, t) - TEM4D(i, j-1, k, t)) / ( 2*grid->dy ) ) * VH(j);

    dum0 = data->rho_4d_chunk;
    // dRho/dy
    // We use k-1 for the base state grid because it does not
    // have a lower ghost zone, so 0 corresponds to the surface 
    // instead of the ghost zone value
    float rho2 = TEM4D(i, j+1, k, t) + grid->rho0[k-1];
    float rho1 = TEM4D(i, j-1, k, t) + grid->rho0[k-1];
    float dalphady = ( ( (1./rho2) - (1./rho1) ) / ( 2*grid->dy ) ) * VH(j);

    // dRho/dx
    rho2 = TEM4D(i+1, j, k, t) + grid->rho0[k-1];
    rho1 = TEM4D(i-1, j, k, t) + grid->rho0[k-1];
    float dalphadx = ( ( (1./rho2) - (1./rho1) ) / ( 2*grid->dx ) ) * UH(i);

    // compute and save to the array
    float *buf0 = data->zvort_solenoid_4d_chunk; 
    BUF4D(i, j, k, t) = (dpdx*dalphady) - (dpdy*dalphadx);
}

/* When doing the parcel trajectory integration, George Bryan does
   some fun stuff with the lower boundaries/ghost zones of the arrays, presumably
   to prevent the parcels from exiting out the bottom of the domain
   or experience artificial values. This sets the ghost zone values. */
__global__ void applyMomentumBC(float *ustag, float *vstag, float *wstag, int NX, int NY, int NZ, int tStart, int tEnd) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    // this is done for easy comparison to CM1 code
    int ni = NX; int nj = NY;

    // this is a lower boundary condition, so only when k is 0
    // also this is on the u staggered mesh
    if (( j < nj+1) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the u stagger macro to handle the
            // proper indexing
            UA4D(i, j, 0, tidx) = UA4D(i, j, 1, tidx);
        }
    }
    
    // do the same but now on the v staggered grid
    if (( j < nj+1) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the v stagger macro to handle the
            // proper indexing
            VA4D(i, j, 0, tidx) = VA4D(i, j, 1, tidx);
        }
    }

    // do the same but now on the w staggered grid
    if (( j < nj+1) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the w stagger macro to handle the
            // proper indexing
            WA4D(i, j, 0, tidx) = -1*WA4D(i, j, 2, tidx);
        }
    }
}


__global__ void doTurbVort(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    // KELTON. FOR THE LOVE OF ALL THAT IS GOOD.
    // STOP CHANGING THE INDEX CHECK CONDITIONS. 
    // YOU'VE DONE THIS LIKE 5 TIMES NOW AND
    // CAUSE SEG FAULTS EVERY TIME. LEARN YOUR 
    // LESSON ALREADY. THIS WORKS. DON'T BREAK.
    // BAD.

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY+1) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvortturb_ten(grid, data, idx_4D, NX, NY, NZ);
        }
    }

    if ((i < NX+1) && (j < NY) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_yvortturb_ten(grid, data, idx_4D, NX, NY, NZ);
        }
    }
    if ((i <= NX+1) && (j <= NY+1) && (k > 0) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvortturb_ten(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

__global__ void calcvort(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    // KELTON. FOR THE LOVE OF ALL THAT IS GOOD.
    // STOP CHANGING THE INDEX CHECK CONDITIONS. 
    // YOU'VE DONE THIS LIKE 5 TIMES NOW AND
    // CAUSE SEG FAULTS EVERY TIME. LEARN YOUR 
    // LESSON ALREADY. THIS WORKS. DON'T BREAK.
    // BAD.

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY+1) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort(grid, data, idx_4D, NX, NY, NZ);
        }
    }

    if ((i < NX+1) && (j < NY) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_yvort(grid, data, idx_4D, NX, NY, NZ);
        }
    }
    if ((i <= NX+1) && (j <= NY+1) && (k > 0) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvort(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the Vorticity Equation */
__global__ void calcvortstretch(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY+1) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort_stretch(grid, data, idx_4D, NX, NY, NZ);
        }
    }

    if ((i < NX+1) && (j < NY) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_yvort_stretch(grid, data, idx_4D, NX, NY, NZ);
        }
    }
    if ((i < NX+1) && (j < NY+1) && (k > 0) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvort_stretch(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the Vorticity Equation */
__global__ void calcxvorttilt(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX+1) && (j < NY+1) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort_tilt(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the Vorticity Equation */
__global__ void calcyvorttilt(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX+1) && (j < NY+1) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_yvort_tilt(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the Vorticity Equation */
__global__ void calczvorttilt(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX+1) && (j < NY+1) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvort_tilt(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the pressure-volume solenoid term */
__global__ void calczvortsolenoid(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    // Even though there are NZ points, it's a center difference
    // and we reach out NZ+1 points to get the derivatives
    if ((i < NX) && (j < NY) && (k < NZ) && ( i > 0 ) && (j > 0) && (k >= 1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvort_solenoid(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Zero out the temporary arrays */
__global__ void zeroTemArrays(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *dum0;
    if (( i < NX+1) && ( j < NY+1) && ( k < NZ+1)) {
        dum0 = data->tem1_4d_chunk;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem2_4d_chunk;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem3_4d_chunk;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem4_4d_chunk;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem5_4d_chunk;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem6_4d_chunk;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
    }
}


/* Apply the free-slip lower boundary condition to the vorticity field. */
__global__ void applyVortBC(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *dum0;

    // NOTE: Not sure if need to use BUF4D or TEM4D. The size of the array
    // will for sure be respected by BUF4D but unsure if it even matters here.

    // This is a lower boundary condition, so only when k is 0.
    // Start with xvort. 
    if (( i < NX+1) && ( j < NY+1) && ( k == 1)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // at this stage, xvort is in the tem1 array
            dum0 = data->tem1_4d_chunk;
            TEM4D(i, j, 1, tidx) = TEM4D(i, j, 2, tidx);
            // at this stage, yvort is in the tem2 array
            dum0 = data->tem2_4d_chunk;
            TEM4D(i, j, 1, tidx) = TEM4D(i, j, 2, tidx);
            // I'm technically ignoring an upper boundary condition
            // here, but we never really guarantee that we're at
            // the top of the model domain because we do a lot of subsetting.
            // So, for now, we assume we're nowehere near the top. 
        }
    }
}

/* Apply the free-slip lower boundary condition to the vorticity field. */
__global__ void applyVortTendBC(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *dum0;

    // NOTE: Not sure if need to use BUF4D or TEM4D. The size of the array
    // will for sure be respected by BUF4D but unsure if it even matters here.

    // This is a lower boundary condition, so only when k is 0.
    // Start with xvort. 
    if (( i < NX+1) && ( j < NY+1) && ( k == 1)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            dum0 = data->tem1_4d_chunk;
            TEM4D(i, j, 1, tidx) = TEM4D(i, j, 2, tidx);
            dum0 = data->tem2_4d_chunk;
            TEM4D(i, j, 1, tidx) = TEM4D(i, j, 2, tidx);
            dum0 = data->tem3_4d_chunk;
            TEM4D(i, j, 1, tidx) = TEM4D(i, j, 2, tidx);
            dum0 = data->tem4_4d_chunk;
            TEM4D(i, j, 1, tidx) = TEM4D(i, j, 2, tidx);
            // I'm technically ignoring an upper boundary condition
            // here, but we never really guarantee that we're at
            // the top of the model domain because we do a lot of subsetting.
            // So, for now, we assume we're nowehere near the top. 
        }
    }
}

/* Average our vorticity values back to the scalar grid for interpolation
   to the parcel paths. We're able to do this in parallel by making use of
   the three temporary arrays allocated on our grid, which means that the
   xvort/yvort/zvort arrays will be averaged into tem1/tem2/tem3. After
   calling this kernel, you MUST set the new pointers appropriately. */
__global__ void doVortAvg(datagrid *grid, integration_data *data, int tStart, int tEnd) {

    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;

    if ((i < NX) && (j < NY) && (k < NZ) && (k > 0)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // average the temporary arrays into the result arrays
            dum0 = data->tem1_4d_chunk;
            buf0 = data->xvort_4d_chunk;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) +\
                                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem2_4d_chunk;
            buf0 = data->yvort_4d_chunk;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) +\
                                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem3_4d_chunk;
            buf0 = data->zvort_4d_chunk;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) +\
                                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );
        }
    }
}

__global__ void doTurbVortAvg(datagrid *grid, integration_data *data, int tStart, int tEnd) {

    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;

    if ((i < NX) && (j < NY) && (k < NZ) && (k > 0)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // average the temporary arrays into the result arrays
            dum0 = data->tem1_4d_chunk;
            buf0 = data->turbxvort_4d_chunk;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) +\
                                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem2_4d_chunk;
            buf0 = data->turbyvort_4d_chunk;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) +\
                                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem3_4d_chunk;
            buf0 = data->turbzvort_4d_chunk;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) +\
                                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );
        }
    }
}

/* Average the derivatives within the temporary arrays used to compute
   the tilting rate and then combine the terms into the final xvtilt
   array. It is assumed that the derivatives have been precomputed into
   the temporary arrays. */
__global__ void doXVortTiltAvg(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;
    float dudy, dwdx, dudz, dvdx;

    // We do the average for each array at a given point
    // and then finish the computation for the zvort tilt
    if ((i < NX) && (j < NY) && (k < NZ) && (k > 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            dum0 = data->tem1_4d_chunk;
            dudy = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );

            dum0 = data->tem2_4d_chunk;
            dwdx = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem3_4d_chunk;
            dudz = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem4_4d_chunk;
            dvdx = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );

            buf0 = data->xvtilt_4d_chunk;
            BUF4D(i, j, k, tidx) = -1.0*((dudy*dwdx) - (dudz*dvdx));
        }
    }
}

/* Average the derivatives within the temporary arrays used to compute
   the tilting rate and then combine the terms into the final yvtilt
   array. It is assumed that the derivatives have been precomputed into
   the temporary arrays. */
__global__ void doYVortTiltAvg(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;
    float dvdx, dwdy, dvdz, dudy;

    // We do the average for each array at a given point
    // and then finish the computation for the zvort tilt
    if ((i < NX) && (j < NY) && (k < NZ) && (k > 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            dum0 = data->tem1_4d_chunk;
            dvdx = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );

            dum0 = data->tem2_4d_chunk;
            dwdy = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem3_4d_chunk;
            dvdz = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem4_4d_chunk;
            dudy = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );

            buf0 = data->yvtilt_4d_chunk;
            BUF4D(i, j, k, tidx) = -1.0*((dvdx*dwdy) - (dvdz*dudy));
        }
    }
}

/* Average the derivatives within the temporary arrays used to compute
   the tilting rate and then combine the terms into the final zvtilt
   array. It is assumed that the derivatives have been precomputed into
   the temporary arrays. */
__global__ void doZVortTiltAvg(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;
    float dwdx, dvdz, dwdy, dudz;

    // We do the average for each array at a given point
    // and then finish the computation for the zvort tilt
    if ((i < NX) && (j < NY) && (k < NZ) && (k > 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            dum0 = data->tem1_4d_chunk;
            dwdx = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem2_4d_chunk;
            dvdz = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem3_4d_chunk;
            dwdy = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem4_4d_chunk;
            dudz = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            buf0 = data->zvtilt_4d_chunk;
            BUF4D(i, j, k, tidx) = (dwdy*dudz)-(dwdx*dvdz);
        }
    }
}


#endif
