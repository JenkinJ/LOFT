#include <vector>
#include "macros.cpp"
#include "datastructs.h"
#include <iostream>
#ifndef DATASTRUCTS
#define DATASTRUCTS
using namespace std;

/* Allocate memory on the CPU and GPU for a grid. There are times,
    like for various MPI ranks, that you don't want to do this on both.
    See the similar function for doing this on just the CPU */
datagrid* allocate_grid_managed( int X0, int X1, int Y0, int Y1, int Z0, int Z1 ) {
    datagrid *grid;
    long NX, NY, NZ;

    hipMallocManaged(&grid, sizeof(datagrid));
    grid->X0 = X0; grid->X1 = X1;
    grid->Y0 = Y0; grid->Y1 = Y1;
    grid->Z0 = Z0; grid->Z1 = Z1;

	NX = grid->X1 - grid->X0 + 1;
	NY = grid->Y1 - grid->Y0 + 1;
	NZ = grid->Z1 - grid->Z0 + 1;

    // set the grid attributes
    grid->NX = NX;
    grid->NY = NY;
    grid->NZ = NZ;

    // allocage grid arrays
    hipMallocManaged(&(grid->xf), (NX+1)*sizeof(float));
    hipMallocManaged(&(grid->xh), NX*sizeof(float));

    hipMallocManaged(&(grid->yf), (NY+1)*sizeof(float));
    hipMallocManaged(&(grid->yh), NY*sizeof(float));

    hipMallocManaged(&(grid->zf), (NZ+1)*sizeof(float));
    hipMallocManaged(&(grid->zh), NZ*sizeof(float));

    hipMallocManaged(&(grid->uf), (NX+2)*sizeof(float));
    hipMallocManaged(&(grid->uh), (NX+2)*sizeof(float));

    hipMallocManaged(&(grid->vf), (NY+2)*sizeof(float));
    hipMallocManaged(&(grid->vh), (NY+2)*sizeof(float));

    hipMallocManaged(&(grid->mf), (NZ+2)*sizeof(float));
    hipMallocManaged(&(grid->mh), (NZ+2)*sizeof(float));

    // allocate base state arrays
    hipMallocManaged(&(grid->qv0), NZ*sizeof(float));
    hipMallocManaged(&(grid->th0), NZ*sizeof(float));
    hipMallocManaged(&(grid->rho0), NZ*sizeof(float));
    hipDeviceSynchronize();

    return grid;
}

/* Allocate arrays only on the CPU for the grid. This is important
   for using with MPI, as only 1 rank should be allocating memory
   on the GPU */
datagrid* allocate_grid_cpu( int X0, int X1, int Y0, int Y1, int Z0, int Z1 ) {
    datagrid *grid = new datagrid();
    long NX, NY, NZ;

    grid->X0 = X0; grid->X1 = X1;
    grid->Y0 = Y0; grid->Y1 = Y1;
    grid->Z0 = Z0; grid->Z1 = Z1;

	NX = grid->X1 - grid->X0 + 1;
	NY = grid->Y1 - grid->Y0 + 1;
	NZ = grid->Z1 - grid->Z0 + 1;

    // set the grid attributes
    grid->NX = NX;
    grid->NY = NY;
    grid->NZ = NZ;

    // allocage grid arrays
    grid->xf = new float[NX+1];
    grid->xh = new float[NX];

    grid->yf = new float[NY+1];
    grid->yh = new float[NY];

    grid->zf = new float[NZ+1];
    grid->zh = new float[NZ];

    grid->uf = new float[NX+2];
    grid->uh = new float[NX+2];

    grid->vf = new float[NY+2];
    grid->vh = new float[NY+2];

    grid->mf = new float[NZ+2];
    grid->mh = new float[NZ+2];

    // allocate base state arrays
    grid->qv0 = new float[NZ];
    grid->th0 = new float[NZ];
    grid->rho0 = new float[NZ];

    return grid;
}

/* Deallocate all of the arrays in the 
   struct for both the GPU and CPU */
void deallocate_grid_managed(datagrid *grid) {
    hipFree(grid->xf);
    hipFree(grid->xh);
    hipFree(grid->yf);
    hipFree(grid->yh);
    hipFree(grid->zf);
    hipFree(grid->zh);
    hipFree(grid->uf);
    hipFree(grid->uh);
    hipFree(grid->vf);
    hipFree(grid->vh);
    hipFree(grid->mf);
    hipFree(grid->mh);
    hipFree(grid->rho0);
    hipFree(grid->th0);
    hipFree(grid->qv0);
    hipDeviceSynchronize();
}

/* Deallocate all of the arrays in the
   struct only for the CPU */
void deallocate_grid_cpu(datagrid *grid) {
    delete[] grid->xf;
    delete[] grid->xh;
    delete[] grid->yf;
    delete[] grid->yh;
    delete[] grid->zf;
    delete[] grid->zh;
    delete[] grid->uf;
    delete[] grid->uh;
    delete[] grid->vf;
    delete[] grid->vh;
    delete[] grid->mf;
    delete[] grid->mh;
    delete[] grid->rho0;
    delete[] grid->th0;
    delete[] grid->qv0;
}

/* Allocate arrays for parcel info on both the CPU and GPU.
   This function should only be called by MPI Rank 0, so
   be sure to use the CPU function for Rank >= 1. */
parcel_pos* allocate_parcels_managed(int NX, int NY, int NZ, int nTotTimes) {
    int nParcels = NX*NY*NZ;
    parcel_pos *parcels;
    // create the struct on both the GPU and the CPU.
    hipMallocManaged(&parcels, sizeof(parcel_pos));

    // allocate memory for the parcels
    // we are integrating for the entirety 
    // of the simulation.
    hipMallocManaged(&(parcels->xpos), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->ypos), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->zpos), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclu), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclv), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclw), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclxvort), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclyvort), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclzvort), nParcels*nTotTimes*sizeof(float)); 
    // set the static variables
    parcels->nParcels = nParcels;
    parcels->nTimes = nTotTimes;
    hipDeviceSynchronize();

    return parcels;
}

/* Allocate arrays only on the CPU for the grid. This is important
   for using with MPI, as only 1 rank should be allocating memory
   on the GPU */
parcel_pos* allocate_parcels_cpu(int NX, int NY, int NZ, int nTotTimes) {
    int nParcels = NX*NY*NZ;
    parcel_pos *parcels = new parcel_pos();

    // allocate memory for the parcels
    // we are integrating for the entirety 
    // of the simulation.
    parcels->xpos = new float[nParcels*nTotTimes]; 
    parcels->ypos = new float[nParcels*nTotTimes]; 
    parcels->zpos = new float[nParcels*nTotTimes]; 
    parcels->pclu = new float[nParcels*nTotTimes]; 
    parcels->pclv = new float[nParcels*nTotTimes]; 
    parcels->pclw = new float[nParcels*nTotTimes]; 
    parcels->pclxvort = new float[nParcels*nTotTimes]; 
    parcels->pclyvort = new float[nParcels*nTotTimes]; 
    parcels->pclzvort = new float[nParcels*nTotTimes]; 
    // set the static variables
    parcels->nParcels = nParcels;
    parcels->nTimes = nTotTimes;

    return parcels;
}

/* Deallocate parcel arrays on both the CPU and the
   GPU */
void deallocate_parcels_managed(parcel_pos *parcels) {
    hipFree(parcels->xpos);
    hipFree(parcels->ypos);
    hipFree(parcels->zpos);
    hipFree(parcels->pclu);
    hipFree(parcels->pclv);
    hipFree(parcels->pclw);
    hipFree(parcels->pclxvort);
    hipFree(parcels->pclyvort);
    hipFree(parcels->pclzvort);
    hipFree(parcels);
    hipDeviceSynchronize();
}

/* Deallocate parcel arrays only on the CPU */
void deallocate_parcels_cpu(parcel_pos *parcels) {
    delete[] parcels->xpos;
    delete[] parcels->ypos;
    delete[] parcels->zpos;
    delete[] parcels->pclu;
    delete[] parcels->pclv;
    delete[] parcels->pclw;
    delete[] parcels->pclxvort;
    delete[] parcels->pclyvort;
    delete[] parcels->pclzvort;
    delete[] parcels;
}

/* Allocate the struct of 4D arrays that store
   fields for integration and calculation. This
   only ever gets called by Rank 0, so there 
   should be no need for a CPU counterpart. */
integration_data* allocate_integration_managed(int bufsize) {
    integration_data *data;
    // create the struct on both the GPU and the CPU.
    hipMallocManaged(&data, sizeof(integration_data));

    // allocate the arrays in the struct
    hipMallocManaged(&(data->u_4d_chunk), bufsize*sizeof(float));
    hipMallocManaged(&(data->v_4d_chunk), bufsize*sizeof(float));
    hipMallocManaged(&(data->w_4d_chunk), bufsize*sizeof(float));
    hipMallocManaged(&(data->pres_4d_chunk), bufsize*sizeof(float));
    hipMallocManaged(&(data->th_4d_chunk), bufsize*sizeof(float));
    hipMallocManaged(&(data->rho_4d_chunk), bufsize*sizeof(float));
    hipMallocManaged(&(data->khh_4d_chunk), bufsize*sizeof(float));
    hipMallocManaged(&(data->xvort_4d_chunk), bufsize*sizeof(float));
    hipMallocManaged(&(data->yvort_4d_chunk), bufsize*sizeof(float));
    hipMallocManaged(&(data->zvort_4d_chunk), bufsize*sizeof(float));
    hipDeviceSynchronize();

    return data;

}

/* Deallocate the struct of 4D arrays that store
   fields for integration and calculation. This 
   only ever gets called by Rank 0, so there
   should be no need for a CPU counterpart. */
void deallocate_integration_managed(integration_data *data) {
    hipFree(data->u_4d_chunk);
    hipFree(data->v_4d_chunk);
    hipFree(data->w_4d_chunk);
    hipFree(data->pres_4d_chunk);
    hipFree(data->th_4d_chunk);
    hipFree(data->rho_4d_chunk);
    hipFree(data->khh_4d_chunk);
    hipFree(data->xvort_4d_chunk);
    hipFree(data->yvort_4d_chunk);
    hipFree(data->zvort_4d_chunk);
    hipDeviceSynchronize();
}
#endif
