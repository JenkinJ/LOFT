#include <vector>
#include "macros.cpp"
#include "datastructs.h"
#include <iostream>
#ifndef DATASTRUCTS
#define DATASTRUCTS
using namespace std;

/* Allocate memory on the CPU and GPU for a grid. There are times,
    like for various MPI ranks, that you don't want to do this on both.
    See the similar function for doing this on just the CPU */
datagrid* allocate_grid_managed( int X0, int X1, int Y0, int Y1, int Z0, int Z1 ) {
    datagrid *grid;
    long NX, NY, NZ;

    hipMallocManaged(&grid, sizeof(datagrid));
    grid->X0 = X0; grid->X1 = X1;
    grid->Y0 = Y0; grid->Y1 = Y1;
    grid->Z0 = Z0; grid->Z1 = Z1;

	NX = grid->X1 - grid->X0 + 1;
	NY = grid->Y1 - grid->Y0 + 1;
	NZ = grid->Z1 - grid->Z0 + 1;

    // set the grid attributes
    grid->NX = NX;
    grid->NY = NY;
    grid->NZ = NZ;

    // allocage grid arrays
    hipMallocManaged(&(grid->xf), (NX+1)*sizeof(float));
    hipMallocManaged(&(grid->xh), NX*sizeof(float));

    hipMallocManaged(&(grid->yf), (NY+1)*sizeof(float));
    hipMallocManaged(&(grid->yh), NY*sizeof(float));

    // +2 is +1 for stagger, +1 for potential bottom ghost zone
    hipMallocManaged(&(grid->zf), (NZ+2)*sizeof(float));
    hipMallocManaged(&(grid->zh), (NZ+2)*sizeof(float));

    hipMallocManaged(&(grid->uf), (NX+2)*sizeof(float));
    hipMallocManaged(&(grid->uh), (NX+2)*sizeof(float));

    hipMallocManaged(&(grid->vf), (NY+2)*sizeof(float));
    hipMallocManaged(&(grid->vh), (NY+2)*sizeof(float));

    hipMallocManaged(&(grid->mf), (NZ+2)*sizeof(float));
    hipMallocManaged(&(grid->mh), (NZ+2)*sizeof(float));

    // allocate base state arrays
    hipMallocManaged(&(grid->u0), NZ*sizeof(float));
    hipMallocManaged(&(grid->v0), NZ*sizeof(float));
    hipMallocManaged(&(grid->qv0), NZ*sizeof(float));
    hipMallocManaged(&(grid->th0), NZ*sizeof(float));
    hipMallocManaged(&(grid->rho0), NZ*sizeof(float));
    hipMallocManaged(&(grid->p0), NZ*sizeof(float));
    hipDeviceSynchronize();

    return grid;
}

/* Allocate arrays only on the CPU for the grid. This is important
   for using with MPI, as only 1 rank should be allocating memory
   on the GPU */
datagrid* allocate_grid_cpu( int X0, int X1, int Y0, int Y1, int Z0, int Z1 ) {
    datagrid *grid = new datagrid();
    long NX, NY, NZ;

    grid->X0 = X0; grid->X1 = X1;
    grid->Y0 = Y0; grid->Y1 = Y1;
    grid->Z0 = Z0; grid->Z1 = Z1;

	NX = grid->X1 - grid->X0 + 1;
	NY = grid->Y1 - grid->Y0 + 1;
	NZ = grid->Z1 - grid->Z0 + 1;

    // set the grid attributes
    grid->NX = NX;
    grid->NY = NY;
    grid->NZ = NZ;

    // allocage grid arrays
    grid->xf = new float[NX+1];
    grid->xh = new float[NX];

    grid->yf = new float[NY+1];
    grid->yh = new float[NY];

    grid->zf = new float[NZ+2];
    grid->zh = new float[NZ+2];

    grid->uf = new float[NX+2];
    grid->uh = new float[NX+2];

    grid->vf = new float[NY+2];
    grid->vh = new float[NY+2];

    grid->mf = new float[NZ+2];
    grid->mh = new float[NZ+2];

    // allocate base state arrays
    grid->u0 = new float[NZ];
    grid->v0 = new float[NZ];
    grid->qv0 = new float[NZ];
    grid->th0 = new float[NZ];
    grid->rho0 = new float[NZ];
    grid->p0 = new float[NZ];

    return grid;
}

/* Deallocate all of the arrays in the 
   struct for both the GPU and CPU */
void deallocate_grid_managed(datagrid *grid) {
    hipFree(grid->xf);
    hipFree(grid->xh);
    hipFree(grid->yf);
    hipFree(grid->yh);
    hipFree(grid->zf);
    hipFree(grid->zh);
    hipFree(grid->uf);
    hipFree(grid->uh);
    hipFree(grid->vf);
    hipFree(grid->vh);
    hipFree(grid->mf);
    hipFree(grid->mh);
    hipFree(grid->u0);
    hipFree(grid->v0);
    hipFree(grid->rho0);
    hipFree(grid->th0);
    hipFree(grid->qv0);
    hipFree(grid->p0);
    hipDeviceSynchronize();
}

/* Deallocate all of the arrays in the
   struct only for the CPU */
void deallocate_grid_cpu(datagrid *grid) {
    delete[] grid->xf;
    delete[] grid->xh;
    delete[] grid->yf;
    delete[] grid->yh;
    delete[] grid->zf;
    delete[] grid->zh;
    delete[] grid->uf;
    delete[] grid->uh;
    delete[] grid->vf;
    delete[] grid->vh;
    delete[] grid->mf;
    delete[] grid->mh;
    delete[] grid->u0;
    delete[] grid->v0;
    delete[] grid->rho0;
    delete[] grid->th0;
    delete[] grid->qv0;
    delete[] grid->p0;
}

/* Allocate arrays for parcel info on both the CPU and GPU.
   This function should only be called by MPI Rank 0, so
   be sure to use the CPU function for Rank >= 1. */
parcel_pos* allocate_parcels_managed(int NX, int NY, int NZ, int nTotTimes) {
    int nParcels = NX*NY*NZ;
    parcel_pos *parcels;
    // create the struct on both the GPU and the CPU.
    hipMallocManaged(&parcels, sizeof(parcel_pos));

    // allocate memory for the parcels
    // we are integrating for the entirety 
    // of the simulation.
    hipMallocManaged(&(parcels->xpos), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->ypos), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->zpos), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclu), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclv), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclw), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclkmh), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pcluturb), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclvturb), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclwturb), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pcludiff), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclvdiff), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclwdiff), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclxvort), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclyvort), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclzvort), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclxvorttilt), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclyvorttilt), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclzvorttilt), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclxvortstretch), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclyvortstretch), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclzvortstretch), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclxvortturb), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclyvortturb), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclzvortturb), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclxvortdiff), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclyvortdiff), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclzvortdiff), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclxvortsolenoid), nParcels*nTotTimes*sizeof(float));
    hipMallocManaged(&(parcels->pclyvortsolenoid), nParcels*nTotTimes*sizeof(float));
    hipMallocManaged(&(parcels->pclzvortsolenoid), nParcels*nTotTimes*sizeof(float));

    hipMallocManaged(&(parcels->pclppert), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclqvpert), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclrhopert), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclthetapert), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclthrhopert), nParcels*nTotTimes*sizeof(float)); 

    hipMallocManaged(&(parcels->pclpbar), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclqvbar), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclrhobar), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclthetabar), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclthrhobar), nParcels*nTotTimes*sizeof(float)); 

    /*
    hipMallocManaged(&(parcels->pclqc), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclqi), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclqs), nParcels*nTotTimes*sizeof(float)); 
    hipMallocManaged(&(parcels->pclqg), nParcels*nTotTimes*sizeof(float)); 
    */

    // set the static variables
    parcels->nParcels = nParcels;
    parcels->nTimes = nTotTimes;
    hipDeviceSynchronize();

    return parcels;
}

/* Allocate arrays only on the CPU for the grid. This is important
   for using with MPI, as only 1 rank should be allocating memory
   on the GPU */
parcel_pos* allocate_parcels_cpu(int NX, int NY, int NZ, int nTotTimes) {
    int nParcels = NX*NY*NZ;
    parcel_pos *parcels = new parcel_pos();

    // allocate memory for the parcels
    // we are integrating for the entirety 
    // of the simulation.
    parcels->xpos = new float[nParcels*nTotTimes]; 
    parcels->ypos = new float[nParcels*nTotTimes]; 
    parcels->zpos = new float[nParcels*nTotTimes]; 
    parcels->pclu = new float[nParcels*nTotTimes]; 
    parcels->pclv = new float[nParcels*nTotTimes]; 
    parcels->pclw = new float[nParcels*nTotTimes]; 
    parcels->pclkmh = new float[nParcels*nTotTimes]; 
    parcels->pcluturb = new float[nParcels*nTotTimes]; 
    parcels->pclvturb = new float[nParcels*nTotTimes]; 
    parcels->pclwturb = new float[nParcels*nTotTimes]; 
    parcels->pcludiff = new float[nParcels*nTotTimes]; 
    parcels->pclvdiff = new float[nParcels*nTotTimes]; 
    parcels->pclwdiff = new float[nParcels*nTotTimes]; 
    parcels->pclxvort = new float[nParcels*nTotTimes]; 
    parcels->pclyvort = new float[nParcels*nTotTimes]; 
    parcels->pclzvort = new float[nParcels*nTotTimes]; 
    parcels->pclxvorttilt = new float[nParcels*nTotTimes]; 
    parcels->pclyvorttilt = new float[nParcels*nTotTimes]; 
    parcels->pclzvorttilt = new float[nParcels*nTotTimes]; 
    parcels->pclxvortstretch = new float[nParcels*nTotTimes]; 
    parcels->pclyvortstretch = new float[nParcels*nTotTimes]; 
    parcels->pclzvortstretch = new float[nParcels*nTotTimes]; 
    parcels->pclxvortturb = new float[nParcels*nTotTimes]; 
    parcels->pclyvortturb = new float[nParcels*nTotTimes]; 
    parcels->pclzvortturb = new float[nParcels*nTotTimes]; 
    parcels->pclxvortdiff = new float[nParcels*nTotTimes]; 
    parcels->pclyvortdiff = new float[nParcels*nTotTimes]; 
    parcels->pclzvortdiff = new float[nParcels*nTotTimes]; 
    parcels->pclxvortsolenoid = new float[nParcels*nTotTimes];
    parcels->pclyvortsolenoid = new float[nParcels*nTotTimes];
    parcels->pclzvortsolenoid = new float[nParcels*nTotTimes];

    parcels->pclppert = new float[nParcels*nTotTimes];
    parcels->pclqvpert = new float[nParcels*nTotTimes];
    parcels->pclrhopert = new float[nParcels*nTotTimes];
    parcels->pclthetapert = new float[nParcels*nTotTimes];
    parcels->pclthrhopert = new float[nParcels*nTotTimes];

    parcels->pclpbar = new float[nParcels*nTotTimes];
    parcels->pclqvbar = new float[nParcels*nTotTimes];
    parcels->pclrhobar = new float[nParcels*nTotTimes];
    parcels->pclthetabar = new float[nParcels*nTotTimes];
    parcels->pclthrhobar = new float[nParcels*nTotTimes];

    /*
    parcels->pclqc = new float[nParcels*nTotTimes];
    parcels->pclqi = new float[nParcels*nTotTimes];
    parcels->pclqs = new float[nParcels*nTotTimes];
    parcels->pclqg = new float[nParcels*nTotTimes];
    */
    // set the static variables
    parcels->nParcels = nParcels;
    parcels->nTimes = nTotTimes;

    return parcels;
}

/* Deallocate parcel arrays on both the CPU and the
   GPU */
void deallocate_parcels_managed(parcel_pos *parcels) {
    hipFree(parcels->xpos);
    hipFree(parcels->ypos);
    hipFree(parcels->zpos);
    hipFree(parcels->pclu);
    hipFree(parcels->pclv);
    hipFree(parcels->pclw);
    hipFree(parcels->pclkmh);
    hipFree(parcels->pcluturb);
    hipFree(parcels->pclvturb);
    hipFree(parcels->pclwturb);
    hipFree(parcels->pcludiff);
    hipFree(parcels->pclvdiff);
    hipFree(parcels->pclwdiff);
    hipFree(parcels->pclxvort);
    hipFree(parcels->pclyvort);
    hipFree(parcels->pclzvort);
    hipFree(parcels->pclxvorttilt);
    hipFree(parcels->pclyvorttilt);
    hipFree(parcels->pclzvorttilt);
    hipFree(parcels->pclxvortstretch);
    hipFree(parcels->pclyvortstretch);
    hipFree(parcels->pclzvortstretch);
    hipFree(parcels->pclxvortturb);
    hipFree(parcels->pclyvortturb);
    hipFree(parcels->pclzvortturb);
    hipFree(parcels->pclxvortdiff);
    hipFree(parcels->pclyvortdiff);
    hipFree(parcels->pclzvortdiff);
    hipFree(parcels->pclxvortsolenoid);
    hipFree(parcels->pclyvortsolenoid);
    hipFree(parcels->pclzvortsolenoid);


    hipFree(parcels->pclppert);
    hipFree(parcels->pclqvpert);
    hipFree(parcels->pclrhopert);
    hipFree(parcels->pclthetapert);
    hipFree(parcels->pclthrhopert);

    hipFree(parcels->pclpbar);
    hipFree(parcels->pclqvbar);
    hipFree(parcels->pclrhobar);
    hipFree(parcels->pclthetabar);
    hipFree(parcels->pclthrhobar);

    /*
    hipFree(parcels->pclqc);
    hipFree(parcels->pclqi);
    hipFree(parcels->pclqs);
    hipFree(parcels->pclqg);
    */
    hipFree(parcels);
    hipDeviceSynchronize();
}

/* Deallocate parcel arrays only on the CPU */
void deallocate_parcels_cpu(parcel_pos *parcels) {
    delete[] parcels->xpos;
    delete[] parcels->ypos;
    delete[] parcels->zpos;
    delete[] parcels->pclu;
    delete[] parcels->pclv;
    delete[] parcels->pclw;
    delete[] parcels->pclkmh;
    delete[] parcels->pcluturb;
    delete[] parcels->pclvturb;
    delete[] parcels->pclwturb;
    delete[] parcels->pcludiff;
    delete[] parcels->pclvdiff;
    delete[] parcels->pclwdiff;
    delete[] parcels->pclxvort;
    delete[] parcels->pclyvort;
    delete[] parcels->pclzvort;
    delete[] parcels->pclxvorttilt;
    delete[] parcels->pclyvorttilt;
    delete[] parcels->pclzvorttilt;
    delete[] parcels->pclxvortstretch;
    delete[] parcels->pclyvortstretch;
    delete[] parcels->pclzvortstretch;
    delete[] parcels->pclxvortturb;
    delete[] parcels->pclyvortturb;
    delete[] parcels->pclzvortturb;
    delete[] parcels->pclxvortdiff;
    delete[] parcels->pclyvortdiff;
    delete[] parcels->pclzvortdiff;
    delete[] parcels->pclxvortsolenoid;
    delete[] parcels->pclyvortsolenoid;
    delete[] parcels->pclzvortsolenoid;

    delete[] parcels->pclppert;
    delete[] parcels->pclqvpert;
    delete[] parcels->pclrhopert;
    delete[] parcels->pclthetapert;
    delete[] parcels->pclthrhopert;

    delete[] parcels->pclpbar;
    delete[] parcels->pclqvbar;
    delete[] parcels->pclrhobar;
    delete[] parcels->pclthetabar;
    delete[] parcels->pclthrhobar;

    /*
    delete[] parcels->pclqc;
    delete[] parcels->pclqi;
    delete[] parcels->pclqs;
    delete[] parcels->pclqg;
    */
    delete[] parcels;
}

/* Allocate the struct of 4D arrays that store
   fields for integration and calculation. This
   only ever gets called by Rank 0, so there 
   should be no need for a CPU counterpart. */
model_data* allocate_model_managed(long bufsize) {
    model_data *data;
    // create the struct on both the GPU and the CPU.
    hipMallocManaged(&data, sizeof(model_data));

    // allocate the arrays in the struct
    hipMallocManaged(&(data->ustag), bufsize*sizeof(float));
    hipMallocManaged(&(data->vstag), bufsize*sizeof(float));
    hipMallocManaged(&(data->wstag), bufsize*sizeof(float));
    hipMallocManaged(&(data->pi), bufsize*sizeof(float));
    hipMallocManaged(&(data->prespert), bufsize*sizeof(float));
    hipMallocManaged(&(data->thrhopert),  bufsize*sizeof(float));
    hipMallocManaged(&(data->thetapert),  bufsize*sizeof(float));
    hipMallocManaged(&(data->rhopert), bufsize*sizeof(float));
    hipMallocManaged(&(data->rhof), bufsize*sizeof(float));
    hipMallocManaged(&(data->kmh), bufsize*sizeof(float));
    hipMallocManaged(&(data->qc), bufsize*sizeof(float));
    hipMallocManaged(&(data->qi), bufsize*sizeof(float));
    hipMallocManaged(&(data->qs), bufsize*sizeof(float));
    hipMallocManaged(&(data->qg), bufsize*sizeof(float));
    hipMallocManaged(&(data->qvpert), bufsize*sizeof(float));
    hipMallocManaged(&(data->turbu), bufsize*sizeof(float));
    hipMallocManaged(&(data->turbv), bufsize*sizeof(float));
    hipMallocManaged(&(data->turbw), bufsize*sizeof(float));
    hipMallocManaged(&(data->diffu), bufsize*sizeof(float));
    hipMallocManaged(&(data->diffv), bufsize*sizeof(float));
    hipMallocManaged(&(data->diffw), bufsize*sizeof(float));
    hipMallocManaged(&(data->tem1), bufsize*sizeof(float));
    hipMallocManaged(&(data->tem2), bufsize*sizeof(float));
    hipMallocManaged(&(data->tem3), bufsize*sizeof(float));
    hipMallocManaged(&(data->tem4), bufsize*sizeof(float));
    hipMallocManaged(&(data->tem5), bufsize*sizeof(float));
    hipMallocManaged(&(data->tem6), bufsize*sizeof(float));
    hipMallocManaged(&(data->xvort), bufsize*sizeof(float));
    hipMallocManaged(&(data->yvort), bufsize*sizeof(float));
    hipMallocManaged(&(data->zvort), bufsize*sizeof(float));
    hipMallocManaged(&(data->xvtilt), bufsize*sizeof(float));
    hipMallocManaged(&(data->yvtilt), bufsize*sizeof(float));
    hipMallocManaged(&(data->zvtilt), bufsize*sizeof(float));
    hipMallocManaged(&(data->xvstretch), bufsize*sizeof(float));
    hipMallocManaged(&(data->yvstretch), bufsize*sizeof(float));
    hipMallocManaged(&(data->zvstretch), bufsize*sizeof(float));
    hipMallocManaged(&(data->turbxvort), bufsize*sizeof(float));
    hipMallocManaged(&(data->turbyvort), bufsize*sizeof(float));
    hipMallocManaged(&(data->turbzvort), bufsize*sizeof(float));
    hipMallocManaged(&(data->diffxvort), bufsize*sizeof(float));
    hipMallocManaged(&(data->diffyvort), bufsize*sizeof(float));
    hipMallocManaged(&(data->diffzvort), bufsize*sizeof(float));
    hipMallocManaged(&(data->xvort_solenoid), bufsize*sizeof(float)); 
    hipMallocManaged(&(data->yvort_solenoid), bufsize*sizeof(float)); 
    hipMallocManaged(&(data->zvort_solenoid), bufsize*sizeof(float)); 

    return data;

}

/* Deallocate the struct of 4D arrays that store
   fields for integration and calculation. This 
   only ever gets called by Rank 0, so there
   should be no need for a CPU counterpart. */
void deallocate_model_managed(model_data *data) {
    hipFree(data->ustag);
    hipFree(data->vstag);
    hipFree(data->wstag);
    hipFree(data->pi);
    hipFree(data->prespert);
    hipFree(data->thetapert);
    hipFree(data->thrhopert);
    hipFree(data->rhopert);
    hipFree(data->rhof);
    hipFree(data->kmh);
    
    hipFree(data->qc);
    hipFree(data->qi);
    hipFree(data->qs);
    hipFree(data->qg);
    hipFree(data->qvpert);
    hipFree(data->turbu);
    hipFree(data->turbv);
    hipFree(data->turbw);
    hipFree(data->diffu);
    hipFree(data->diffv);
    hipFree(data->diffw);
    hipFree(data->tem1);
    hipFree(data->tem2);
    hipFree(data->tem3);
    hipFree(data->tem4);
    hipFree(data->tem5);
    hipFree(data->tem6);
    hipFree(data->xvort);
    hipFree(data->yvort);
    hipFree(data->zvort);
    hipFree(data->xvtilt);
    hipFree(data->yvtilt);
    hipFree(data->zvtilt);
    hipFree(data->xvstretch);
    hipFree(data->yvstretch);
    hipFree(data->zvstretch);
    hipFree(data->turbxvort);
    hipFree(data->turbyvort);
    hipFree(data->turbzvort);
    hipFree(data->diffxvort);
    hipFree(data->diffyvort);
    hipFree(data->diffzvort);
    hipFree(data->xvort_solenoid); 
    hipFree(data->yvort_solenoid); 
    hipFree(data->zvort_solenoid); 
}
#endif
