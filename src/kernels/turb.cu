#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "../include/datastructs.h"
#include "../include/macros.h"
#include "../calc/calcturb.cu"
#ifndef TURB_CU
#define TURB_CU

__global__ void doCalcRf(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY) && (k < NZ+1)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calcrf(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}
__global__ void doCalcDef(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY) && (k < NZ) && (i > 0) && (j > 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calcdef(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

__global__ void doGetTau(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY) && (k < NZ) && (i > 0) && (j > 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            gettau(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}
__global__ void doCalcTurb(datagrid *grid, model_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY) && (k < NZ) && (i > 0) && (j > 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_turbu(grid, data, idx_4D, NX, NY, NZ);
        }
    }
    if ((i < NX) && (j < NY) && (k < NZ) && (j > 0) && (i > 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_turbv(grid, data, idx_4D, NX, NY, NZ);
        }
    }
    if ((i < NX) && (j < NY) && (k < NZ+1) && (k > 0) && (i > 0) && (j > 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_turbw(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

#endif
