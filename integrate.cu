#include "hip/hip_runtime.h"
#include <iostream>
#include "datastructs.cpp"
#include "interp.cu"
#ifndef INTEGRATE_CU
#define INTEGRATE_CU

using namespace std;
// this is an error checking helper function for processes
// that run on the GPU. Without calling this, the GPU can
// fail to execute but the program won't crash or report it.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      cout << hipGetErrorString(code) << endl;
      if (abort) exit(code);
   }
}


__global__ void test(datagrid grid, parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, int MX, int MY, int MZ, int nT, int tChunk, int totTime) {
	int parcel_id = blockIdx.x;
    // safety check to make sure our thread index doesn't
    // go out of our array bounds
    if (parcel_id < parcels.nParcels) {
        bool is_ugrd = false;
        bool is_vgrd = false;
        bool is_wgrd = false;

        float pcl_u, pcl_v, pcl_w;
        float point[3];

        // loop over the number of time steps we are
        // integrating over
        for (int tidx = 0; tidx < nT; ++tidx) {
            point[0] = parcels.xpos[tidx + (totTime * parcel_id)];
            point[1] = parcels.ypos[tidx + (totTime * parcel_id)];
            point[2] = parcels.zpos[tidx + (totTime * parcel_id)];
            //printf("My Point to Integrate: x = %f\t y = %f\t z = %f\t parcel_id = %d\t time = %d\n", point[0], point[1], point[2], parcel_id, tidx);


            is_ugrd = true;
            is_vgrd = false;
            is_wgrd = false;
            pcl_u = interp3D(grid.xh, grid.yh, grid.zh, u_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);

            is_ugrd = false;
            is_vgrd = true;
            is_wgrd = false;
            pcl_v = interp3D(grid.xh, grid.yh, grid.zh, v_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);

            is_ugrd = false;
            is_vgrd = false;
            is_wgrd = true;
            pcl_w = interp3D(grid.xh, grid.yh, grid.zh, w_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            //printf("My Parcel Vector Field: u = %f\t v = %f\t w = %f\n", pcl_u, pcl_v, pcl_w);

            // if the parcel has left the domain, exit
            if ((pcl_u == -999.0) || (pcl_v == -999.0) || (pcl_w == -999.0)) {
                return;
            }
            else {

                // integrate X position forward by the U wind
                point[0] += pcl_u * (1.0f/6.0f);
                // integrate Y position forward by the V wind
                point[1] += pcl_v * (1.0f/6.0f);
                // integrate Z position forward by the W wind
                point[2] += pcl_w * (1.0f/6.0f);


                parcels.xpos[(tidx + 1) + (totTime * parcel_id)] = point[0]; 
                parcels.ypos[(tidx + 1) + (totTime * parcel_id)] = point[1];
                parcels.zpos[(tidx + 1) + (totTime * parcel_id)] = point[2];
            }
        }
    }
}

/*This function handles allocating memory on the GPU, transferring the CPU
arrays to GPU global memory, calling the integrate GPU kernel, and then
updating the position vectors with the new stuff*/
void cudaIntegrateParcels(datagrid grid, parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, \
                            int MX, int MY, int MZ, int nT, int tChunk, int totTime) {
    // pointers to device memory
    float *device_u_time_chunk, *device_v_time_chunk, *device_w_time_chunk;

    parcel_pos device_parcels;
    datagrid device_grid;

    // copy over our integer and long
    // constants to our device struct
    device_grid.X0 = grid.X0; device_grid.X1 = grid.X1;
    device_grid.Y0 = grid.Y0; device_grid.Y1 = grid.Y1;
    device_grid.Z0 = grid.Z0; device_grid.Z1 = grid.Z1;
    device_grid.NX = grid.NX; device_grid.NY = grid.NY;
    device_grid.NZ = grid.NZ; 
    device_parcels.nParcels = parcels.nParcels;

    // allocate device memory for our grid arrays
    gpuErrchk( hipMalloc(&(device_grid.xh), device_grid.NX*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.yh), device_grid.NY*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.zh), device_grid.NZ*sizeof(float)) );
    // allocate the device memory for U/V/W
    gpuErrchk( hipMalloc(&device_u_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_v_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_w_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    // allocate device memory for our parcel positions
    gpuErrchk( hipMalloc(&(device_parcels.xpos), parcels.nParcels * (nT+1) * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.ypos), parcels.nParcels * (nT+1) * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.zpos), parcels.nParcels * (nT+1) * sizeof(float)) );

    // copy the arrays to device memory
    gpuErrchk( hipMemcpy(device_u_time_chunk, u_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_v_time_chunk, v_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_w_time_chunk, w_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(device_parcels.xpos, parcels.xpos, parcels.nParcels * totTime * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_parcels.ypos, parcels.ypos, parcels.nParcels * totTime * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_parcels.zpos, parcels.zpos, parcels.nParcels * totTime * sizeof(float), hipMemcpyHostToDevice) );

    // don't need the staggered mesh sent
    gpuErrchk( hipMemcpy(device_grid.xh, grid.xh, device_grid.NX*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.yh, grid.yh, device_grid.NY*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.zh, grid.zh, device_grid.NZ*sizeof(float), hipMemcpyHostToDevice) );

    test<<<parcels.nParcels,1>>>(device_grid, device_parcels, device_u_time_chunk, device_v_time_chunk, device_w_time_chunk, MX, MY, MZ, nT, tChunk, totTime);
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipMemcpy(parcels.xpos, device_parcels.xpos, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.ypos, device_parcels.ypos, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.zpos, device_parcels.zpos, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipDeviceSynchronize() );

    hipFree(device_grid.xh);
    hipFree(device_grid.yh);
    hipFree(device_grid.zh);
    hipFree(device_parcels.xpos);
    hipFree(device_parcels.ypos);
    hipFree(device_parcels.zpos);


    cout << "FINISHED CUDA" << endl;
}

#endif
