#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "datastructs.cpp"
#include "macros.cpp"
#include "interp.cu"
#ifndef INTEGRATE_CU
#define INTEGRATE_CU

using namespace std;
// this is an error checking helper function for processes
// that run on the GPU. Without calling this, the GPU can
// fail to execute but the program won't crash or report it.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      cout << hipGetErrorString(code) << endl;
      if (abort) exit(code);
   }
}



__device__ void calc_xvort(datagrid grid, float *wstag, float *vstag, float *xvort, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];
}

__device__ void calc_yvort(datagrid grid, float *ustag, float *wstag, float *yvort, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

}

__device__ void calc_zvort(datagrid grid, float *ustag, float *vstag, float *zvort, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

}

/* When doing the parcel trajectory integration, George Bryan does
   some fun stuff with the lower boundaries of the arrays, presumably
   to prevent the parcels from exiting out the bottom of the domain
   or experience artificial values */
__global__ void applyMomentumBC(float *ustag, float *vstag, float *wstag, int NX, int NY, int NZ, int tStart, int tEnd) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    // this is done for easy comparison to CM1 code
    int ni = NX; int nj = NY; int nk = NZ;

    // this is a lower boundary condition, so only when k is 0
    // also this is on the u staggered mesh
    if (( j >= 0 ) && ( i >= 0) && ( j < nj+1) && ( i < ni+2) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the u stagger macro to handle the
            // proper indexing
            UA(i, j, 0, tidx) = UA(i, j, 1, tidx);
        }
    }
    
    // do the same but now on the v staggered grid
    if (( j >= 0 ) && ( i >= 0) && ( j < nj+2) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the v stagger macro to handle the
            // proper indexing
            VA(i, j, 0, tidx) = VA(i, j, 1, tidx);
        }
    }

    // do the same but now on the w staggered grid
    if (( j >= 0 ) && ( i >= 0) && ( j < nj+1) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the w stagger macro to handle the
            // proper indexing
            WA(i, j, 0, tidx) = -1*WA(i, j, 2, tidx);
        }
    }
}

/* Kernel for computing the components of vorticity
    and vorticity forcing terms. We do this using our domain subset containing the parcels
    instead of doing it locally for each parcel, as it would scale poorly for large 
    numbers of parcels. */
__global__ void calcvort(datagrid grid, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, \
                        float *xvort, float *yvort, float *zvort, \
                        int MX, int MY, int MZ, int tStart, int tEnd, int totTime) {

    // get our 3D index based on our blocks/threads
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int idx_4D[4];
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < MX-1) && (j < MY-1) && (k < MZ-1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort(grid, w_time_chunk, v_time_chunk, xvort, idx_4D, MX, MY, MZ);
            calc_yvort(grid, u_time_chunk, w_time_chunk, yvort, idx_4D, MX, MY, MZ);
            calc_zvort(grid, u_time_chunk, v_time_chunk, zvort, idx_4D, MX, MY, MZ);
        }
    }
}

__global__ void integrate(datagrid grid, parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, \
                    int MX, int MY, int MZ, int tStart, int tEnd, int totTime, int direct) {

	int parcel_id = blockIdx.x;
    // safety check to make sure our thread index doesn't
    // go out of our array bounds
    if (parcel_id < parcels.nParcels) {
        bool is_ugrd = false;
        bool is_vgrd = false;
        bool is_wgrd = false;

        float pcl_u, pcl_v, pcl_w;
        float point[3];

        // loop over the number of time steps we are
        // integrating over
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // GPU sanity test of data integrity
            point[0] = parcels.xpos[P2(tidx, parcel_id, totTime)];
            point[1] = parcels.ypos[P2(tidx, parcel_id, totTime)];
            point[2] = parcels.zpos[P2(tidx, parcel_id, totTime)];

            is_ugrd = true;
            is_vgrd = false;
            is_wgrd = false;
            pcl_u = interp3D(grid, u_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);

            is_ugrd = false;
            is_vgrd = true;
            is_wgrd = false;
            pcl_v = interp3D(grid, v_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);

            is_ugrd = false;
            is_vgrd = false;
            is_wgrd = true;
            pcl_w = interp3D(grid, w_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            
            // integrate X position forward by the U wind
            point[0] += pcl_u * (1.0f/6.0f) * direct;
            // integrate Y position forward by the V wind
            point[1] += pcl_v * (1.0f/6.0f) * direct;
            // integrate Z position forward by the W wind
            point[2] += pcl_w * (1.0f/6.0f) * direct;
            if ((pcl_u == -999.0) || (pcl_v == -999.0) || (pcl_w == -999.0)) {
                printf("Warning: missing values detected at x: %f y:%f z:%f with ground bounds X0: %f Y0: %f Z0: %f X1: %f Y1: %f Z1: %f\n", \
                    point[0], point[1], point[2], grid.xh[0], grid.yh[0], grid.zh[0], grid.xh[grid.NX-1], grid.yh[grid.NY-1], grid.zh[grid.NZ-1]);
                return;
            }


            parcels.xpos[P2(tidx+1, parcel_id, totTime)] = point[0]; 
            parcels.ypos[P2(tidx+1, parcel_id, totTime)] = point[1];
            parcels.zpos[P2(tidx+1, parcel_id, totTime)] = point[2];
        }
    }
}

/*This function handles allocating memory on the GPU, transferring the CPU
arrays to GPU global memory, calling the integrate GPU kernel, and then
updating the position vectors with the new stuff*/
void cudaIntegrateParcels(datagrid grid, parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, \
                         float *p_time_chunk, float *th_time_chunk, float *rho_time_chunk, float *khh_time_chunk, \
                         int MX, int MY, int MZ, int nT, int totTime, int direct) {
    // pointers to device memory
    float *device_u_time_chunk, *device_v_time_chunk, *device_w_time_chunk;
    float *device_xvort_time_chunk, *device_yvort_time_chunk, *device_zvort_time_chunk;
    parcel_pos device_parcels;
    datagrid device_grid;

    int tStart, tEnd;
    tStart = 0;
    tEnd = nT;
}

#endif
