#include "hip/hip_runtime.h"
#include <iostream>
#include "datastructs.cpp"
#ifndef INTEGRATE_CU
#define INTEGRATE_CU

using namespace std;
// this is an error checking helper function for processes
// that run on the GPU. Without calling this, the GPU can
// fail to execute but the program won't crash or report it.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      cout << hipGetErrorString(code) << endl;
      if (abort) exit(code);
   }
}

// This is a GPU kernel for integrating a fluid parcel forward in time by increment dt.
// It expects the X, Y, and Z array pointers as well as the U, V, and W wind component pointers,
// which are all of length nParcels
__global__ void integrate(float *x_arr, float *y_arr, float *z_arr, float *u_arr, float *v_arr, float *w_arr, int nParcels, float dt) {
	// use the thread index to index the array
	int pidx = blockIdx.x*blockDim.x + threadIdx.x;

	// safety check to not access array memory out of bounds
	if (pidx < nParcels) {
		// integrate X position forward by the U wind
		x_arr[pidx] = x_arr[pidx] + u_arr[pidx] * dt;
		// integrate Y position forward by the V wind
		y_arr[pidx] = y_arr[pidx] + v_arr[pidx] * dt;
		// integrate Z position forward by the W wind
		z_arr[pidx] = z_arr[pidx] + w_arr[pidx] * dt;
	}
}

__global__ void test(datagrid grid, parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, int MX, int MY, int MZ, int nT) {
	int parcel_id = blockIdx.x*blockDim.x + threadIdx.x;
    // safety check to make sure our thread index doesn't
    // go out of our array bounds
    if (parcel_id < parcels.nParcels) {
        // loop over the number of time steps we are
        // integrating over
        for (int tidx = 0; tidx < nT; ++tidx) {
            printf("X0: %f\t X1: %f\tParcel Number: %d/%d\tParcel X: %f\tParcel Y: %f\t Parcel Z: %f\n", grid.xf[0], grid.xf[grid.NX-1], parcel_id, \
                    parcels.nParcels, parcels.xpos[parcel_id], parcels.ypos[parcel_id], parcels.zpos[parcel_id]);
        // interpolate U/V/W values at parcel location
        }
    }
}

/*This function handles allocating memory on the GPU, transferring the CPU
arrays to GPU global memory, calling the integrate GPU kernel, and then
updating the position vectors with the new stuff*/
void cudaIntegrateParcels(datagrid grid, parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, int MX, int MY, int MZ, int nT) {
    // pointers to device memory
    float *device_u_time_chunk, *device_v_time_chunk, *device_w_time_chunk;
    parcel_pos device_parcels;
    datagrid device_grid;

    // copy over our integer and long
    // constants to our device struct
    device_grid.X0 = grid.X0; device_grid.X1 = grid.X1;
    device_grid.Y0 = grid.Y0; device_grid.Y1 = grid.Y1;
    device_grid.Z0 = grid.Z0; device_grid.Z1 = grid.Z1;
    device_grid.NX = grid.NX; device_grid.NY = grid.NY;
    device_grid.NZ = grid.NZ; 
    device_parcels.nParcels = parcels.nParcels;

    // allocate device memory for our grid arrays
    gpuErrchk( hipMalloc(&(device_grid.xf), device_grid.NX*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.yf), device_grid.NY*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.zf), device_grid.NZ*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.xh), device_grid.NX*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.yh), device_grid.NY*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.zh), device_grid.NZ*sizeof(float)) );
    // allocate the device memory for U/V/W
    gpuErrchk( hipMalloc(&device_u_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_v_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_w_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    // allocate device memory for our parcel positions
    gpuErrchk( hipMalloc(&(device_parcels.xpos), parcels.nParcels * nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.ypos), parcels.nParcels * nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.zpos), parcels.nParcels * nT*sizeof(float)) );

    // copy the arrays to device memory
    gpuErrchk( hipMemcpy(device_u_time_chunk, u_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_v_time_chunk, v_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_w_time_chunk, w_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(device_parcels.xpos, parcels.xpos, parcels.nParcels * nT * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_parcels.ypos, parcels.ypos, parcels.nParcels * nT * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_parcels.zpos, parcels.zpos, parcels.nParcels * nT * sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(device_grid.xf, grid.xf, device_grid.NX*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.yf, grid.yf, device_grid.NY*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.zf, grid.zf, device_grid.NZ*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.xh, grid.xh, device_grid.NX*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.yh, grid.yh, device_grid.NY*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.zh, grid.zh, device_grid.NZ*sizeof(float), hipMemcpyHostToDevice) );

    test<<<parcels.nParcels,1>>>(device_grid, device_parcels, device_u_time_chunk, device_v_time_chunk, device_w_time_chunk, MX, MY, MZ, nT);
    gpuErrchk( hipDeviceSynchronize() );



}

#endif
