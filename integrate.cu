#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "datastructs.cpp"
#include "interp.cu"
#ifndef INTEGRATE_CU
#define INTEGRATE_CU

using namespace std;
#define P2(t,p,mt) (((p)*(mt))+(t))
// this is an error checking helper function for processes
// that run on the GPU. Without calling this, the GPU can
// fail to execute but the program won't crash or report it.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      cout << hipGetErrorString(code) << endl;
      if (abort) exit(code);
   }
}

__device__ void calc_xvort(datagrid grid, float *warr, float *varr, float *xvort, int *idx_4D, int MX, int MY, int MZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dz = grid.zh[k+1] - grid.zh[k];
    float dy = grid.yh[j+1] - grid.yh[j];
    float dw = warr[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - warr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float dv = varr[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - varr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    xvort[arrayIndex(i, j, k, t, MX, MY, MZ)] = ( dw / dy ) - ( dv / dz);

}

__device__ void calc_yvort(datagrid grid, float *uarr, float *warr, float *yvort, int *idx_4D, int MX, int MY, int MZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dz = grid.zh[k+1] - grid.zh[k];
    float dx = grid.xh[j+1] - grid.xh[j];
    float dw = warr[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - warr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float du = uarr[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - uarr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    yvort[arrayIndex(i, j, k, t, MX, MY, MZ)] = ( du / dz ) - ( dw / dx);

}

__device__ void calc_zvort(datagrid grid, float *uarr, float *varr, float *zvort, int *idx_4D, int MX, int MY, int MZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dx = grid.xh[i+1] - grid.xh[i];
    float dy = grid.yh[j+1] - grid.yh[j];
    float dv = varr[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - varr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float du = uarr[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - uarr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    zvort[arrayIndex(i, j, k, t, MX, MY, MZ)] = ( dv / dx ) - ( du / dy);
    //printf("%f, %i, %i, %i, %i\n", zvort[arrayIndex(i, j, k, t, MX, MY, MZ)], i, j, k, t);
}

/* For the vorticity calculations, you have to do a 4 point average
of the neighbors to get the point onto the scalar grid. Calcvort does
the initial pass, and then this gets called at the end to make sure that
the averaging happens */
__global__ void doAvg(float *xvort, float *yvort, float *zvort, int MX, int MY, int MZ, int tStart, int tEnd, int totTime) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    if ((i < MX) && (j < MY) && (k < MZ)) { 
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            xvort[arrayIndex(i, j, k, tidx, MX, MY, MZ)] = 0.25*(xvort[arrayIndex(i, j, k, tidx, MX, MY, MZ)] + \
                xvort[arrayIndex(i, j+1, k, tidx, MX, MY, MZ)] + xvort[arrayIndex(i, j, k+1, tidx, MX, MY, MZ)] + \
                xvort[arrayIndex(i, j+1, k+1, tidx, MX, MY, MZ)]);

            yvort[arrayIndex(i, j, k, tidx, MX, MY, MZ)] = 0.25*(yvort[arrayIndex(i, j, k, tidx, MX, MY, MZ)] + \
                yvort[arrayIndex(i+1, j, k, tidx, MX, MY, MZ)] + yvort[arrayIndex(i, j, k+1, tidx, MX, MY, MZ)] + \
                yvort[arrayIndex(i+1, j, k+1, tidx, MX, MY, MZ)]);

            zvort[arrayIndex(i, j, k, tidx, MX, MY, MZ)] = 0.25*(zvort[arrayIndex(i, j, k, tidx, MX, MY, MZ)] + \
                zvort[arrayIndex(i+1, j, k, tidx, MX, MY, MZ)] + zvort[arrayIndex(i, j+1, k, tidx, MX, MY, MZ)] + \
                zvort[arrayIndex(i+1, j+1, k, tidx, MX, MY, MZ)]);
        }
    }
}

/* Kernel for computing the components of vorticity
    and vorticity forcing terms. We do this using our domain subset containing the parcels
    instead of doing it locally for each parcel, as it would scale poorly for large 
    numbers of parcels. */
__global__ void calcvort(datagrid grid, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, \
                        float *xvort, float *yvort, float *zvort, \
                        int MX, int MY, int MZ, int tStart, int tEnd, int totTime) {

    // get our 3D index based on our blocks/threads
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int idx_4D[4];
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < MX) && (j < MY) && (k < MZ)) { 
        if ((i+1 > MX) || (j+1) > MY) printf("i+1 or j+1 out of bounds\n");
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort(grid, w_time_chunk, v_time_chunk, xvort, idx_4D, MX, MY, MZ);
            calc_yvort(grid, u_time_chunk, w_time_chunk, yvort, idx_4D, MX, MY, MZ);
            // calculate the Z component of vorticity
            //printf("%i, %i, %i, %i, %i, %i, %i\n", i, j, k, tidx, MX, MY, MZ);
            calc_zvort(grid, u_time_chunk, v_time_chunk, zvort, idx_4D, MX, MY, MZ);
        }
    }


}

__global__ void test(datagrid grid, parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, \
                    int MX, int MY, int MZ, int tStart, int tEnd, int totTime) {
	int parcel_id = blockIdx.x;
    // safety check to make sure our thread index doesn't
    // go out of our array bounds
    if (parcel_id < parcels.nParcels) {
        bool is_ugrd = false;
        bool is_vgrd = false;
        bool is_wgrd = false;

        float pcl_u, pcl_v, pcl_w;
        float point[3];

        // loop over the number of time steps we are
        // integrating over
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // GPU sanity test of data integrity
            point[0] = parcels.xpos[P2(tidx, parcel_id, totTime)];
            point[1] = parcels.ypos[P2(tidx, parcel_id, totTime)];
            point[2] = parcels.zpos[P2(tidx, parcel_id, totTime)];

            is_ugrd = true;
            is_vgrd = false;
            is_wgrd = false;
            pcl_u = interp3D(grid, u_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);

            is_ugrd = false;
            is_vgrd = true;
            is_wgrd = false;
            pcl_v = interp3D(grid, v_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);

            is_ugrd = false;
            is_vgrd = false;
            is_wgrd = true;
            pcl_w = interp3D(grid, w_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);


            // integrate X position forward by the U wind
            point[0] += pcl_u * (1.0f/6.0f);
            // integrate Y position forward by the V wind
            point[1] += pcl_v * (1.0f/6.0f);
            // integrate Z position forward by the W wind
            point[2] += pcl_w * (1.0f/6.0f);
            if ((pcl_u == -999.0) || (pcl_v == -999.0) || (pcl_w == -999.0)) {
                printf("Warning: missing values detected at x: %f y:%f z:%f with ground bounds X0: %f Y0: %f Z0: %f X1: %f Y1: %f Z1: %f\n", \
                    point[0], point[1], point[2], grid.xh[0], grid.yh[0], grid.zh[0], grid.xh[grid.NX-1], grid.yh[grid.NY-1], grid.zh[grid.NZ-1]);
                return;
            }

            if (point[2] < grid.zf[0]) point[2] = grid.zf[0];


            parcels.xpos[P2(tidx+1, parcel_id, totTime)] = point[0]; 
            parcels.ypos[P2(tidx+1, parcel_id, totTime)] = point[1];
            parcels.zpos[P2(tidx+1, parcel_id, totTime)] = point[2];
            parcels.pclu[P2(tidx, parcel_id, totTime)] = pcl_u;
            parcels.pclv[P2(tidx, parcel_id, totTime)] = pcl_v;
            parcels.pclw[P2(tidx, parcel_id, totTime)] = pcl_w;
        }
    }
}

/*This function handles allocating memory on the GPU, transferring the CPU
arrays to GPU global memory, calling the integrate GPU kernel, and then
updating the position vectors with the new stuff*/
void cudaIntegrateParcels(datagrid grid, parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, int MX, int MY, int MZ, int nT, int totTime) {
    // pointers to device memory
    float *device_u_time_chunk, *device_v_time_chunk, *device_w_time_chunk;
    float *device_xvort_time_chunk, *device_yvort_time_chunk, *device_zvort_time_chunk;

    parcel_pos device_parcels;
    datagrid device_grid;

    int tStart, tEnd;
    tStart = 0;
    tEnd = nT;

    // copy over our integer and long
    // constants to our device struct
    device_grid.X0 = grid.X0; device_grid.X1 = grid.X1;
    device_grid.Y0 = grid.Y0; device_grid.Y1 = grid.Y1;
    device_grid.Z0 = grid.Z0; device_grid.Z1 = grid.Z1;
    device_grid.NX = grid.NX; device_grid.NY = grid.NY;
    device_grid.NZ = grid.NZ; 
    device_parcels.nParcels = parcels.nParcels;

    // allocate device memory for our grid arrays
    gpuErrchk( hipMalloc(&(device_grid.xh), device_grid.NX*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.yh), device_grid.NY*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.zh), device_grid.NZ*sizeof(float)) );

    gpuErrchk( hipMalloc(&(device_grid.xf), device_grid.NX*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.yf), device_grid.NY*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.zf), device_grid.NZ*sizeof(float)) );
    // allocate the device memory for U/V/W
    gpuErrchk( hipMalloc(&device_u_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_v_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_w_time_chunk, MX*MY*MZ*nT*sizeof(float)) );

    //vorticity device arrays we have to calculate
    gpuErrchk( hipMalloc(&device_xvort_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_yvort_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_zvort_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    // allocate device memory for our parcel positions
    gpuErrchk( hipMalloc(&(device_parcels.xpos), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.ypos), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.zpos), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclu), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclv), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclw), parcels.nParcels * totTime * sizeof(float)) );

    // copy the arrays to device memory
    gpuErrchk( hipMemcpy(device_u_time_chunk, u_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_v_time_chunk, v_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_w_time_chunk, w_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(device_parcels.xpos, parcels.xpos, parcels.nParcels * totTime * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_parcels.ypos, parcels.ypos, parcels.nParcels * totTime * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_parcels.zpos, parcels.zpos, parcels.nParcels * totTime * sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(device_grid.xh, grid.xh, device_grid.NX*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.yh, grid.yh, device_grid.NY*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.zh, grid.zh, device_grid.NZ*sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(device_grid.xf, grid.xf, device_grid.NX*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.yf, grid.yf, device_grid.NY*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.zf, grid.zf, device_grid.NZ*sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipDeviceSynchronize() );
    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((MX/threadsPerBlock.x)+1, (MY/threadsPerBlock.y)+1, (MZ/threadsPerBlock.z)+1); 
    cout << "Calculating vorticity" << endl;
    calcvort<<<numBlocks, threadsPerBlock>>>(device_grid, device_u_time_chunk, device_v_time_chunk, device_w_time_chunk, device_xvort_time_chunk, device_yvort_time_chunk, device_zvort_time_chunk, MX, MY, MZ, tStart, tEnd, totTime);
    gpuErrchk( hipDeviceSynchronize() );
    doAvg<<<numBlocks, threadsPerBlock>>>(device_xvort_time_chunk, device_yvort_time_chunk, device_zvort_time_chunk, MX, MY, MZ, tStart, tEnd, totTime);
    gpuErrchk( hipDeviceSynchronize() );
    cout << "End vorticity calc" << endl;
    test<<<parcels.nParcels,1>>>(device_grid, device_parcels, device_u_time_chunk, device_v_time_chunk, device_w_time_chunk, MX, MY, MZ, tStart, tEnd, totTime);
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipMemcpy(parcels.xpos, device_parcels.xpos, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.ypos, device_parcels.ypos, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.zpos, device_parcels.zpos, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclu, device_parcels.pclu, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclv, device_parcels.pclv, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclw, device_parcels.pclw, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );

    gpuErrchk( hipDeviceSynchronize() );

    hipFree(device_grid.xh);
    hipFree(device_grid.yh);
    hipFree(device_grid.zh);
    hipFree(device_grid.xf);
    hipFree(device_grid.yf);
    hipFree(device_grid.zf);
    hipFree(device_parcels.xpos);
    hipFree(device_parcels.ypos);
    hipFree(device_parcels.zpos);
    hipFree(device_parcels.pclu);
    hipFree(device_parcels.pclv);
    hipFree(device_parcels.pclw);
    hipFree(device_u_time_chunk);
    hipFree(device_v_time_chunk);
    hipFree(device_w_time_chunk);
    hipFree(device_zvort_time_chunk);

    gpuErrchk( hipDeviceSynchronize() );
    cout << "FINISHED CUDA" << endl;
}

#endif
