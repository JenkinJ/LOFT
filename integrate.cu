#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "datastructs.cu"
#include "macros.cpp"
#include "interp.cu"
#include "momentum.cu"
#include "turb.cu"
#include "vort.cu"
#include "diff6.cu"
#ifndef INTEGRATE_CU
#define INTEGRATE_CU

// this is an error checking helper function for processes
// that run on the GPU. Without calling this, the GPU can
// fail to execute but the program won't crash or report it.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
       std::cout << hipGetErrorString(code) << std::endl;
      if (abort) exit(code);
   }
}


/*  Execute all of the required kernels on the GPU that are necessary for computing the 3
    components of vorticity. The idea here is that we're building wrappers on wrappers to
    simplify the process for the end user that just wants to calculate vorticity. This is
    also a necessary adjustment because the tendency calculations will require multiple
    steps, so transitioning this block of code as a proof of concept for how the programming
    model should work. */
void doCalcVort(datagrid *grid, model_data *data, int tStart, int tEnd, dim3 numBlocks, dim3 threadsPerBlock, hipStream_t stream) {
    // calculate the three compionents of vorticity
    calcvort<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream) );
    gpuErrchk( hipPeekAtLastError() );

    // Average the vorticity to the scalar grid using the temporary
    // arrays we allocated. After doing the averaging, we have to 
    // set the pointers to the temporary arrays as the new xvort,
    // yvort, and zvort, and set the old x/y/zvort arrays as the new
    // temporary arrays. Note: may have to zero those out in the future...
    doVortAvg<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );

    zeroTemArrays<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
} 

void doMomentumBud(datagrid *grid, model_data *data, int tStart, int tEnd, dim3 numBlocks, dim3 threadsPerBlock, hipStream_t stream) {
    calcpi<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk(hipPeekAtLastError());
    calcpgradw<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
}

void doCalcVortTend(datagrid *grid, model_data *data, int tStart, int tEnd, dim3 numBlocks, dim3 threadsPerBlock, hipStream_t stream) {

    doCalcRf<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    calcpi<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk(hipPeekAtLastError());

    // Compute the vorticity tendency due to stretching. These conveniently
    // end up on the scalar grid, and no extra steps are required. This will
    // compute the tendency for all 3 components of vorticity. 
    calcvortstretch<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );

    // Compute the vertical vorticity tendency due to tilting. We have to do 
    // each component individually because we have to average the arrays back
    // to the scalar grid. It's a mess. 
    calcxvorttilt<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doXVortTiltAvg<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    zeroTemArrays<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );

    calcyvorttilt<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doYVortTiltAvg<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    zeroTemArrays<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );

    calczvorttilt<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doZVortTiltAvg<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    zeroTemArrays<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );

    // Do the SGS turbulence closure calculations
    doCalcDef<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doGetTau<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doCalcTurb<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );

    zeroTemArrays<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doTurbVort<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );

    // Average the vorticity to the scalar grid using the temporary
    // arrays we allocated. After doing the averaging, we have to 
    // set the pointers to the temporary arrays as the new xvort,
    // yvort, and zvort, and set the old x/y/zvort arrays as the new
    // temporary arrays. Note: may have to zero those out in the future...
    doTurbVortAvg<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );

    zeroTemArrays<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    calcvortsolenoid<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk(hipPeekAtLastError());


    zeroTemArrays<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    calcvortbaro<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk(hipPeekAtLastError());



    /* U momentum tendency due to 6th order numerical diffusion */
    zeroTemArrays<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doCalcDiffUXYZ<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doCalcDiffU<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );

    /* V momentum tendency due to 6th order numerical diffusion */
    zeroTemArrays<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doCalcDiffVXYZ<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doCalcDiffV<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );

    /* W momentum tendency due to 6th order numerical diffusion */
    zeroTemArrays<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doCalcDiffWXYZ<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doCalcDiffW<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );

    /* Vorticity tendency due to 6th order numerical diffusion */
    zeroTemArrays<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doDiffVort<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    doDiffVortAvg<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
    zeroTemArrays<<<numBlocks, threadsPerBlock, 0, stream>>>(grid, data, tStart, tEnd);
    //gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk( hipPeekAtLastError() );
}

__global__ void integrate(datagrid *grid, parcel_pos *parcels, model_data *data, \
                          int tStart, int tEnd, int totTime, int direct) {

	//int parcel_id = blockIdx.x;
    int parcel_id = blockIdx.x * blockDim.x + threadIdx.x;

    // safety check to make sure our thread index doesn't
    // go out of our array bounds
    if (parcel_id < parcels->nParcels) {
        bool is_ugrd = false;
        bool is_vgrd = false;
        bool is_wgrd = false;

        float pcl_x, pcl_y, pcl_z;
        float pcl_u, pcl_v, pcl_w;
        float uu1, vv1, ww1;
        float point[3];

        // loop over the number of time steps we are
        // integrating over
        float dt = grid->dt; 
        float dt2 = dt / 2.;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {

            // get the current values of various fields interpolated
            // to the parcel before we integrate using the RK2 step
            point[0] = parcels->xpos[PCL(tidx, parcel_id, totTime)];
            point[1] = parcels->ypos[PCL(tidx, parcel_id, totTime)];
            point[2] = parcels->zpos[PCL(tidx, parcel_id, totTime)];
            pcl_x = point[0];
            pcl_y = point[1];
            pcl_z = point[2];
            if (( pcl_x > xf(grid->NX-4) ) || ( pcl_y > yf(grid->NY-4) ) || ( pcl_z > zf(grid->NZ-4) ) \
             || ( pcl_x < xf(0) )        || ( pcl_y < yf(0) )        || ( pcl_z < 0. ) ) {
                break;
            }


            is_ugrd = true;
            is_vgrd = false;
            is_wgrd = false;
            pcl_u = interp3D(grid, data->ustag, point, is_ugrd, is_vgrd, is_wgrd, tidx);

            is_ugrd = false;
            is_vgrd = true;
            is_wgrd = false;
            pcl_v = interp3D(grid, data->vstag, point, is_ugrd, is_vgrd, is_wgrd, tidx);

            is_ugrd = false;
            is_vgrd = false;
            is_wgrd = true;
            pcl_w = interp3D(grid, data->wstag, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            parcels->pclu[PCL(tidx,   parcel_id, totTime)] = pcl_u;
            parcels->pclv[PCL(tidx,   parcel_id, totTime)] = pcl_v;
            parcels->pclw[PCL(tidx,   parcel_id, totTime)] = pcl_w;

            // Now we use an RK2 scheme to integrate forward
            // in time. Values are interpolated to the parcel 
            // at the beginning of the next data time step. 
            for (int nkrp = 1; nkrp <= 2; ++nkrp) {        
                if (nkrp == 1) {
                    // integrate X position forward by the U wind
                    point[0] = pcl_x + pcl_u * dt * direct;
                    // integrate Y position forward by the V wind
                    point[1] = pcl_y + pcl_v * dt * direct;
                    // integrate Z position forward by the W wind
                    point[2] = pcl_z + pcl_w * dt * direct;
                    if ((pcl_u == -999.0) || (pcl_v == -999.0) || (pcl_w == -999.0)) {
                        printf("Warning: missing values detected at x: %f y:%f z:%f with ground bounds X0: %f Y0: %f Z0: %f X1: %f Y1: %f Z1: %f\n", \
                            point[0], point[1], point[2], xh(0), yh(0), zh(0), xh(grid->NX-1), yh(grid->NY-1), zh(grid->NZ-1));
                        return;
                    }
                    uu1 = pcl_u;
                    vv1 = pcl_v;
                    ww1 = pcl_w;
                }
                else {
                    is_ugrd = true;
                    is_vgrd = false;
                    is_wgrd = false;
                    pcl_u = interp3D(grid, data->ustag, point, is_ugrd, is_vgrd, is_wgrd, tidx);

                    is_ugrd = false;
                    is_vgrd = true;
                    is_wgrd = false;
                    pcl_v = interp3D(grid, data->vstag, point, is_ugrd, is_vgrd, is_wgrd, tidx);

                    is_ugrd = false;
                    is_vgrd = false;
                    is_wgrd = true;
                    pcl_w = interp3D(grid, data->wstag, point, is_ugrd, is_vgrd, is_wgrd, tidx);

                    // integrate X position forward by the U wind
                    point[0] = pcl_x + (pcl_u + uu1) * dt2 * direct;
                    // integrate Y position forward by the V wind
                    point[1] = pcl_y + (pcl_v + vv1) * dt2 * direct;
                    // integrate Z position forward by the W wind
                    point[2] = pcl_z + (pcl_w + ww1) * dt2 * direct;
                    if ((pcl_u == -999.0) || (pcl_v == -999.0) || (pcl_w == -999.0)) {
                        printf("Warning: missing values detected at x: %f y:%f z:%f with ground bounds X0: %f Y0: %f Z0: %f X1: %f Y1: %f Z1: %f\n", \
                            point[0], point[1], point[2], xh(0), yh(0), zh(0), xh(grid->NX-1), yh(grid->NY-1), zh(grid->NZ-1));
                        return;
                    }
                }
            } // end RK loop

            parcels->xpos[PCL(tidx+1, parcel_id, totTime)] = point[0]; 
            parcels->ypos[PCL(tidx+1, parcel_id, totTime)] = point[1];
            parcels->zpos[PCL(tidx+1, parcel_id, totTime)] = point[2];
        } // end time loop
    } // end index check
}

__global__ void parcel_interp(datagrid *grid, parcel_pos *parcels, model_data *data, \
                          int tStart, int tEnd, int totTime, int direct) {

	//int parcel_id = blockIdx.x;
    int parcel_id = blockIdx.x * blockDim.x + threadIdx.x;
    // get the io config from the user namelist
    iocfg *io = parcels->io;

    // safety check to make sure our thread index doesn't
    // go out of our array bounds
    if (parcel_id < parcels->nParcels) {
        bool is_ugrd = false;
        bool is_vgrd = false;
        bool is_wgrd = false;

        float point[3];

        // loop over the number of time steps we are
        // integrating over
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            point[0] = parcels->xpos[PCL(tidx, parcel_id, totTime)];
            point[1] = parcels->ypos[PCL(tidx, parcel_id, totTime)];
            point[2] = parcels->zpos[PCL(tidx, parcel_id, totTime)];
            if (io->output_kmh) {
                is_ugrd = false;
                is_vgrd = false;
                is_wgrd = true;
                float pclkmh = interp3D(grid, data->kmh, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclkmh[PCL(tidx, parcel_id, totTime)] = pclkmh;
            }

            if (io->output_momentum_budget) {
                is_ugrd = true;
                is_vgrd = false;
                is_wgrd = false;
                float pclupgrad = interp3D(grid, data->pgradu, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pcluturb = interp3D(grid, data->turbu, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pcludiff = interp3D(grid, data->diffu, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                is_ugrd = false;
                is_vgrd = true;
                is_wgrd = false;
                float pclvpgrad = interp3D(grid, data->pgradv, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclvturb = interp3D(grid, data->turbv, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclvdiff = interp3D(grid, data->diffv, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                is_ugrd = false;
                is_vgrd = false;
                is_wgrd = true;
                float pclwpgrad = interp3D(grid, data->pgradw, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclwturb = interp3D(grid, data->turbw, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclwdiff = interp3D(grid, data->diffw, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclupgrad[PCL(tidx,   parcel_id, totTime)] = pclupgrad;
                parcels->pclvpgrad[PCL(tidx,   parcel_id, totTime)] = pclvpgrad;
                parcels->pclwpgrad[PCL(tidx,   parcel_id, totTime)] = pclwpgrad;
                parcels->pcluturb[PCL(tidx,   parcel_id, totTime)] = pcluturb;
                parcels->pclvturb[PCL(tidx,   parcel_id, totTime)] = pclvturb;
                parcels->pclwturb[PCL(tidx,   parcel_id, totTime)] = pclwturb;
                parcels->pcludiff[PCL(tidx,   parcel_id, totTime)] = pcludiff;
                parcels->pclvdiff[PCL(tidx,   parcel_id, totTime)] = pclvdiff;
                parcels->pclwdiff[PCL(tidx,   parcel_id, totTime)] = pclwdiff;
            }


            // interpolate scalar values to the parcel point
            is_ugrd = false;
            is_vgrd = false;
            is_wgrd = false;
            if (io->output_vorticity_budget || io->output_xvort) {
                float pclxvort = interp3D(grid, data->xvort, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclxvort[PCL(tidx, parcel_id, totTime)] = pclxvort;
            }
            if (io->output_vorticity_budget || io->output_yvort) {
                float pclyvort = interp3D(grid, data->yvort, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclyvort[PCL(tidx, parcel_id, totTime)] = pclyvort;
            }
            if (io->output_vorticity_budget || io->output_zvort) {
                float pclzvort = interp3D(grid, data->zvort, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclzvort[PCL(tidx, parcel_id, totTime)] = pclzvort;
            }
            if (io->output_vorticity_budget) {
                float pclxvorttilt = interp3D(grid, data->xvtilt, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclyvorttilt = interp3D(grid, data->yvtilt, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclzvorttilt = interp3D(grid, data->zvtilt, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclxvortstretch = interp3D(grid, data->xvstretch, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclyvortstretch = interp3D(grid, data->yvstretch, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclzvortstretch = interp3D(grid, data->zvstretch, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclxvortturb = interp3D(grid, data->turbxvort, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclyvortturb = interp3D(grid, data->turbyvort, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclzvortturb = interp3D(grid, data->turbzvort, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclxvortdiff = interp3D(grid, data->diffxvort, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclyvortdiff = interp3D(grid, data->diffyvort, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclzvortdiff = interp3D(grid, data->diffzvort, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclxvortbaro = interp3D(grid, data->xvort_baro, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclyvortbaro = interp3D(grid, data->yvort_baro, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclxvortsolenoid = interp3D(grid, data->xvort_solenoid, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclyvortsolenoid = interp3D(grid, data->yvort_solenoid, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                float pclzvortsolenoid = interp3D(grid, data->zvort_solenoid, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                // Store the vorticity in the parcel
                parcels->pclxvorttilt[PCL(tidx, parcel_id, totTime)] = pclxvorttilt;
                parcels->pclyvorttilt[PCL(tidx, parcel_id, totTime)] = pclyvorttilt;
                parcels->pclzvorttilt[PCL(tidx, parcel_id, totTime)] = pclzvorttilt;
                parcels->pclxvortstretch[PCL(tidx, parcel_id, totTime)] = pclxvortstretch;
                parcels->pclyvortstretch[PCL(tidx, parcel_id, totTime)] = pclyvortstretch;
                parcels->pclzvortstretch[PCL(tidx, parcel_id, totTime)] = pclzvortstretch;
                parcels->pclxvortturb[PCL(tidx, parcel_id, totTime)] = pclxvortturb;
                parcels->pclyvortturb[PCL(tidx, parcel_id, totTime)] = pclyvortturb;
                parcels->pclzvortturb[PCL(tidx, parcel_id, totTime)] = pclzvortturb;
                parcels->pclxvortdiff[PCL(tidx, parcel_id, totTime)] = pclxvortdiff;
                parcels->pclyvortdiff[PCL(tidx, parcel_id, totTime)] = pclyvortdiff;
                parcels->pclzvortdiff[PCL(tidx, parcel_id, totTime)] = pclzvortdiff;
                parcels->pclxvortbaro[PCL(tidx, parcel_id, totTime)] = pclxvortbaro;
                parcels->pclyvortbaro[PCL(tidx, parcel_id, totTime)] = pclyvortbaro;
                parcels->pclxvortsolenoid[PCL(tidx, parcel_id, totTime)] = pclxvortsolenoid;
                parcels->pclyvortsolenoid[PCL(tidx, parcel_id, totTime)] = pclyvortsolenoid;
                parcels->pclzvortsolenoid[PCL(tidx, parcel_id, totTime)] = pclzvortsolenoid;
            }

            // Now do the scalars
            if (io->output_ppert) {
                float pclppert = interp3D(grid, data->prespert, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclppert[PCL(tidx, parcel_id, totTime)] = pclppert;
            }
            if (io->output_qvpert) {
                float pclqvpert = interp3D(grid, data->qvpert, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclqvpert[PCL(tidx, parcel_id, totTime)] = pclqvpert;
            }
            if (io->output_rhopert) {
                float pclrhopert = interp3D(grid, data->rhopert, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclrhopert[PCL(tidx, parcel_id, totTime)] = pclrhopert;
            }
            if (io->output_thetapert) {
                float pclthetapert = interp3D(grid, data->thetapert, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclthetapert[PCL(tidx, parcel_id, totTime)] = pclthetapert;
            }
            if (io->output_thrhopert) {
                float pclthrhopert = interp3D(grid, data->thrhopert, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclthrhopert[PCL(tidx, parcel_id, totTime)] = pclthrhopert;
            }

            if (io->output_pbar) {
                float pclpbar = interp1D(grid, grid->p0, point[2], is_wgrd, tidx);
                parcels->pclpbar[PCL(tidx, parcel_id, totTime)] = pclpbar;
            }
            if (io->output_qvbar) {
                float pclqvbar = interp1D(grid, grid->qv0, point[2], is_wgrd, tidx);
                parcels->pclqvbar[PCL(tidx, parcel_id, totTime)] = pclqvbar;
            }
            if (io->output_rhobar) {
                float pclrhobar = interp1D(grid, grid->rho0, point[2], is_wgrd, tidx);
                parcels->pclrhobar[PCL(tidx, parcel_id, totTime)] = pclrhobar;
            }
            if (io->output_thetabar) {
                float pclthetabar = interp1D(grid, grid->th0, point[2], is_wgrd, tidx);
                parcels->pclthetabar[PCL(tidx, parcel_id, totTime)] = pclthetabar;
            }
            if (io->output_rhobar) {
                float pclthrhobar = interp1D(grid, grid->th0, point[2], is_wgrd, tidx);
                parcels->pclthrhobar[PCL(tidx, parcel_id, totTime)] = pclthrhobar;
            }

            if (io->output_qc) {
                float pclqc = interp3D(grid, data->qc, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclqc[PCL(tidx, parcel_id, totTime)] = pclqc;
            }
            if (io->output_qi) {
                float pclqi = interp3D(grid, data->qi, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclqi[PCL(tidx, parcel_id, totTime)] = pclqi;
            }
            if (io->output_qs) {
                float pclqs = interp3D(grid, data->qs, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclqs[PCL(tidx, parcel_id, totTime)] = pclqs;
            }
            if (io->output_qg) {
                float pclqg = interp3D(grid, data->qg, point, is_ugrd, is_vgrd, is_wgrd, tidx);
                parcels->pclqg[PCL(tidx, parcel_id, totTime)] = pclqg;
            }
        }
    }
}

/*This function handles allocating memory on the GPU, transferring the CPU
arrays to GPU global memory, calling the integrate GPU kernel, and then
updating the position vectors with the new stuff*/
void cudaIntegrateParcels(datagrid *grid, model_data *data, parcel_pos *parcels, int nT, int totTime, int direct) {

    int tStart, tEnd;
    tStart = 0;
    tEnd = nT;
    int NX, NY, NZ;
    // set the NX, NY, NZ
    // variables for calculations
    NX = grid->NX;
    NY = grid->NY;
    NZ = grid->NZ;
    iocfg *io = parcels->io;

    hipStream_t calStream;
    hipStream_t intStream;
    hipStreamCreate(&calStream);
    hipStreamCreate(&intStream);


    // set the thread/block execution strategy for the kernels

    // Okay, so I think the last remaining issue might lie here. For some reason, some blocks 
    // must not be executing or something, seemingly related to the threadsPerBlock size. 
    // Changing to 4x4x4 fixed for xvort, but not yvort. I think we need to dynamically set
    // threadsPerBloc(x, y, z) based on the size of our grid at a given time step. 
    dim3 threadsPerBlock(8, 8, 6);
    dim3 numBlocks((int)ceil(NX/threadsPerBlock.x)+1, (int)ceil(NY/threadsPerBlock.y)+1, (int)ceil(NZ/threadsPerBlock.z)+1); 

    // Calculate the three compionents of vorticity
    // and do the necessary averaging. This is a wrapper that
    // calls the necessary kernels and assigns the pointers
    // appropriately such that the "user" only has to call this method.
    if (io->output_xvort || io->output_yvort || io->output_zvort || io->output_vorticity_budget) {
        doCalcVort(grid, data, tStart, tEnd, numBlocks, threadsPerBlock, calStream);
    }
    

    // Calculate the vorticity forcing terms for each of the 3 components.
    // This is a wrapper that calls the necessary kernels to compute the
    // derivatives and average them back to the scalar grid where necessary. 
    if (io->output_vorticity_budget || io->output_momentum_budget) doCalcVortTend(grid, data, tStart, tEnd, numBlocks, threadsPerBlock, calStream);
    if (io->output_momentum_budget) doMomentumBud(grid, data, tStart, tEnd, numBlocks, threadsPerBlock, calStream);


    // Before integrating the trajectories, George Bryan sets some below-grid/surface conditions 
    // that we need to consider. This handles applying those boundary conditions. 
    //applyMomentumBC<<<numBlocks, threadsPerBlock>>>(data->ustag, data->vstag, data->wstag, NX, NY, NZ, tStart, tEnd);
    //gpuErrchk(hipDeviceSynchronize() );
    //gpuErrchk( hipPeekAtLastError() );

    // integrate the parcels forward in time and interpolate
    // calculations to trajectories. 
    int nThreads = 256;
    int nPclBlocks = int(parcels->nParcels / nThreads) + 1;
    integrate<<<nPclBlocks, nThreads, 0, intStream>>>(grid, parcels, data, tStart, tEnd, totTime, direct);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );

    parcel_interp<<<nPclBlocks, nThreads, 0, intStream>>>(grid, parcels, data, tStart, tEnd, totTime, direct);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );
}
#endif

