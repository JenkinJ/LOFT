#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "datastructs.cu"
#include "macros.cpp"
#include "interp.cu"
#ifndef INTEGRATE_CU
#define INTEGRATE_CU

using namespace std;
// this is an error checking helper function for processes
// that run on the GPU. Without calling this, the GPU can
// fail to execute but the program won't crash or report it.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      cout << hipGetErrorString(code) << endl;
      if (abort) exit(code);
   }
}



__device__ void calc_xvort(datagrid *grid, float *wstag, float *vstag, float *xvort, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];
}

__device__ void calc_yvort(datagrid *grid, float *ustag, float *wstag, float *yvort, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

}

__device__ void calc_zvort(datagrid *grid, float *ustag, float *vstag, float *zvort, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

}

/* When doing the parcel trajectory integration, George Bryan does
   some fun stuff with the lower boundaries of the arrays, presumably
   to prevent the parcels from exiting out the bottom of the domain
   or experience artificial values */
__global__ void applyMomentumBC(float *ustag, float *vstag, float *wstag, int NX, int NY, int NZ, int tStart, int tEnd) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    // this is done for easy comparison to CM1 code
    int ni = NX; int nj = NY; int nk = NZ;

    // this is a lower boundary condition, so only when k is 0
    // also this is on the u staggered mesh
    if (( j >= 0 ) && ( i >= 0) && ( j < nj+1) && ( i < ni+2) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the u stagger macro to handle the
            // proper indexing
            UA4D(i, j, 0, tidx) = UA4D(i, j, 1, tidx);
        }
    }
    
    // do the same but now on the v staggered grid
    if (( j >= 0 ) && ( i >= 0) && ( j < nj+2) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the v stagger macro to handle the
            // proper indexing
            VA4D(i, j, 0, tidx) = VA4D(i, j, 1, tidx);
        }
    }

    // do the same but now on the w staggered grid
    if (( j >= 0 ) && ( i >= 0) && ( j < nj+1) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the w stagger macro to handle the
            // proper indexing
            WA4D(i, j, 0, tidx) = -1*WA4D(i, j, 2, tidx);
        }
    }
}

/* Kernel for computing the components of vorticity
    and vorticity forcing terms. We do this using our domain subset containing the parcels
    instead of doing it locally for each parcel, as it would scale poorly for large 
    numbers of parcels. */
__global__ void calcvort(datagrid *grid, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, \
                        float *xvort, float *yvort, float *zvort, \
                        int MX, int MY, int MZ, int tStart, int tEnd, int totTime) {

    // get our 3D index based on our blocks/threads
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int idx_4D[4];
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < MX-1) && (j < MY-1) && (k < MZ-1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort(grid, w_time_chunk, v_time_chunk, xvort, idx_4D, MX, MY, MZ);
            calc_yvort(grid, u_time_chunk, w_time_chunk, yvort, idx_4D, MX, MY, MZ);
            calc_zvort(grid, u_time_chunk, v_time_chunk, zvort, idx_4D, MX, MY, MZ);
        }
    }
}

__global__ void integrate(datagrid *grid, parcel_pos *parcels, integration_data *data, \
                          int tStart, int tEnd, int totTime, int direct) {

	int parcel_id = blockIdx.x;
    // safety check to make sure our thread index doesn't
    // go out of our array bounds
    if (parcel_id < parcels->nParcels) {
        bool is_ugrd = false;
        bool is_vgrd = false;
        bool is_wgrd = false;

        float pcl_u, pcl_v, pcl_w;
        float point[3];

        // loop over the number of time steps we are
        // integrating over
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            point[0] = parcels->xpos[PCL(tidx, parcel_id, totTime)];
            point[1] = parcels->ypos[PCL(tidx, parcel_id, totTime)];
            point[2] = parcels->zpos[PCL(tidx, parcel_id, totTime)];
            //printf("My Point Is: X = %f Y = %f Z = %f t = %d nParcels = %d\n", point[0], point[1], point[2], tidx, parcels->nParcels);

            is_ugrd = true;
            is_vgrd = false;
            is_wgrd = false;
            pcl_u = interp3D(grid, data->u_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);

            is_ugrd = false;
            is_vgrd = true;
            is_wgrd = false;
            pcl_v = interp3D(grid, data->v_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);

            is_ugrd = false;
            is_vgrd = false;
            is_wgrd = true;
            pcl_w = interp3D(grid, data->w_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            //printf("pcl u: %f pcl v: %f pcl w: %f\n", pcl_u, pcl_v, pcl_w);
            
            // integrate X position forward by the U wind
            point[0] += pcl_u * (1.0f/6.0f) * direct;
            // integrate Y position forward by the V wind
            point[1] += pcl_v * (1.0f/6.0f) * direct;
            // integrate Z position forward by the W wind
            point[2] += pcl_w * (1.0f/6.0f) * direct;
            if ((pcl_u == -999.0) || (pcl_v == -999.0) || (pcl_w == -999.0)) {
                printf("Warning: missing values detected at x: %f y:%f z:%f with ground bounds X0: %f Y0: %f Z0: %f X1: %f Y1: %f Z1: %f\n", \
                    point[0], point[1], point[2], grid->xh[0], grid->yh[0], grid->zh[0], grid->xh[grid->NX-1], grid->yh[grid->NY-1], grid->zh[grid->NZ-1]);
                return;
            }


            parcels->xpos[PCL(tidx+1, parcel_id, totTime)] = point[0]; 
            parcels->ypos[PCL(tidx+1, parcel_id, totTime)] = point[1];
            parcels->zpos[PCL(tidx+1, parcel_id, totTime)] = point[2];
            parcels->pclu[PCL(tidx,   parcel_id, totTime)] = pcl_u;
            parcels->pclv[PCL(tidx,   parcel_id, totTime)] = pcl_v;
            parcels->pclw[PCL(tidx,   parcel_id, totTime)] = pcl_w;
        }
    }
}

/*This function handles allocating memory on the GPU, transferring the CPU
arrays to GPU global memory, calling the integrate GPU kernel, and then
updating the position vectors with the new stuff*/
void cudaIntegrateParcels(datagrid *grid, integration_data *data, parcel_pos *parcels, int nT, int totTime, int direct) {

    int tStart, tEnd;
    tStart = 0;
    tEnd = nT;
    int NX, NY, NZ;
    // set the NX, NY, NZ
    // variables for calculations
    NX = grid->NX;
    NY = grid->NY;
    NZ = grid->NZ;


    // set the thread/block execution strategy for the kernels
    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((NX/threadsPerBlock.x)+1, (NY/threadsPerBlock.y)+1, (NZ/threadsPerBlock.z)+1); 

    // we synchronize the device before doing anything to make sure all
    // array memory transfers have safely completed. This is probably 
    // unnecessary but I'm doing it anyways because overcaution never
    // goes wrong. Ever.
    gpuErrchk( hipDeviceSynchronize() );

    // Before integrating the trajectories, George Bryan sets some below-grid/surface conditions 
    // that we need to consider. This handles applying those boundary conditions. 
    applyMomentumBC<<<numBlocks, threadsPerBlock>>>(data->u_4d_chunk, data->v_4d_chunk, data->w_4d_chunk, NX, NY, NZ, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize() );

    // integrate the parcels forward in time and interpolate
    // calculations to trajectories. 
    integrate<<<parcels->nParcels, 1>>>(grid, parcels, data, tStart, tEnd, totTime, direct);
    gpuErrchk(hipDeviceSynchronize() );

}

#endif
