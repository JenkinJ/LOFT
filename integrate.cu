#include "hip/hip_runtime.h"
#include <iostream>
#include "datastructs.cpp"
#ifndef INTEGRATE_CU
#define INTEGRATE_CU

using namespace std;
// this is an error checking helper function for processes
// that run on the GPU. Without calling this, the GPU can
// fail to execute but the program won't crash or report it.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      cout << hipGetErrorString(code) << endl;
      if (abort) exit(code);
   }
}

// This is a GPU kernel for integrating a fluid parcel forward in time by increment dt.
// It expects the X, Y, and Z array pointers as well as the U, V, and W wind component pointers,
// which are all of length nParcels
__global__ void integrate(float *x_arr, float *y_arr, float *z_arr, float *u_arr, float *v_arr, float *w_arr, int nParcels, float dt) {
	// use the thread index to index the array
	int pidx = blockIdx.x*blockDim.x + threadIdx.x;

	// safety check to not access array memory out of bounds
	if (pidx < nParcels) {
		// integrate X position forward by the U wind
		x_arr[pidx] = x_arr[pidx] + u_arr[pidx] * dt;
		// integrate Y position forward by the V wind
		y_arr[pidx] = y_arr[pidx] + v_arr[pidx] * dt;
		// integrate Z position forward by the W wind
		z_arr[pidx] = z_arr[pidx] + w_arr[pidx] * dt;
	}
}

__global__ void test(float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, int MX, int MY, int MZ, int nT) {
    int N = MX*MY*MZ*nT;
    float umax = -999.0;
    float vmax = -999.0;
    float wmax = -999.0;

    for (int i = 0; i < N; ++i) {
        if (u_time_chunk[i] > umax) umax = u_time_chunk[i];
        if (v_time_chunk[i] > vmax) vmax = v_time_chunk[i];
        if (w_time_chunk[i] > wmax) wmax = w_time_chunk[i];
    }
    printf("Umax: %f\tVmax: %f\tWmax: %f\n", umax, vmax, wmax);
}

/*This function handles allocating memory on the GPU, transferring the CPU
arrays to GPU global memory, calling the integrate GPU kernel, and then
updating the position vectors with the new stuff*/
void cudaIntegrateParcels(parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, int MX, int MY, int MZ, int nT) {
    // pointers to device memory
    float *device_u_time_chunk, *device_v_time_chunk, *device_w_time_chunk;
    // allocate the device memory
    gpuErrchk( hipMalloc(&device_u_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_v_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_w_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    // copy the arrays to device memory
    gpuErrchk( hipMemcpy(device_u_time_chunk, u_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_v_time_chunk, v_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_w_time_chunk, w_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );

    test<<<1,1>>>(device_u_time_chunk, device_v_time_chunk, device_w_time_chunk, MX, MY, MZ, nT);
    gpuErrchk( hipDeviceSynchronize() );



}

#endif
