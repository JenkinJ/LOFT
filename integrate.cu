#include "hip/hip_runtime.h"
#include <iostream>
#include "datastructs.cpp"
#ifndef INTEGRATE_CU
#define INTEGRATE_CU

using namespace std;
// this is an error checking helper function for processes
// that run on the GPU. Without calling this, the GPU can
// fail to execute but the program won't crash or report it.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      cout << hipGetErrorString(code) << endl;
      if (abort) exit(code);
   }
}

// This is a GPU kernel for integrating a fluid parcel forward in time by increment dt.
// It expects the X, Y, and Z array pointers as well as the U, V, and W wind component pointers,
// which are all of length nParcels
__global__ void integrate(float *x_arr, float *y_arr, float *z_arr, float *u_arr, float *v_arr, float *w_arr, int nParcels, float dt) {
	// use the thread index to index the array
	int pidx = blockIdx.x*blockDim.x + threadIdx.x;

	// safety check to not access array memory out of bounds
	if (pidx < nParcels) {
		// integrate X position forward by the U wind
		x_arr[pidx] = x_arr[pidx] + u_arr[pidx] * dt;
		// integrate Y position forward by the V wind
		y_arr[pidx] = y_arr[pidx] + v_arr[pidx] * dt;
		// integrate Z position forward by the W wind
		z_arr[pidx] = z_arr[pidx] + w_arr[pidx] * dt;
	}
}

__global__ void test(parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, int MX, int MY, int MZ, int nT) {
    int parcel_id = blockIdx.x + threadIdx.x;
    if (parcel_id < parcels.nParcels) {
        printf("Parcel Number: %d\tParcel X: %f\tParcel Y: %f\t Parcel Z: %f\n", parcel_id, parcels.xpos[parcel_id], parcels.ypos[parcel_id], parcels.zpos[parcel_id]);
    }
}

/*This function handles allocating memory on the GPU, transferring the CPU
arrays to GPU global memory, calling the integrate GPU kernel, and then
updating the position vectors with the new stuff*/
void cudaIntegrateParcels(parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, int MX, int MY, int MZ, int nT) {
    // pointers to device memory
    float *device_u_time_chunk, *device_v_time_chunk, *device_w_time_chunk;
    parcel_pos device_parcels;
    device_parcels.nParcels = parcels.nParcels;

    // allocate the device memory
    gpuErrchk( hipMalloc(&device_u_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_v_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_w_time_chunk, MX*MY*MZ*nT*sizeof(float)) );

    gpuErrchk( hipMalloc(&(device_parcels.xpos), parcels.nParcels * nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.ypos), parcels.nParcels * nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.zpos), parcels.nParcels * nT*sizeof(float)) );

    // copy the arrays to device memory
    gpuErrchk( hipMemcpy(device_u_time_chunk, u_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_v_time_chunk, v_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_w_time_chunk, w_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(device_parcels.xpos, parcels.xpos, parcels.nParcels * nT * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_parcels.ypos, parcels.ypos, parcels.nParcels * nT * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_parcels.zpos, parcels.zpos, parcels.nParcels * nT * sizeof(float), hipMemcpyHostToDevice) );

    test<<<parcels.nParcels,1>>>(device_parcels, device_u_time_chunk, device_v_time_chunk, device_w_time_chunk, MX, MY, MZ, nT);
    gpuErrchk( hipDeviceSynchronize() );



}

#endif
