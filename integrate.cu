#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "datastructs.cpp"
#include "interp.cu"
#ifndef INTEGRATE_CU
#define INTEGRATE_CU

using namespace std;
#define P2(t,p,mt) (((p)*(mt))+(t))
// this is an error checking helper function for processes
// that run on the GPU. Without calling this, the GPU can
// fail to execute but the program won't crash or report it.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      cout << hipGetErrorString(code) << endl;
      if (abort) exit(code);
   }
}


// rf is the rho field on the vertical staggered mesh. George does this for turbulence closure
// since K terms and derivatives are all done on the W mesh. This is calculated by doing an extrapolation 
// from the scalar mesh to the W staggered mesh. As described below, c1 and c2 are distances of the staggered mesh from
// the scalar mesh normalized by the grid spacing dz, and since in an isotropic mesh the stagger is exactly half way
// between scalar points, is 0.5. This is hard coded, but even in our stretch zone above 10km, it's really close
// to 0.5. This is probably only violated for wildly stretched meshes, which we don't use because they're dumb.
// Mostly just noting this for future reference, because this will need correcting for stretched meshes.
__device__ void calcrf(datagrid grid, float *rho, float *rf, int *idx_4D, int MX, int MY, int MZ) {

    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];
    // c1 and c2 are both 0.5 for isotropic staggered meshes. We are hard coding this to be the case here for our data,
    // but is not necessarily true for all simulations.
    float c1 = 0.5; float c2 = 0.5;
    if ( k == 0) {
        float rho1 = grid.rho0[k] + rho[arrayIndex(i, j, 1, t, MX, MY, MZ)]; 
        float rho2 = grid.rho0[k] + rho[arrayIndex(i, j, 2, t, MX, MY, MZ)];
        float rho3 = grid.rho0[k] + rho[arrayIndex(i, j, 3, t, MX, MY, MZ)];
        rf[arrayIndex(i, j, 0, t, MX, MY, MZ)] = (1.75*rho1-rho2+0.25*rho3);
    }
    else {

        float rho1 = grid.rho0[k] + rho[arrayIndex(i, j, k, t, MX, MY, MZ)]; 
        float rho2 = grid.rho0[k] + rho[arrayIndex(i, j, k+1, t, MX, MY, MZ)];
        rf[arrayIndex(i, j, k, t, MX, MY, MZ)] = ( c1*rho1 + c2*rho2);
    }
    // there's technically a top boundary condition in CM1, but we're ignoring because we hope to be far away from the upper boundary.
}

// calculate the deformation terms for the turbulence diagnostics. They get stored in the 
// arrays later designated for tau stress tensors and variables are named according to
// tensor notation
__device__ void calcdef(datagrid grid, float *uarr, float *varr, float *warr, float *rho, float *rf,\
                        float *s11, float *s12, float *s13, float *s22, float *s23, float *s33, \
                        int *idx_4D, int MX, int MY, int MZ) {

    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dx = grid.xf[i+1] - grid.xf[i];

    // tau 11. Derivative is du/dx therefore use the staggered mesh and forward difference it to get it on the scalar mesh
    float rho1 = grid.rho0[k] + rho[arrayIndex(i, j, k, t, MX, MY, MZ)];
    s11[arrayIndex(i, j, k, t, MX, MY, MZ)] = rho1*( uarr[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - uarr[arrayIndex(i, j, k, t, MX, MY, MZ)] ) / dx;


    // tau 12. Derivatives are no longer on the staggered meshes since it's du/dy and dv/dx. Therefore, an averaging step
    // will be required later at some point.
    float dy = grid.yh[j+1] - grid.yh[j];
    dx = grid.xh[i+1] - grid.xh[i];
    float rho2 = grid.rho0[k] + rho[arrayIndex(i+1, j+1, k, t, MX, MY, MZ)];
    float rho3 = grid.rho0[k] + rho[arrayIndex(i, j+1, k, t, MX, MY, MZ)];
    float rho4 = grid.rho0[k] + rho[arrayIndex(i+1, j, k, t, MX, MY, MZ)];
    s12[arrayIndex(i, j, k, t, MX, MY, MZ)] = 0.5 * ( ( uarr[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - uarr[arrayIndex(i, j, k, t, MX, MY, MZ)] ) / dy \
                                            + ( varr[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - varr[arrayIndex(i, j, k, t, MX, MY, MZ)] ) / dx ) \
                                            * 0.25*( (rho1 + rho2) + (rho3 + rho4) );

   // tau 22. Derivative is dv/dy therefore use the staggered mesh and forward difference it to get it on the scalar mesh
   dy = grid.yf[j+1] - grid.yf[j];
   s22[arrayIndex(i, j, k, t, MX, MY, MZ)] = rho1 * ( varr[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - varr[arrayIndex(i, j, k, t, MX, MY, MZ)] ) / dy;


   // tau 33. Derivative is du/dz and therefore use the staggered mesh and forward difference it to get it on the scalar mesh
   float dz = grid.zf[k+1] - grid.zf[k];
   s33[arrayIndex(i, j, k, t, MX, MY, MZ)] = rho1 * ( warr[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - warr[arrayIndex(i, k, k, t, MX, MY, MZ)] ) / dz;

   // data above the lower boundary
   if ( k >= 1.) {

       // tau 13. Derivative is no longer on staggered mesh and will require an average to correct the data to the scalar mesh
       dx = grid.xh[i+1] - grid.xh[i];
       dz = grid.zh[k+1] - grid.zh[k];
       s13[arrayIndex(i, j, k, t, MX, MY, MZ)] = 0.5*( ( warr[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - warr[arrayIndex(i, j, k, t, MX, MY, MZ)] ) / dx \
                                               + ( uarr[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - uarr[arrayIndex(i, j, k, t, MX, MY, MZ)] ) / dz ) \
                                                 *0.5*( rf[arrayIndex(i, j, k, t, MX, MY, MZ)] + rf[arrayIndex(i+1, j, k, t, MX, MY, MZ)]);

       // tau 23. Derivative is no longer on staggered mesh and will require an average to correct the data to the scalar mesh
       dy = grid.yh[j+1] - grid.yh[j];
       s23[arrayIndex(i, j, k, t, MX, MY, MZ)] = 0.5*( ( warr[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - warr[arrayIndex(i, j, k, t, MX, MY, MZ)] ) / dy \
                                               + ( varr[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - varr[arrayIndex(i, j, k, t, MX, MY, MZ)] ) / dz) \
                                               *0.5*( rf[arrayIndex(i, j, k, t, MX, MY, MZ)]+rf[arrayIndex(i, j+1, k, t, MX, MY, MZ)] );

   }

   // lower boundary condition
   else {
       s13[arrayIndex(i, j, k, t, MX, MY, MZ)] = 0.0;
       s23[arrayIndex(i, j, k, t, MX, MY, MZ)] = 0.0;
   }
}

// take the output from calcdef and compute the full stress tensor tau
// !NOTE: turb coefficients are defined on w points
__device__ void gettau(datagrid grid, float *rho, float *khh, \
                        float *t11, float *t12, float *t13, float *t22, float *t23, float *t33, \
                        int *idx_4D, int MX, int MY, int MZ) {

    // get the i,j,k,t index of where we are doing
    // our differencing
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];


    // kmh = Pr * khh and Pr is 1./3.
    float tem = (1./3.)*khh[arrayIndex(i, k, k, t, MX, MY, MZ)] + (1./3.)*khh[arrayIndex(i, j, k+1, t, MX, MY, MZ)];


    // these are conveniently on points we know... but that convecience will end shortly
    t11[arrayIndex(i, j, k, t, MX, MY, MZ)] = t11[arrayIndex(i, j, k, t, MX, MY, MZ)] * tem;

    t22[arrayIndex(i, j, k, t, MX, MY, MZ)] = t22[arrayIndex(i, j, k, t, MX, MY, MZ)] * tem;

    t33[arrayIndex(i, j, k, t, MX, MY, MZ)] = t33[arrayIndex(i, j, k, t, MX, MY, MZ)] * tem;

    // do some 8 point averaging of our kmh in space
    float kmh1 = khh[arrayIndex(i-1, j-1, k, t, MX, MY, MZ )] * 1./3.;
    float kmh2 = khh[arrayIndex(i, j, k, t, MX, MY, MZ )] * 1./3.;
    float kmh3 = khh[arrayIndex(i-1, j, k, t, MX, MY, MZ )] * 1./3.;
    float kmh4 = khh[arrayIndex(i, j-1, k, t, MX, MY, MZ  )] * 1./3.;
    float kmh5 = khh[arrayIndex(i-1, j-1, k+1, t, MX, MY, MZ )] * 1./3.;
    float kmh6 = khh[arrayIndex(i, j, k+1, t, MX, MY, MZ )] * 1./3.;
    float kmh7 = khh[arrayIndex(i-1, j, k+1, t, MX, MY, MZ )] * 1./3.;
    float kmh8 = khh[arrayIndex(i,j-1,k+1, t, MX, MY, MZ)] * 1./3.;
    t12[arrayIndex(i, j, k, t, MX, MY, MZ)] = t12[arrayIndex(i, j, k, t, MX, MY, MZ)]*.025 \
    *( ( (kmh1+kmh2)+(kmh3+kmh4) )+( (kmh5+kmh6)+(kmh7+kmh8) ) );

    if (k >= 1) {
        float kmv1 = kmh2; 
        float kmv2 = khh[arrayIndex(i+1, j, k, t, MX, MY, MZ)] * 1./3.;
        float kmv3 = khh[arrayIndex(i, j+1, k, t, MX, MY, MZ)] * 1./3.;
        t13[arrayIndex(i, j, k, t, MX, MY, MZ)] = t13[arrayIndex(i, j, k, t, MX, MY, MZ)] * ( kmv1 + kmv2 );
        t23[arrayIndex(i, j, k, t, MX, MY, MZ)] = t23[arrayIndex(i, j, k, t, MX, MY, MZ)] * ( kmv1 + kmv3 ); 

    }
    else {

        // lower boundary condition
        t13[arrayIndex(i, j, k, t, MX, MY, MZ)] = 0.0;
        t23[arrayIndex(i, j, k, t, MX, MY, MZ)] = 0.0;

    }
    
}

// NOTE: Care has been taken to best preserve the numerical accuracy of how things are done in CM1, 
// but some areas where George does a backwards difference have been replaced with a forward difference.
// It is unknown to me at this time how much this will impact the overall result, but it could result in
// a pesky "off by one error" of sorts where in cartesian space, the values are slightly off. One day
// I'll probably revisit this and refactor it to make it 100% accurate with CM1, but Fortran's inconsistency
// with 0 indexed vs 1 indexed arrays confuse me sometimes, and we're not dealing with ghost zones

// calculate the turbulence term of the momentum equation for U. Eventually this will be modified to be
// the turbulence term for a component of vorticity (I think). 
__device__ void calc_turbu(datagrid grid, float *t11, float *t12, float *t13, float *turbx, float *turby, float *turbz, \
                        int *idx_4D, int MX, int MY, int MZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dx = grid.xh[i+1] - grid.xh[i];
    float dy = grid.yh[j+1] - grid.yh[j];
    float dz = grid.zh[k+1] - grid.zh[k];
    turbx[arrayIndex(i, j, k, t, MX, MY, MZ)] = (t11[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - t11[arrayIndex(i, j, k, t, MX, MY, MZ)])/dx;
    turby[arrayIndex(i, j, k, t, MX, MY, MZ)] = (t12[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - t12[arrayIndex(i, j, k, t, MX, MY, MZ)])/dy;
    turbz[arrayIndex(i, j, k, t, MX, MY, MZ)] = (t13[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - t13[arrayIndex(i, j, k, t, MX, MY, MZ)])/dz;

}


// calculate the turbulence term of the momentum equation for V. Eventually this will be modified to be
// the turbulence term for a component of vorticity (I think). 
__device__ void calc_turbv(datagrid grid, float *t12, float *t22, float *t23, float *turbx, float *turby, float *turbz,
                        int *idx_4D, int MX, int MY, int MZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dx = grid.xh[i+1] - grid.xh[i];
    float dy = grid.yh[j+1] - grid.yh[j];
    float dz = grid.zh[k+1] - grid.zh[k];
    turbx[arrayIndex(i, j, k, t, MX, MY, MZ)] = (t12[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - t12[arrayIndex(i, j, k, t, MX, MY, MZ)]) / dx;
    turby[arrayIndex(i, j, k, t, MX, MY, MZ)] = (t22[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - t22[arrayIndex(i, j, k, t, MX, MY, MZ)]) / dy;
    turbz[arrayIndex(i, j, k, t, MX, MY, MZ)] = (t23[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - t23[arrayIndex(i, j, k, t, MX, MY, MZ)]) / dz;

}

// calculate the turbulence term of the momentum equation for W. Eventually this will be modified to be
// the turbulence term for a component of vorticity (I think). 
__device__ void calc_turbw(datagrid grid, float *t13, float *t23, float *t33, float *turbx, float *turby, float *turbz,
                        int *idx_4D, int MX, int MY, int MZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dx = grid.xh[i+1] - grid.xh[i];
    float dy = grid.yh[j+1] - grid.yh[j];
    float dz = grid.zh[k+1] - grid.zh[k];
    turbx[arrayIndex(i, j, k, t, MX, MY, MZ)] = (t13[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - t13[arrayIndex(i, j, k, t, MX, MY, MZ)]) / dx;
    turby[arrayIndex(i, j, k, t, MX, MY, MZ)] = (t23[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - t23[arrayIndex(i, j, k, t, MX, MY, MZ)]) / dy;
    turbz[arrayIndex(i, j, k, t, MX, MY, MZ)] = (t33[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - t33[arrayIndex(i, j, k, t, MX, MY, MZ)]) / dz;

}

__device__ void calc_xvort(datagrid grid, float *warr, float *varr, float *xvort, int *idx_4D, int MX, int MY, int MZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dz = grid.zh[k+1] - grid.zh[k];
    float dy = grid.yh[j+1] - grid.yh[j];
    float dw = warr[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - warr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float dv = varr[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - varr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    xvort[arrayIndex(i, j, k, t, MX, MY, MZ)] = ( dw / dy ) - ( dv / dz);

}

__device__ void calc_yvort(datagrid grid, float *uarr, float *warr, float *yvort, int *idx_4D, int MX, int MY, int MZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dz = grid.zh[k+1] - grid.zh[k];
    float dx = grid.xh[i+1] - grid.xh[i];
    float dw = warr[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - warr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float du = uarr[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - uarr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    yvort[arrayIndex(i, j, k, t, MX, MY, MZ)] = ( du / dz ) - ( dw / dx);

}

__device__ void calc_zvort(datagrid grid, float *uarr, float *varr, float *zvort, int *idx_4D, int MX, int MY, int MZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dx = grid.xh[i+1] - grid.xh[i];
    float dy = grid.yh[j+1] - grid.yh[j];
    float dv = varr[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - varr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float du = uarr[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - uarr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    zvort[arrayIndex(i, j, k, t, MX, MY, MZ)] = ( dv / dx ) - ( du / dy);
    //printf("%f, %i, %i, %i, %i\n", zvort[arrayIndex(i, j, k, t, MX, MY, MZ)], i, j, k, t);
}
// tilting terms: do center difference, not forward
__device__ void calc_zvorttilt(datagrid grid, float *zvorttilt, float *xvort, float *yvort, float*warr, int *idx_4D, int MX, int MY, int MZ) {
	int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dx = grid.xh[i+1] - grid.xh[i-1];
    float dy = grid.yh[j+1] - grid.yh[j-1];
    float dwdx = (warr[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - warr[arrayIndex(i-1, j, k, t, MX, MY, MZ)]) / dx;
    float dwdy = (warr[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - warr[arrayIndex(i, j-1, k, t, MX, MY, MZ)]) / dy;
    float xv = xvort[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float yv = yvort[arrayIndex(i, j, k, t, MX, MY, MZ)];

    zvorttilt[arrayIndex(i, j, k, t, MX, MY, MZ)] = xv*dwdx + yv*dwdy;
    if (i == 1) zvorttilt[arrayIndex(0, j, k, t, MX, MY, MZ)] = zvorttilt[arrayIndex(i, j, k, t, MX, MY, MZ)];
    if (j == 1) zvorttilt[arrayIndex(i, 0, k, t, MX, MY, MZ)] = zvorttilt[arrayIndex(i, j, k, t, MX, MY, MZ)];

}

__device__ void calc_xvorttilt(datagrid grid, float *xvorttilt, float *yvort, float *zvort, float*uarr, int *idx_4D, int MX, int MY, int MZ) {
	int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dy = grid.yh[j+1] - grid.yh[j-1];
    float dz = grid.zh[k+1] - grid.zh[k-1];
    float dudy = (uarr[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - uarr[arrayIndex(i, j-1, k, t, MX, MY, MZ)]) / dy;
    float dudz = (uarr[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - uarr[arrayIndex(i, j, k-1, t, MX, MY, MZ)]) / dz;
    float yv = yvort[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float zv = zvort[arrayIndex(i, j, k, t, MX, MY, MZ)];

    xvorttilt[arrayIndex(i, j, k, t, MX, MY, MZ)] = yv*dudy + zv*dudz;
    if (j == 1) xvorttilt[arrayIndex(i, 0, k, t, MX, MY, MZ)] = xvorttilt[arrayIndex(i, j, k, t, MX, MY, MZ)];
    if (k == 1) xvorttilt[arrayIndex(i, j, 0, t, MX, MY, MZ)] = xvorttilt[arrayIndex(i, j, k, t, MX, MY, MZ)];

}

__device__ void calc_yvorttilt(datagrid grid, float *yvorttilt, float *xvort, float *zvort, float*varr, int *idx_4D, int MX, int MY, int MZ) {
	int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dx = grid.xh[i+1] - grid.xh[i-1];
    float dz = grid.zh[k+1] - grid.zh[k-1];
    float dvdx = (varr[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - varr[arrayIndex(i-1, j, k, t, MX, MY, MZ)]) / dx;
    float dvdz = (varr[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - varr[arrayIndex(i, j, k-1, t, MX, MY, MZ)]) / dz;
    float xv = xvort[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float zv = zvort[arrayIndex(i, j, k, t, MX, MY, MZ)];

    yvorttilt[arrayIndex(i, j, k, t, MX, MY, MZ)] = xv*dvdx + zv*dvdz;
    if (i == 1) yvorttilt[arrayIndex(0, j, k, t, MX, MY, MZ)] = yvorttilt[arrayIndex(i, j, k, t, MX, MY, MZ)];
    if (k == 1) yvorttilt[arrayIndex(i, j, 0, t, MX, MY, MZ)] = yvorttilt[arrayIndex(i, j, k, t, MX, MY, MZ)];

}

/* calculate the stretching term for vertical vorticity. The stretching functions naturally result on
scalar grid points, nothing needs to be done after this*/
__device__ void calc_zvortstretch(datagrid grid, float *zvortstretch, float *zvort, float *uarr, float *varr, int *idx_4D, int MX, int MY, int MZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dx = grid.xf[i+1] - grid.xf[i];
    float dy = grid.yf[j+1] - grid.yf[j];
    float du = uarr[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - uarr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float dv = varr[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - varr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float zv = zvort[arrayIndex(i, j, k, t, MX, MY, MZ)];

    // du/dx and dv/dy end up on the scalar grid, and zvort is already averaged to the
    // scalar grid, so we're done here.
    zvortstretch[arrayIndex(i, j, k, t, MX, MY, MZ)] = -1.*zv*( (du/dx) + (dv/dy) );

}

/* calculate the stretching term for vertical vorticity. The stretching functions*/
__device__ void calc_yvortstretch(datagrid grid, float *yvortstretch, float *yvort, float *uarr, float *warr, int *idx_4D, int MX, int MY, int MZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float dx = grid.xf[i+1] - grid.xf[i];
    float dz = grid.zf[k+1] - grid.zf[k];
    float du = uarr[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - uarr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float dw = warr[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - warr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float yv = yvort[arrayIndex(i, j, k, t, MX, MY, MZ)];

    // du/dx and dv/dy end up on the scalar grid, and zvort is already averaged to the
    // scalar grid, so we're done here.
    yvortstretch[arrayIndex(i, j, k, t, MX, MY, MZ)] = -1.*yv*( (du/dx) + (dw/dz) );

}

/* calculate the stretching term for vertical vorticity. The stretching functions*/
__device__ void calc_xvortstretch(datagrid grid, float *xvortstretch, float *xvort, float *varr, float *warr, int *idx_4D, int MX, int MY, int MZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];
    
    //printf("%i %i %i %i\n", i, j, k ,t);
   	float dz = grid.zf[k+1] - grid.zf[k];
    float dy = grid.yf[j+1] - grid.yf[j];

    


    float dv = varr[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - varr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float dw = warr[arrayIndex(i, j, k+1, t, MX, MY, MZ)] - warr[arrayIndex(i, j, k, t, MX, MY, MZ)];
    float xv = xvort[arrayIndex(i, j, k, t, MX, MY, MZ)];


    // du/dx and dv/dy end up on the scalar grid, and zvort is already averaged to the
    // scalar grid, so we're done here.
    xvortstretch[arrayIndex(i, j, k, t, MX, MY, MZ)] =  -1.*xv*( (dv/dy) + (dw/dz) );

}


__device__ void calc_xvortbaro(datagrid grid, float *xbaro, float *thrhopert, int *idx_4D, int MX, int MY, int MZ) {

	int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float reps = 461.5/287.04;
    float coeff = 9.81 / (grid.th0[k] * (1.0 + reps*grid.qv0[k]/(1.0+grid.qv0[k])));
    float dy = grid.yh[j+1] - grid.yh[j-1];

    xbaro[arrayIndex(i, j, k, t, MX, MY, MZ)] = coeff * (thrhopert[arrayIndex(i, j+1, k, t, MX, MY, MZ)] - thrhopert[arrayIndex(i, j-1, k, t, MX, MY, MZ)]) / dy;

}

__device__ void calc_yvortbaro(datagrid grid, float *ybaro, float *thrhopert, int *idx_4D, int MX, int MY, int MZ) {

	int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float reps = 461.5/287.04;
    float coeff = 9.81 / (grid.th0[k] * (1.0 + reps*grid.qv0[k]/(1.0+grid.qv0[k])));
    float dx = grid.xh[i+1] - grid.xh[i-1];

    ybaro[arrayIndex(i, j, k, t, MX, MY, MZ)] = -1.*coeff * (thrhopert[arrayIndex(i+1, j, k, t, MX, MY, MZ)] - thrhopert[arrayIndex(i-1, j, k, t, MX, MY, MZ)]) / dx;

}


__global__ void doCalcrf(datagrid grid, float *rho_time_chunk, float *rhof, int MX, int MY, int MZ, int tStart, int tEnd, int totTime) {

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int idx_4D[4];

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < MX-1) && (j < MY-1) && (k < MZ-1)) { 
        if ((i+1 > MX) || (j+1 > MY) || (k+1 > MZ)) printf("i+1 or j+1 out of bounds\n");
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calcrf(grid, rho_time_chunk, rhof, idx_4D, MX, MY, MZ);
        }
    }
}

__global__ void doCalcdef(datagrid grid, float *uarr, float *varr, float *warr, float *rho, float *rf, \
                        float *s11, float *s12, float *s13, float *s22, float *s23, float *s33, \
                        int MX, int MY, int MZ, int tStart, int tEnd, int totTime) {

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int idx_4D[4];

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < MX-1) && (j < MY-1) && (k < MZ-1)) { 
        if ((i+1 > MX) || (j+1 > MY) || (k+1 > MZ)) printf("i+1 or j+1 out of bounds\n");
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calcdef(grid, uarr, varr, warr, rho, rf, s11, s12, s13, s22, s23, s33, idx_4D, MX, MY, MZ);
        }
    }
}

/*
__global__ void doGettau() {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int idx_4D[4];

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < MX-1) && (j < MY-1) && (k < MZ-1)) { 
        if ((i+1 > MX) || (j+1 > MY) || (k+1 > MZ)) printf("i+1 or j+1 out of bounds\n");
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
        }
    }
}

__global__ void goTurbu() {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int idx_4D[4];

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < MX-1) && (j < MY-1) && (k < MZ-1)) { 
        if ((i+1 > MX) || (j+1 > MY) || (k+1 > MZ)) printf("i+1 or j+1 out of bounds\n");
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
        }
    }
}

__global__ void doTurbv() { 
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int idx_4D[4];

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < MX-1) && (j < MY-1) && (k < MZ-1)) { 
        if ((i+1 > MX) || (j+1 > MY) || (k+1 > MZ)) printf("i+1 or j+1 out of bounds\n");
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
        }
    }
}

__global__ void doTurbw() { 
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int idx_4D[4];

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < MX-1) && (j < MY-1) && (k < MZ-1)) { 
        if ((i+1 > MX) || (j+1 > MY) || (k+1 > MZ)) printf("i+1 or j+1 out of bounds\n");
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
        }
    }
}
*/

/* NOTE FOR VORTICITY KERNELS!!!!!!
Our subset doesn't have ghost zones per-say, but we assume
that our parcels aren't on the borders of the arrays. This is
partly becasue 1) we subset around the parcels by several
gridpoints in each direction and 2) don't really care about parcels 
leaving the domain anyway
*/

/* a kernel that applies the lower boundary conditions for
   xvort and yvort*/
__global__ void applyVortBCs(float *xvort, float *yvort, int MX, int MY, int MZ, int tStart, int tEnd, int totTime) { 
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    if ((i < MX-1) && (j < MY-1) && (k == 0)) { 
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            xvort[arrayIndex(i, j, 0, tidx, MX, MY, MZ)] = xvort[arrayIndex(i, j, 1, tidx, MX, MY, MZ)];
            yvort[arrayIndex(i, j, 0, tidx, MX, MY, MZ)] = yvort[arrayIndex(i, j, 1, tidx, MX, MY, MZ)];
        }
    }
}

/* For the vorticity calculations, you have to do a 4 point average
of the neighbors to get the point onto the scalar grid. Calcvort does
the initial pass, and then this gets called at the end to make sure that
the averaging happens */
__global__ void doVortAvg(float *xvort, float *yvort, float *zvort, int MX, int MY, int MZ, int tStart, int tEnd, int totTime) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    if ((i < MX-1) && (j < MY-1) && (k < MZ-1)) { 
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            xvort[arrayIndex(i, j, k, tidx, MX, MY, MZ)] = 0.25*(xvort[arrayIndex(i, j, k, tidx, MX, MY, MZ)] + \
                xvort[arrayIndex(i, j+1, k, tidx, MX, MY, MZ)] + xvort[arrayIndex(i, j, k+1, tidx, MX, MY, MZ)] + \
                xvort[arrayIndex(i, j+1, k+1, tidx, MX, MY, MZ)]);

            yvort[arrayIndex(i, j, k, tidx, MX, MY, MZ)] = 0.25*(yvort[arrayIndex(i, j, k, tidx, MX, MY, MZ)] + \
                yvort[arrayIndex(i+1, j, k, tidx, MX, MY, MZ)] + yvort[arrayIndex(i, j, k+1, tidx, MX, MY, MZ)] + \
                yvort[arrayIndex(i+1, j, k+1, tidx, MX, MY, MZ)]);

            zvort[arrayIndex(i, j, k, tidx, MX, MY, MZ)] = 0.25*(zvort[arrayIndex(i, j, k, tidx, MX, MY, MZ)] + \
                zvort[arrayIndex(i+1, j, k, tidx, MX, MY, MZ)] + zvort[arrayIndex(i, j+1, k, tidx, MX, MY, MZ)] + \
                zvort[arrayIndex(i+1, j+1, k, tidx, MX, MY, MZ)]);
        }
    }
}

__global__ void doVortTendAvg(float *xvorttilt, float *yvorttilt, float *zvorttilt, int MX, int MY, int MZ, int tStart, int tEnd, int totTime) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    if ((i < MX-1) && (j < MY-1) && (k < MZ-1)) { 
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            xvorttilt[arrayIndex(i, j, k, tidx, MX, MY, MZ)] = 0.5*(xvorttilt[arrayIndex(i, j, k, tidx, MX, MY, MZ)] + \
                xvorttilt[arrayIndex(i+1, j, k, tidx, MX, MY, MZ)]);

            yvorttilt[arrayIndex(i, j, k, tidx, MX, MY, MZ)] = 0.5*(yvorttilt[arrayIndex(i, j, k, tidx, MX, MY, MZ)] + \
                yvorttilt[arrayIndex(i, j+1, k, tidx, MX, MY, MZ)]);

            zvorttilt[arrayIndex(i, j, k, tidx, MX, MY, MZ)] = 0.5*(zvorttilt[arrayIndex(i, j, k, tidx, MX, MY, MZ)] + \
                zvorttilt[arrayIndex(i, j, k+1, tidx, MX, MY, MZ)]);
        }
    }
}


/* Kernel for computing the components of vorticity
    and vorticity forcing terms. We do this using our domain subset containing the parcels
    instead of doing it locally for each parcel, as it would scale poorly for large 
    numbers of parcels. */
__global__ void calcvort(datagrid grid, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, \
                        float *xvort, float *yvort, float *zvort, \
                        int MX, int MY, int MZ, int tStart, int tEnd, int totTime) {

    // get our 3D index based on our blocks/threads
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int idx_4D[4];
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < MX-1) && (j < MY-1) && (k < MZ-1)) { 
        if ((i+1 > MX) || (j+1 > MY) || (k+1 > MZ)) printf("i+1 or j+1 out of bounds\n");
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort(grid, w_time_chunk, v_time_chunk, xvort, idx_4D, MX, MY, MZ);
            calc_yvort(grid, u_time_chunk, w_time_chunk, yvort, idx_4D, MX, MY, MZ);
            // calculate the Z component of vorticity
            //printf("%i, %i, %i, %i, %i, %i, %i\n", i, j, k, tidx, MX, MY, MZ);
            calc_zvort(grid, u_time_chunk, v_time_chunk, zvort, idx_4D, MX, MY, MZ);

        }
    }
}

/* Calculate the vorticity tendency terms. Needs to be called after the calcvort function, the 4 point average, 
and the boundary conditions have been applied to xvort and yvort. Then this should all be fine? */
__global__ void calcvorttend(datagrid grid, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, \
							float *xvort, float *yvort, float *zvort, \
							float *xvortstretch, float *yvortstretch, float *zvortstretch, \
							float *xvorttilt, float *yvorttilt, float *zvorttilt, \
                            float *xvortbaro, float *yvortbaro, float *thrhopert, \
							int MX, int MY, int MZ, int tStart, int tEnd, int totTime) {
    // get our 3D index based on our blocks/threads
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int idx_4D[4];
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    
    if ((i < MX-1) && (j < MY-1) && (k < MZ-1)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;

			calc_xvortstretch(grid, xvortstretch, xvort, v_time_chunk, w_time_chunk, idx_4D, MX, MY, MZ );
			calc_yvortstretch(grid, yvortstretch, yvort, u_time_chunk, w_time_chunk, idx_4D, MX, MY, MZ );
			calc_zvortstretch(grid, zvortstretch, zvort, u_time_chunk, v_time_chunk, idx_4D, MX, MY, MZ );
        }
    }

    // these are centered differences and have different boundary requirements
    if ((i < MX-2) && (j < MY-2) && (k < MZ-2) && \
    	(i >= 1) && (j >= 1) && (k >=1)) { 
        if ((i+1 > MX) || (j+1 > MY) || (k+1 > MZ)) printf("i+1 or j+1 out of bounds\n");
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;

			calc_zvorttilt(grid, zvorttilt, xvort, yvort, w_time_chunk, idx_4D, MX, MY, MZ);
			calc_xvorttilt(grid, xvorttilt, yvort, zvort, u_time_chunk, idx_4D, MX, MY, MZ);
			calc_yvorttilt(grid, yvorttilt, xvort, zvort, v_time_chunk, idx_4D, MX, MY, MZ);
            calc_xvortbaro(grid, xvortbaro, thrhopert, idx_4D, MX, MY, MZ);
            calc_yvortbaro(grid, yvortbaro, thrhopert, idx_4D, MX, MY, MZ);
        }
    }
}

__global__ void test(datagrid grid, parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, \
                    float *p_time_chunk, float *th_time_chunk, \
                    float *xvort_time_chunk, float *yvort_time_chunk, float *zvort_time_chunk, \
                    float *xvorttilt_chunk, float *yvorttilt_chunk, float *zvorttilt_chunk, \
                    float *xvortstretch_chunk, float *yvortstretch_chunk, float *zvortstretch_chunk, \
                    float *xvortbaro_chunk, float *yvortbaro_chunk, \
                    int MX, int MY, int MZ, int tStart, int tEnd, int totTime, int direct) {

	int parcel_id = blockIdx.x;
    // safety check to make sure our thread index doesn't
    // go out of our array bounds
    if (parcel_id < parcels.nParcels) {
        bool is_ugrd = false;
        bool is_vgrd = false;
        bool is_wgrd = false;

        float pcl_u, pcl_v, pcl_w;
        float pcl_ppert, pcl_thrhoprime;
        float pcl_xvort, pcl_yvort, pcl_zvort;
        float pcl_xvorttilt, pcl_yvorttilt, pcl_zvorttilt;
        float pcl_xvortstretch, pcl_yvortstretch, pcl_zvortstretch;
        float pcl_xvortbaro, pcl_yvortbaro;
        float point[3];

        // loop over the number of time steps we are
        // integrating over
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // GPU sanity test of data integrity
            point[0] = parcels.xpos[P2(tidx, parcel_id, totTime)];
            point[1] = parcels.ypos[P2(tidx, parcel_id, totTime)];
            point[2] = parcels.zpos[P2(tidx, parcel_id, totTime)];

            is_ugrd = true;
            is_vgrd = false;
            is_wgrd = false;
            pcl_u = interp3D(grid, u_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);

            is_ugrd = false;
            is_vgrd = true;
            is_wgrd = false;
            pcl_v = interp3D(grid, v_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);

            is_ugrd = false;
            is_vgrd = false;
            is_wgrd = true;
            pcl_w = interp3D(grid, w_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);

            // get the vorticity components
            is_ugrd = false;
            is_vgrd = false;
            is_wgrd = false;
            pcl_xvort = interp3D(grid, xvort_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            pcl_yvort = interp3D(grid, yvort_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            pcl_zvort = interp3D(grid, zvort_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            pcl_xvorttilt = interp3D(grid, xvorttilt_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            pcl_yvorttilt = interp3D(grid, yvorttilt_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            pcl_zvorttilt = interp3D(grid, zvorttilt_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            pcl_xvortstretch = interp3D(grid, xvortstretch_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            pcl_yvortstretch = interp3D(grid, yvortstretch_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            pcl_zvortstretch = interp3D(grid, zvortstretch_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            pcl_xvortbaro = interp3D(grid, xvortbaro_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            pcl_yvortbaro = interp3D(grid, yvortbaro_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);

            pcl_ppert = interp3D(grid, p_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            pcl_thrhoprime = interp3D(grid, th_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);



            // integrate X position forward by the U wind
            point[0] += pcl_u * (1.0f/6.0f) * direct;
            // integrate Y position forward by the V wind
            point[1] += pcl_v * (1.0f/6.0f) * direct;
            // integrate Z position forward by the W wind
            point[2] += pcl_w * (1.0f/6.0f) * direct;
            if ((pcl_u == -999.0) || (pcl_v == -999.0) || (pcl_w == -999.0)) {
                printf("Warning: missing values detected at x: %f y:%f z:%f with ground bounds X0: %f Y0: %f Z0: %f X1: %f Y1: %f Z1: %f\n", \
                    point[0], point[1], point[2], grid.xh[0], grid.yh[0], grid.zh[0], grid.xh[grid.NX-1], grid.yh[grid.NY-1], grid.zh[grid.NZ-1]);
                return;
            }

            if (point[2] < grid.zf[0]) point[2] = grid.zf[0];


            parcels.xpos[P2(tidx+1, parcel_id, totTime)] = point[0]; 
            parcels.ypos[P2(tidx+1, parcel_id, totTime)] = point[1];
            parcels.zpos[P2(tidx+1, parcel_id, totTime)] = point[2];
            parcels.pclu[P2(tidx, parcel_id, totTime)] = pcl_u;
            parcels.pclv[P2(tidx, parcel_id, totTime)] = pcl_v;
            parcels.pclw[P2(tidx, parcel_id, totTime)] = pcl_w;
            parcels.pclppert[P2(tidx, parcel_id, totTime)] = pcl_ppert;
            parcels.pclthrhoprime[P2(tidx, parcel_id, totTime)] = pcl_thrhoprime;

            parcels.pclxvort[P2(tidx, parcel_id, totTime)] = pcl_xvort;
            parcels.pclyvort[P2(tidx, parcel_id, totTime)] = pcl_yvort;
            parcels.pclzvort[P2(tidx, parcel_id, totTime)] = pcl_zvort;
            parcels.pclxvorttilt[P2(tidx, parcel_id, totTime)] = pcl_xvorttilt;
            parcels.pclyvorttilt[P2(tidx, parcel_id, totTime)] = pcl_yvorttilt;
            parcels.pclzvorttilt[P2(tidx, parcel_id, totTime)] = pcl_zvorttilt;
            parcels.pclxvortstretch[P2(tidx, parcel_id, totTime)] = pcl_xvortstretch;
            parcels.pclyvortstretch[P2(tidx, parcel_id, totTime)] = pcl_yvortstretch;
            parcels.pclzvortstretch[P2(tidx, parcel_id, totTime)] = pcl_zvortstretch;
            parcels.pclxvortbaro[P2(tidx, parcel_id, totTime)] = pcl_xvortbaro;
            parcels.pclyvortbaro[P2(tidx, parcel_id, totTime)] = pcl_yvortbaro;
        }
    }
}

/*This function handles allocating memory on the GPU, transferring the CPU
arrays to GPU global memory, calling the integrate GPU kernel, and then
updating the position vectors with the new stuff*/
void cudaIntegrateParcels(datagrid grid, parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, \
                         float *p_time_chunk, float *th_time_chunk, float *rho_time_chunk, \
                         int MX, int MY, int MZ, int nT, int totTime, int direct) {
    // pointers to device memory
    float *device_u_time_chunk, *device_v_time_chunk, *device_w_time_chunk;
    float *device_p_time_chunk, *device_th_time_chunk, *device_rho_time_chunk;
    float *device_xvort_time_chunk, *device_yvort_time_chunk, *device_zvort_time_chunk;
    float *device_xvortstretch_chunk, *device_yvortstretch_chunk, *device_zvortstretch_chunk;
    float *device_xvorttilt_chunk, *device_yvorttilt_chunk, *device_zvorttilt_chunk;
    float *device_xvortbaro_chunk, *device_yvortbaro_chunk, *device_rhof_time_chunk;
    float *device_t11_time_chunk, *device_t12_time_chunk, *device_t13_time_chunk;
    float *device_t22_time_chunk, *device_t23_time_chunk, *device_t33_time_chunk;

    parcel_pos device_parcels;
    datagrid device_grid;

    int tStart, tEnd;
    tStart = 0;
    tEnd = nT;

    // copy over our integer and long
    // constants to our device struct
    device_grid.X0 = grid.X0; device_grid.X1 = grid.X1;
    device_grid.Y0 = grid.Y0; device_grid.Y1 = grid.Y1;
    device_grid.Z0 = grid.Z0; device_grid.Z1 = grid.Z1;
    device_grid.NX = grid.NX; device_grid.NY = grid.NY;
    device_grid.NZ = grid.NZ; device_grid.nz = grid.nz;
    device_parcels.nParcels = parcels.nParcels;

    // allocate device memory for our grid arrays
    gpuErrchk( hipMalloc(&(device_grid.xh), device_grid.NX*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.yh), device_grid.NY*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.zh), device_grid.NZ*sizeof(float)) );

    gpuErrchk( hipMalloc(&(device_grid.th0), device_grid.nz*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.qv0), device_grid.nz*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.rho0), device_grid.nz*sizeof(float)) );

    gpuErrchk( hipMalloc(&(device_grid.xf), device_grid.NX*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.yf), device_grid.NY*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.zf), device_grid.NZ*sizeof(float)) );
    // allocate the device memory for U/V/W
    gpuErrchk( hipMalloc(&device_u_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_v_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_w_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_p_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_th_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_rho_time_chunk, MX*MY*MZ*nT*sizeof(float)) );

    //vorticity device arrays we have to calculate
    gpuErrchk( hipMalloc(&device_xvort_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_yvort_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_zvort_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    // vorticity tendency arrays
    gpuErrchk( hipMalloc(&device_xvortstretch_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_yvortstretch_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_zvortstretch_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_xvorttilt_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_yvorttilt_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_zvorttilt_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_xvortbaro_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_yvortbaro_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_rhof_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    // turbulent stress arrays
    gpuErrchk( hipMalloc(&device_t11_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_t12_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_t13_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_t22_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_t23_time_chunk, MX*MY*MZ*nT*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_t33_time_chunk, MX*MY*MZ*nT*sizeof(float)) );

    // allocate device memory for our parcel positions
    gpuErrchk( hipMalloc(&(device_parcels.xpos), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.ypos), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.zpos), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclu), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclv), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclw), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclppert), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclthrhoprime), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclxvort), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclyvort), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclzvort), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclxvorttilt), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclyvorttilt), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclzvorttilt), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclxvortstretch), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclyvortstretch), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclzvortstretch), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclxvortbaro), parcels.nParcels * totTime * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.pclyvortbaro), parcels.nParcels * totTime * sizeof(float)) );

    // copy the arrays to device memory
    gpuErrchk( hipMemcpy(device_u_time_chunk, u_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_v_time_chunk, v_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_w_time_chunk, w_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_p_time_chunk, p_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_th_time_chunk, th_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_rho_time_chunk, rho_time_chunk, MX*MY*MZ*nT*sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(device_parcels.xpos, parcels.xpos, parcels.nParcels * totTime * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_parcels.ypos, parcels.ypos, parcels.nParcels * totTime * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_parcels.zpos, parcels.zpos, parcels.nParcels * totTime * sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(device_grid.xh, grid.xh, device_grid.NX*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.yh, grid.yh, device_grid.NY*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.zh, grid.zh, device_grid.NZ*sizeof(float), hipMemcpyHostToDevice) );
    
    gpuErrchk( hipMemcpy(device_grid.th0, grid.th0, device_grid.nz*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.qv0, grid.qv0, device_grid.nz*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.rho0, grid.rho0, device_grid.nz*sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(device_grid.xf, grid.xf, device_grid.NX*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.yf, grid.yf, device_grid.NY*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.zf, grid.zf, device_grid.NZ*sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipDeviceSynchronize() );
    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((MX/threadsPerBlock.x)+1, (MY/threadsPerBlock.y)+1, (MZ/threadsPerBlock.z)+1); 

    cout << "Calculating rhof" << endl;

    doCalcrf<<<numBlocks, threadsPerBlock>>>(device_grid, device_rho_time_chunk, device_rhof_time_chunk, MX, MY, MZ, tStart, tEnd, totTime);
    gpuErrchk( hipDeviceSynchronize() );

    cout << "Calculating Deformation for Turbulence" << endl;
    doCalcdef<<<numBlocks, threadsPerBlock>>>(device_grid, device_u_time_chunk, device_v_time_chunk, device_w_time_chunk, device_rho_time_chunk, device_rhof_time_chunk, \
                        device_t11_time_chunk, device_t12_time_chunk, device_t13_time_chunk, device_t22_time_chunk, device_t23_time_chunk, device_t33_time_chunk, \
                        MX, MY, MZ, tStart, tEnd, totTime);
    gpuErrchk( hipDeviceSynchronize() );

    cout << "Calculating vorticity" << endl;
    calcvort<<<numBlocks, threadsPerBlock>>>(device_grid, device_u_time_chunk, device_v_time_chunk, device_w_time_chunk, \
                                            device_xvort_time_chunk, device_yvort_time_chunk, device_zvort_time_chunk, \
                                            MX, MY, MZ, tStart, tEnd, totTime);
    gpuErrchk( hipDeviceSynchronize() );

    applyVortBCs<<<numBlocks, threadsPerBlock>>>(device_xvort_time_chunk, device_yvort_time_chunk, MX, MY, MZ, tStart, tEnd, totTime);
    gpuErrchk( hipDeviceSynchronize() );
    
    doVortAvg<<<numBlocks, threadsPerBlock>>>(device_xvort_time_chunk, device_yvort_time_chunk, device_zvort_time_chunk, MX, MY, MZ, tStart, tEnd, totTime);
    gpuErrchk( hipDeviceSynchronize() );
    cout << "doing tendencies" << endl;
    calcvorttend<<<numBlocks, threadsPerBlock>>>(device_grid, device_u_time_chunk, device_v_time_chunk, device_w_time_chunk, \
    				device_xvort_time_chunk, device_yvort_time_chunk, device_zvort_time_chunk, \
    				device_xvortstretch_chunk, device_yvortstretch_chunk, device_zvortstretch_chunk, \
    				device_xvorttilt_chunk, device_yvorttilt_chunk, device_zvorttilt_chunk, \
                    device_xvortbaro_chunk, device_yvortbaro_chunk, device_th_time_chunk, \
    				MX, MY, MZ, tStart, tEnd, totTime);
    gpuErrchk( hipDeviceSynchronize() );

    doVortTendAvg<<<numBlocks, threadsPerBlock>>>(device_xvorttilt_chunk, device_yvorttilt_chunk, device_zvorttilt_chunk, MX, MY, MZ, tStart, tEnd, totTime);
    gpuErrchk( hipDeviceSynchronize() );
    cout << "ending tendencies" << endl;
    
    cout << "End vorticity calc" << endl;
    test<<<parcels.nParcels,1>>>(device_grid, device_parcels, device_u_time_chunk, device_v_time_chunk, device_w_time_chunk, \
                                device_p_time_chunk, device_th_time_chunk, \
                                device_xvort_time_chunk, device_yvort_time_chunk, device_zvort_time_chunk, \
                                device_xvorttilt_chunk, device_yvorttilt_chunk, device_zvorttilt_chunk, \
                                device_xvortstretch_chunk, device_yvortstretch_chunk, device_zvortstretch_chunk, \
                                device_xvortbaro_chunk, device_yvortbaro_chunk, \
                                MX, MY, MZ, tStart, tEnd, totTime, direct);
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipMemcpy(parcels.xpos, device_parcels.xpos, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.ypos, device_parcels.ypos, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.zpos, device_parcels.zpos, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclu, device_parcels.pclu, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclv, device_parcels.pclv, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclw, device_parcels.pclw, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclppert, device_parcels.pclppert, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclthrhoprime, device_parcels.pclthrhoprime, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclxvort, device_parcels.pclxvort, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclyvort, device_parcels.pclyvort, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclzvort, device_parcels.pclzvort, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclxvorttilt, device_parcels.pclxvorttilt, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclyvorttilt, device_parcels.pclyvorttilt, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclzvorttilt, device_parcels.pclzvorttilt, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclxvortstretch, device_parcels.pclxvortstretch, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclyvortstretch, device_parcels.pclyvortstretch, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclzvortstretch, device_parcels.pclzvortstretch, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclxvortbaro, device_parcels.pclxvortbaro, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.pclyvortbaro, device_parcels.pclyvortbaro, parcels.nParcels * totTime * sizeof(float), hipMemcpyDeviceToHost) );

    gpuErrchk( hipDeviceSynchronize() );

    hipFree(device_grid.xh);
    hipFree(device_grid.yh);
    hipFree(device_grid.zh);
    hipFree(device_grid.xf);
    hipFree(device_grid.yf);
    hipFree(device_grid.zf);
    hipFree(device_grid.th0);
    hipFree(device_grid.qv0);
    hipFree(device_grid.rho0);
    hipFree(device_parcels.xpos);
    hipFree(device_parcels.ypos);
    hipFree(device_parcels.zpos);
    hipFree(device_parcels.pclu);
    hipFree(device_parcels.pclv);
    hipFree(device_parcels.pclw);
    hipFree(device_parcels.pclppert);
    hipFree(device_parcels.pclthrhoprime);
    hipFree(device_parcels.pclxvort);
    hipFree(device_parcels.pclyvort);
    hipFree(device_parcels.pclzvort);
    hipFree(device_parcels.pclxvorttilt);
    hipFree(device_parcels.pclyvorttilt);
    hipFree(device_parcels.pclzvorttilt);
    hipFree(device_parcels.pclxvortstretch);
    hipFree(device_parcels.pclyvortstretch);
    hipFree(device_parcels.pclzvortstretch);
    hipFree(device_parcels.pclxvortbaro);
    hipFree(device_parcels.pclyvortbaro);
    hipFree(device_u_time_chunk);
    hipFree(device_v_time_chunk);
    hipFree(device_w_time_chunk);
    hipFree(device_p_time_chunk);
    hipFree(device_th_time_chunk);
    hipFree(device_rho_time_chunk);
    hipFree(device_xvort_time_chunk);
    hipFree(device_yvort_time_chunk);
    hipFree(device_zvort_time_chunk);
    hipFree(device_xvorttilt_chunk);
    hipFree(device_yvorttilt_chunk);
	hipFree(device_zvorttilt_chunk);
    hipFree(device_xvortstretch_chunk);
    hipFree(device_yvortstretch_chunk);
	hipFree(device_zvortstretch_chunk);
	hipFree(device_xvortbaro_chunk);
	hipFree(device_yvortbaro_chunk);
	hipFree(device_rhof_time_chunk);
	hipFree(device_t11_time_chunk);
	hipFree(device_t12_time_chunk);
	hipFree(device_t13_time_chunk);
	hipFree(device_t22_time_chunk);
	hipFree(device_t23_time_chunk);
	hipFree(device_t33_time_chunk);



    gpuErrchk( hipDeviceSynchronize() );
    cout << "FINISHED CUDA" << endl;
}

#endif
