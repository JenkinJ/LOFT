#include "hip/hip_runtime.h"
#include <iostream>
#include "datastructs.cpp"
#include "interp.cu"
#include <fstream>
#ifndef INTEGRATE_CU
#define INTEGRATE_CU

using namespace std;
// this is an error checking helper function for processes
// that run on the GPU. Without calling this, the GPU can
// fail to execute but the program won't crash or report it.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      cout << hipGetErrorString(code) << endl;
      if (abort) exit(code);
   }
}


__global__ void test(datagrid grid, parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, int MX, int MY, int MZ, int nT) {
	int parcel_id = blockIdx.x;
    // safety check to make sure our thread index doesn't
    // go out of our array bounds
    if (parcel_id < parcels.nParcels) {
        bool is_ugrd = false;
        bool is_vgrd = false;
        bool is_wgrd = false;

        float pcl_u, pcl_v, pcl_w;
        float point[3];

        // loop over the number of time steps we are
        // integrating over
        for (int tidx = 0; tidx < nT-1; ++tidx) {
            point[0] = parcels.xpos[tidx + (nT * parcel_id)];
            point[1] = parcels.ypos[tidx + (nT * parcel_id)];
            point[2] = parcels.zpos[tidx + (nT * parcel_id)];
            //printf("My Point to Integrate: x = %f\t y = %f\t z = %f\t parcel_id = %d\t time = %d\n", point[0], point[1], point[2], parcel_id, tidx);


            is_ugrd = true;
            is_vgrd = false;
            is_wgrd = false;
            pcl_u = interp3D(grid.xh, grid.yh, grid.zh, u_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);

            is_ugrd = false;
            is_vgrd = true;
            is_wgrd = false;
            pcl_v = interp3D(grid.xh, grid.yh, grid.zh, v_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);

            is_ugrd = false;
            is_vgrd = false;
            is_wgrd = true;
            pcl_w = interp3D(grid.xh, grid.yh, grid.zh, w_time_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx, MX, MY, MZ);
            //printf("My Parcel Vector Field: u = %f\t v = %f\t w = %f\n", pcl_u, pcl_v, pcl_w);

            // if the parcel has left the domain, exit
            if ((pcl_u == -999.0) || (pcl_v == -999.0) || (pcl_w == -999.0)) {
                return;
            }
            else {

                // integrate X position forward by the U wind
                point[0] += pcl_u * (1.0f/6.0f);
                // integrate Y position forward by the V wind
                point[1] += pcl_v * (1.0f/6.0f);
                // integrate Z position forward by the W wind
                point[2] += pcl_w * (1.0f/6.0f);


                parcels.xpos[(tidx + 1) + (nT * parcel_id)] = point[0]; 
                parcels.ypos[(tidx + 1) + (nT * parcel_id)] = point[1];
                parcels.zpos[(tidx + 1) + (nT * parcel_id)] = point[2];
            }
        }
    }
}

/*This function handles allocating memory on the GPU, transferring the CPU
arrays to GPU global memory, calling the integrate GPU kernel, and then
updating the position vectors with the new stuff*/
void cudaIntegrateParcels(datagrid grid, parcel_pos parcels, float *u_time_chunk, float *v_time_chunk, float *w_time_chunk, int MX, int MY, int MZ, int nT) {
    // pointers to device memory
    float *device_u_time_chunk, *device_v_time_chunk, *device_w_time_chunk;
    parcel_pos device_parcels;
    datagrid device_grid;

    // copy over our integer and long
    // constants to our device struct
    device_grid.X0 = grid.X0; device_grid.X1 = grid.X1;
    device_grid.Y0 = grid.Y0; device_grid.Y1 = grid.Y1;
    device_grid.Z0 = grid.Z0; device_grid.Z1 = grid.Z1;
    device_grid.NX = grid.NX; device_grid.NY = grid.NY;
    device_grid.NZ = grid.NZ; 
    device_parcels.nParcels = parcels.nParcels;

    // allocate device memory for our grid arrays
    gpuErrchk( hipMalloc(&(device_grid.xh), device_grid.NX*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.yh), device_grid.NY*sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_grid.zh), device_grid.NZ*sizeof(float)) );
    // allocate the device memory for U/V/W
    gpuErrchk( hipMalloc(&device_u_time_chunk, MX*MY*MZ*(nT-1)*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_v_time_chunk, MX*MY*MZ*(nT-1)*sizeof(float)) );
    gpuErrchk( hipMalloc(&device_w_time_chunk, MX*MY*MZ*(nT-1)*sizeof(float)) );
    // allocate device memory for our parcel positions
    gpuErrchk( hipMalloc(&(device_parcels.xpos), parcels.nParcels * nT * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.ypos), parcels.nParcels * nT * sizeof(float)) );
    gpuErrchk( hipMalloc(&(device_parcels.zpos), parcels.nParcels * nT * sizeof(float)) );

    // copy the arrays to device memory
    gpuErrchk( hipMemcpy(device_u_time_chunk, u_time_chunk, MX*MY*MZ*(nT-1)*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_v_time_chunk, v_time_chunk, MX*MY*MZ*(nT-1)*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_w_time_chunk, w_time_chunk, MX*MY*MZ*(nT-1)*sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipMemcpy(device_parcels.xpos, parcels.xpos, parcels.nParcels * nT * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_parcels.ypos, parcels.ypos, parcels.nParcels * nT * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_parcels.zpos, parcels.zpos, parcels.nParcels * nT * sizeof(float), hipMemcpyHostToDevice) );

    // don't need the staggered mesh sent
    gpuErrchk( hipMemcpy(device_grid.xh, grid.xh, device_grid.NX*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.yh, grid.yh, device_grid.NY*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(device_grid.zh, grid.zh, device_grid.NZ*sizeof(float), hipMemcpyHostToDevice) );

    test<<<parcels.nParcels,1>>>(device_grid, device_parcels, device_u_time_chunk, device_v_time_chunk, device_w_time_chunk, MX, MY, MZ, nT);
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipMemcpy(parcels.xpos, device_parcels.xpos, parcels.nParcels * nT * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.ypos, device_parcels.ypos, parcels.nParcels * nT * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(parcels.zpos, device_parcels.zpos, parcels.nParcels * nT * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipDeviceSynchronize() );

    hipFree(device_grid.xh);
    hipFree(device_grid.yh);
    hipFree(device_grid.zh);
    hipFree(device_parcels.xpos);
    hipFree(device_parcels.ypos);
    hipFree(device_parcels.zpos);


    ofstream outfile;
    outfile.open("./result.csv");
    
    // loop over each parcel
    for (int pcl = 0; pcl < parcels.nParcels; ++ pcl) {
        // print the parcel start flag 
        outfile << "!Parcel " << pcl << endl; 
        // loop over the times
        for (int t = 0; t < nT; ++t) {
            // for each row: x position, y position, z position
            for (int row = 0; row < 3; ++row) {
                if (row == 0) outfile << parcels.xpos[t + (pcl*nT)] << ", ";
                if (row == 1) outfile << parcels.ypos[t + (pcl*nT)] << ", ";
                if (row == 2) outfile << parcels.zpos[t + (pcl*nT)] << endl;
            }
        }
        // parcel end flag
        outfile << "!End " << pcl << endl;
    }
}

#endif
