#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "datastructs.cu"
#include "macros.cpp"
#include "interp.cu"
#ifndef INTEGRATE_CU
#define INTEGRATE_CU

using namespace std;
// this is an error checking helper function for processes
// that run on the GPU. Without calling this, the GPU can
// fail to execute but the program won't crash or report it.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      cout << hipGetErrorString(code) << endl;
      if (abort) exit(code);
   }
}

// rf is the rho field on the vertical staggered mesh. George does this for turbulence closure
// since K terms and derivatives are all done on the W mesh. This is calculated by doing an extrapolation 
// from the scalar mesh to the W staggered mesh. As described below, c1 and c2 are distances of the staggered mesh from
// the scalar mesh normalized by the grid spacing dz, and since in an isotropic mesh the stagger is exactly half way
// between scalar points, is 0.5. This is hard coded, but even in our stretch zone above 10km, it's really close
// to 0.5. This is probably only violated for wildly stretched meshes, which we don't use because they're dumb.
// Mostly just noting this for future reference, because this will need correcting for stretched meshes.
__device__ void calcrf(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    // c1 and c2 are both 0.5 for isotropic staggered meshes. We are hard coding this to be the case here for our data,
    // but is not necessarily true for all simulations.
    float c1 = 0.5; float c2 = 0.5;
    float *buf0;
    if (k == 0) {
        buf0 = data->rho_4d_chunk;
        float rho1 = grid->rho0[k] + BUF4D(i, j, 0, t);
        float rho2 = grid->rho0[k+1] + BUF4D(i, j, 1, t);
        float rho3 = grid->rho0[k+2] + BUF4D(i, j, 2, t);
        buf0 = data->rhof_4d_chunk;
        BUF4D(i, j, k, t) = (1.75*rho1) - rho2 + (0.25*rho3);
    }

    else { 
        buf0 = data->rho_4d_chunk;
        float rho1 = grid->rho0[k-1] + BUF4D(i, j, k-1, t);
        float rho2 = grid->rho0[k] + BUF4D(i, j, k, t);
        buf0 = data->rhof_4d_chunk;
        BUF4D(i, j, k, t) = ( c1*rho1 + c2*rho2);
    }
    // there's technically a top boundary condition in CM1, but we're ignoring because we hope to be far away from the upper boundary.
}

// calculate the deformation terms for the turbulence diagnostics. They get stored in the 
// arrays later designated for tau stress tensors and variables are named according to
// tensor notation
__device__ void calcdef(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *dum0, *buf0;
    float *ustag, *vstag, *wstag;
    ustag = data->u_4d_chunk;
    vstag = data->v_4d_chunk;
    wstag = data->w_4d_chunk;

    // tau 11. Derivative is du/dx and therefore the derivative on the staggered mesh results on the scalar point.
    dum0 = data->tem1_4d_chunk;
    TEM4D(i, j, k, t) = ( ( UA4D(i+1, j, k, t) - UA4D(i, j, k, t) ) / grid->dx ) * UH(i);

    // tau 12. Derivatives are no longer on the staggered mesh since it's du/dy and dv/dx. Therefore, and
    // averaging step must take place on the TEM array after calculation. 

    dum0 = data->tem2_4d_chunk;
    TEM4D(i, j, k, t) = ( ( ( UA4D(i, j, k, t) - UA4D(i, j-1, k, t) ) / grid->dy ) * VF(j) ) \
                        + ( ( ( VA4D(i, j, k, t) - VA4D(i-1, j, k, t) ) / grid->dx ) * UF(i) );

    // tau 22. Once again back on the scalar mesh. 
    dum0 = data->tem3_4d_chunk;
    TEM4D(i, j, k, t) = ( ( VA4D(i, j+1, k, t) - VA4D(i, j, k, t) ) / grid->dy ) * VH(j);

    // tau 33. On the scalar mesh. 
    dum0 = data->tem4_4d_chunk;
    TEM4D(i, j, k, t) = ( ( WA4D(i, j, k+1, t) - WA4D(i, j, k, t) ) / grid->dz ) * MH(k);

    if (k >= 1) {

        // tau 13 is not on the scalar mesh
        dum0 = data->tem6_4d_chunk;
        TEM4D(i, j, k, t) = ( ( ( WA4D(i, j, k, t) - WA4D(i-1, j, k, t) ) / grid->dx ) * UF(i) ) \
                           +( ( ( WA4D(i, j, k, t) - WA4D(i, j, k-1, t) ) / grid->dz ) * MF(k) );

        // tau 23 is not on the scalar mesh
        dum0 = data->tem6_4d_chunk;
        TEM4D(i, j, k, t) = ( ( ( WA4D(i, j, k, t) - WA4D(i, j-1, k, t) ) / grid->dy ) * VF(j) ) \
                           +( ( ( VA4D(i, j, k, t) - VA4D(i, j, k-1, t) ) / grid->dz ) * MF(k) );

    }
}

/* Compute the Exner function / nondimensionalized pressure */
__device__ void calc_pi(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    // This is p'
    float *buf0 = data->pres_4d_chunk;
    float pi = powf( BUF4D(i, j, k, t) / 1000., 0.28571426);
    buf0 = data->pi_4d_chunk;
    BUF4D(i, j, k, t) = pi;
}


/* Compute the x component of vorticity. After this is called by the calvort kernel, you must also run 
   the kernel for applying the lower boundary condition and then the kernel for averaging to the
   scalar grid. */
__device__ void calc_xvort(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *vstag = data->v_4d_chunk;
    float *wstag = data->w_4d_chunk;
    float *dum0 = data->tem1_4d_chunk;

    float dwdy = ( ( WA4D(i, j, k, t) - WA4D(i, j-1, k, t) )/grid->dy ) * VF(j);
    float dvdz = ( ( VA4D(i, j, k, t) - VA4D(i, j, k-1, t) )/grid->dz ) * MF(k);
    TEM4D(i, j, k, t) = dwdy - dvdz; 
}

/* Compute the y component of vorticity. After this is called by the calvort kernel, you must also run 
   the kernel for applying the lower boundary condition and then the kernel for averaging to the
   scalar grid. */
__device__ void calc_yvort(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *wstag = data->w_4d_chunk;
    float *dum0 = data->tem2_4d_chunk;

    float dwdx = ( ( WA4D(i, j, k, t) - WA4D(i-1, j, k, t) )/grid->dx ) * UF(i);
    float dudz = ( ( UA4D(i, j, k, t) - UA4D(i, j, k-1, t) )/grid->dz ) * MF(k);
    TEM4D(i, j, k, t) = dudz - dwdx;
}

/* Compute the z component of vorticity. After this is called by the calvort kernel, you must also run 
   the kernel for applying the lower boundary condition and then the kernel for averaging to the
   scalar grid. */
__device__ void calc_zvort(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *vstag = data->v_4d_chunk;
    float *dum0 = data->tem3_4d_chunk;

    float dvdx = ( ( VA4D(i, j, k, t) - VA4D(i-1, j, k, t) )/grid->dx) * UF(i);
    float dudy = ( ( UA4D(i, j, k, t) - UA4D(i, j-1, k, t) )/grid->dy) * VF(j);
    TEM4D(i, j, k, t) = dvdx - dudy;
}

/* Compute the X component of vorticity tendency due
   to tilting Y and Z components into the X direction */
__device__ void calc_xvort_tilt(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *vstag = data->v_4d_chunk;
    float *wstag = data->w_4d_chunk;

    // dudy in tem1
    float *dum0 = data->tem1_4d_chunk;
    TEM4D(i, j, k, t) = ( ( UA4D(i, j, k, t) - UA4D(i, j-1, k, t) ) / grid->dy ) * VF(j);

    // dwdx in tem2
    dum0 = data->tem2_4d_chunk;
    TEM4D(i, j, k, t) = ( ( WA4D(i, j, k, t) - WA4D(i-1, j, k, t) ) / grid->dx ) * UF(i);

    // dudz in tem3
    dum0 = data->tem3_4d_chunk;
    TEM4D(i, j, k, t) = ( ( UA4D(i, j, k, t) - UA4D(i, j, k-1, t) ) / grid->dz ) * MF(k);

    // dvdx in tem4
    dum0 = data->tem4_4d_chunk;
    TEM4D(i, j, k, t) = ( ( VA4D(i, j, k, t) - VA4D(i-1, j, k, t) ) / grid->dx ) * UF(i);
}

/* Compute the Y component of vorticity tendency due
   to tilting X and Z components into the X direction */
__device__ void calc_yvort_tilt(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *vstag = data->v_4d_chunk;
    float *wstag = data->w_4d_chunk;
    
    // dvdx in tem1
    float *dum0 = data->tem1_4d_chunk;
    TEM4D(i, j, k, t) = ( ( VA4D(i, j, k, t) - VA4D(i-1, j, k, t) ) / grid->dx ) * UF(i);

    // dwdy in tem2
    dum0 = data->tem2_4d_chunk;
    TEM4D(i, j, k, t) = ( ( WA4D(i, j, k, t) - WA4D(i, j-1, k, t) ) / grid->dy ) * VF(j);

    // dvdz in tem3
    dum0 = data->tem3_4d_chunk;
    TEM4D(i, j, k, t) = ( ( VA4D(i, j, k, t) - VA4D(i, j, k-1, t) ) / grid->dz ) * MF(k);

    // dudy in tem4
    dum0 = data->tem4_4d_chunk;
    TEM4D(i, j, k, t) = ( ( UA4D(i, j, k, t) - UA4D(i, j-1, k, t) ) / grid->dy ) * VF(j);
}

/* Compute the Z component of vorticity tendency due
   to tilting X and Y components into the X direction */
__device__ void calc_zvort_tilt(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *vstag = data->v_4d_chunk;
    float *wstag = data->w_4d_chunk;

    // Compute dw/dx and put it in the tem1 array. The derivatives
    // land on weird places so we have to average each derivative back
    // to the scalar grid, resulting in this clunky approach
    float *dum0 = data->tem1_4d_chunk;
    TEM4D(i, j, k, t) = ( ( WA4D(i, j, k, t) - WA4D(i-1, j, k, t) ) / grid->dx ) * UF(i);

    // put dv/dz in tem2
    dum0 = data->tem2_4d_chunk;
    TEM4D(i, j, k, t) = ( ( VA4D(i, j, k, t) - VA4D(i, j, k-1, t) ) / grid->dz ) * MF(k);

    // put dw/dy in tem3
    dum0 = data->tem3_4d_chunk;
    TEM4D(i, j, k, t) = ( ( WA4D(i, j, k, t) - WA4D(i, j-1, k, t) ) / grid->dy ) * VF(j);

    // put du/dz in tem4
    dum0 = data->tem4_4d_chunk;
    TEM4D(i, j, k, t) = ( ( UA4D(i, j, k, t) - UA4D(i, j, k-1, t) ) / grid->dz ) * MF(k);
}

/* Compute the X component of vorticity tendency due
   to stretching of the vorticity along the X axis. */
__device__ void calc_xvort_stretch(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *vstag = data->v_4d_chunk;
    float *wstag = data->w_4d_chunk;
    float *xvort = data->xvort_4d_chunk;
    float *xvort_stretch = data->xvstretch_4d_chunk;

    // this stencil conveniently lands itself on the scalar grid,
    // so we won't have to worry about doing any averaging. I think.
    float *buf0 = xvort;
    float xv = BUF4D(i, j, k, t);
    float dvdy = ( ( VA4D(i, j, k, t) - VA4D(i, j-1, k, t) )/grid->dy) * VF(j);
    float dwdz = ( ( WA4D(i, j, k, t) - WA4D(i, j, k-1, t) )/grid->dz) * MF(k);

    buf0 = xvort_stretch;
    BUF4D(i, j, k, t) = -1.0*xv*( dvdy + dwdz);

}

/* Compute the Y component of vorticity tendency due
   to stretching of the vorticity along the Y axis. */
__device__ void calc_yvort_stretch(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *wstag = data->w_4d_chunk;
    float *yvort = data->yvort_4d_chunk;
    float *yvort_stretch = data->yvstretch_4d_chunk;

    // this stencil conveniently lands itself on the scalar grid,
    // so we won't have to worry about doing any averaging. I think.
    float *buf0 = yvort;
    float yv = BUF4D(i, j, k, t);
    float dudx = ( ( UA4D(i, j, k, t) - UA4D(i-1, j, k, t) )/grid->dx) * UF(i);
    float dwdz = ( ( WA4D(i, j, k, t) - WA4D(i, j, k-1, t) )/grid->dz) * MF(k);

    buf0 = yvort_stretch;
    BUF4D(i, j, k, t) = -1.0*yv*( dudx + dwdz);
}

/* Compute the Z component of vorticity tendency due
   to stretching of the vorticity along the Z axis. */
__device__ void calc_zvort_stretch(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    float *ustag = data->u_4d_chunk;
    float *vstag = data->v_4d_chunk;
    float *zvort = data->zvort_4d_chunk;
    float *zvort_stretch = data->zvstretch_4d_chunk;

    // this stencil conveniently lands itself on the scalar grid,
    // so we won't have to worry about doing any averaging. I think.
    float *buf0 = zvort;
    float zv = BUF4D(i, j, k, t);
    float dudx = ( ( UA4D(i, j, k, t) - UA4D(i-1, j, k, t) )/grid->dx) * UF(i);
    float dvdy = ( ( VA4D(i, j, k, t) - VA4D(i, j-1, k, t) )/grid->dy) * VF(j);

    buf0 = zvort_stretch;
    BUF4D(i, j, k, t) = -1.0*zv*( dudx + dvdy);
}

__device__ void calc_zvort_solenoid(datagrid *grid, integration_data *data, int *idx_4D, int NX, int NY, int NZ) {
    int i = idx_4D[0];
    int j = idx_4D[1];
    int k = idx_4D[2];
    int t = idx_4D[3];

    // We can use p' here with no problems
    float *dum0 = data->pres_4d_chunk;
    // dP/dx
    float dpdx = ( (TEM4D(i+1, j, k, t) - TEM4D(i-1, j, k, t)) / ( 2*grid->dx ) ) * UH(i);
    // dP/dy
    float dpdy = ( (TEM4D(i, j+1, k, t) - TEM4D(i, j-1, k, t)) / ( 2*grid->dy ) ) * VH(j);

    dum0 = data->rho_4d_chunk;
    // dRho/dy
    float rho2 = TEM4D(i, j+1, k, t) + grid->rho0[k];
    float rho1 = TEM4D(i, j-1, k, t) + grid->rho0[k];
    float dalphady = ( ( (1./rho2) - (1./rho1) ) / ( 2*grid->dy ) ) * VH(j);

    // dRho/dx
    rho2 = TEM4D(i+1, j, k, t) + grid->rho0[k];
    rho1 = TEM4D(i-1, j, k, t) + grid->rho0[k];
    float dalphadx = ( ( (1./rho2) - (1./rho1) ) / ( 2*grid->dx ) ) * UH(i);

    // compute and save to the array
    float *buf0 = data->zvort_solenoid_4d_chunk; 
    BUF4D(i, j, k, t) = (dpdx*dalphady) - (dpdy*dalphadx);
}

/* When doing the parcel trajectory integration, George Bryan does
   some fun stuff with the lower boundaries/ghost zones of the arrays, presumably
   to prevent the parcels from exiting out the bottom of the domain
   or experience artificial values. This sets the ghost zone values. */
__global__ void applyMomentumBC(float *ustag, float *vstag, float *wstag, int NX, int NY, int NZ, int tStart, int tEnd) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    // this is done for easy comparison to CM1 code
    int ni = NX; int nj = NY;

    // this is a lower boundary condition, so only when k is 0
    // also this is on the u staggered mesh
    if (( j < nj+1) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the u stagger macro to handle the
            // proper indexing
            UA4D(i, j, 0, tidx) = UA4D(i, j, 1, tidx);
        }
    }
    
    // do the same but now on the v staggered grid
    if (( j < nj+1) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the v stagger macro to handle the
            // proper indexing
            VA4D(i, j, 0, tidx) = VA4D(i, j, 1, tidx);
        }
    }

    // do the same but now on the w staggered grid
    if (( j < nj+1) && ( i < ni+1) && ( k == 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // use the w stagger macro to handle the
            // proper indexing
            WA4D(i, j, 0, tidx) = -1*WA4D(i, j, 2, tidx);
        }
    }
}


__global__ void calcvort(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY+1) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort(grid, data, idx_4D, NX, NY, NZ);
        }
    }

    if ((i < NX+1) && (j < NY) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_yvort(grid, data, idx_4D, NX, NY, NZ);
        }
    }
    if ((i <= NX+1) && (j <= NY+1) && (k > 0) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvort(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the Vorticity Equation */
__global__ void calcvortstretch(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX) && (j < NY+1) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort_stretch(grid, data, idx_4D, NX, NY, NZ);
        }
    }

    if ((i < NX+1) && (j < NY) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_yvort_stretch(grid, data, idx_4D, NX, NY, NZ);
        }
    }
    if ((i < NX+1) && (j < NY+1) && (k > 0) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvort_stretch(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the Vorticity Equation */
__global__ void calcxvorttilt(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX+1) && (j < NY+1) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_xvort_tilt(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the Vorticity Equation */
__global__ void calcyvorttilt(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX+1) && (j < NY+1) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_yvort_tilt(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the Vorticity Equation */
__global__ void calczvorttilt(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    if ((i < NX+1) && (j < NY+1) && (k > 1) && (k < NZ+1)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvort_tilt(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Compute the forcing tendencies from the pressure-volume solenoid term */
__global__ void calczvortsolenoid(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our 3D index based on our blocks/threads
    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;
    int k = (blockIdx.z*blockDim.z) + threadIdx.z;
    int idx_4D[4];
    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    //printf("%i, %i, %i\n", i, j, k);

    idx_4D[0] = i; idx_4D[1] = j; idx_4D[2] = k;
    // Even though there are NZ points, it's a center difference
    // and we reach out NZ+1 points to get the derivatives
    if ((i < NX) && (j < NY) && (k < NZ) && ( i > 0 ) && (j > 0) && (k > 0)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            idx_4D[3] = tidx;
            calc_zvort_solenoid(grid, data, idx_4D, NX, NY, NZ);
        }
    }
}

/* Zero out the temporary arrays */
__global__ void zeroTemArrays(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *dum0;
    if (( i < NX+1) && ( j < NY+1) && ( k < NZ+1)) {
        dum0 = data->tem1_4d_chunk;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem2_4d_chunk;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem3_4d_chunk;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
        dum0 = data->tem4_4d_chunk;
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            TEM4D(i, j, k, tidx) = 0.0;
        }
    }
}


/* Apply the free-slip lower boundary condition to the vorticity field. */
__global__ void applyVortBC(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *dum0;

    // NOTE: Not sure if need to use BUF4D or TEM4D. The size of the array
    // will for sure be respected by BUF4D but unsure if it even matters here.

    // This is a lower boundary condition, so only when k is 0.
    // Start with xvort. 
    if (( i < NX+1) && ( j < NY+1) && ( k == 1)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // at this stage, xvort is in the tem1 array
            dum0 = data->tem1_4d_chunk;
            TEM4D(i, j, 1, tidx) = TEM4D(i, j, 2, tidx);
            // at this stage, yvort is in the tem2 array
            dum0 = data->tem2_4d_chunk;
            TEM4D(i, j, 1, tidx) = TEM4D(i, j, 2, tidx);
            // I'm technically ignoring an upper boundary condition
            // here, but we never really guarantee that we're at
            // the top of the model domain because we do a lot of subsetting.
            // So, for now, we assume we're nowehere near the top. 
        }
    }
}

/* Apply the free-slip lower boundary condition to the vorticity field. */
__global__ void applyVortTendBC(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *dum0;

    // NOTE: Not sure if need to use BUF4D or TEM4D. The size of the array
    // will for sure be respected by BUF4D but unsure if it even matters here.

    // This is a lower boundary condition, so only when k is 0.
    // Start with xvort. 
    if (( i < NX+1) && ( j < NY+1) && ( k == 1)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            dum0 = data->tem1_4d_chunk;
            TEM4D(i, j, 1, tidx) = TEM4D(i, j, 2, tidx);
            dum0 = data->tem2_4d_chunk;
            TEM4D(i, j, 1, tidx) = TEM4D(i, j, 2, tidx);
            dum0 = data->tem3_4d_chunk;
            TEM4D(i, j, 1, tidx) = TEM4D(i, j, 2, tidx);
            dum0 = data->tem4_4d_chunk;
            TEM4D(i, j, 1, tidx) = TEM4D(i, j, 1, tidx);
            // I'm technically ignoring an upper boundary condition
            // here, but we never really guarantee that we're at
            // the top of the model domain because we do a lot of subsetting.
            // So, for now, we assume we're nowehere near the top. 
        }
    }
}


/* Average our vorticity values back to the scalar grid for interpolation
   to the parcel paths. We're able to do this in parallel by making use of
   the three temporary arrays allocated on our grid, which means that the
   xvort/yvort/zvort arrays will be averaged into tem1/tem2/tem3. After
   calling this kernel, you MUST set the new pointers appropriately. */
__global__ void doVortAvg(datagrid *grid, integration_data *data, int tStart, int tEnd) {

    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;

    if ((i < NX) && (j < NY) && (k < NZ) && (k > 0)) {
        // loop over the number of time steps we have in memory
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            // average the temporary arrays into the result arrays
            dum0 = data->tem1_4d_chunk;
            buf0 = data->xvort_4d_chunk;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) +\
                                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem2_4d_chunk;
            buf0 = data->yvort_4d_chunk;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) +\
                                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem3_4d_chunk;
            buf0 = data->zvort_4d_chunk;
            BUF4D(i, j, k, tidx) = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) +\
                                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );
        }
    }
}

/* Average the derivatives within the temporary arrays used to compute
   the tilting rate and then combine the terms into the final xvtilt
   array. It is assumed that the derivatives have been precomputed into
   the temporary arrays. */
__global__ void doXVortTiltAvg(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;
    float dudy, dwdx, dudz, dvdx;

    // We do the average for each array at a given point
    // and then finish the computation for the zvort tilt
    if ((i < NX) && (j < NY) && (k < NZ) && (k > 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            dum0 = data->tem1_4d_chunk;
            dudy = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );

            dum0 = data->tem2_4d_chunk;
            dwdx = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem3_4d_chunk;
            dudz = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem4_4d_chunk;
            dvdx = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );

            buf0 = data->xvtilt_4d_chunk;
            BUF4D(i, j, k, tidx) = -1.0*((dudy*dwdx) - (dudz*dvdx));
        }
    }
}

/* Average the derivatives within the temporary arrays used to compute
   the tilting rate and then combine the terms into the final yvtilt
   array. It is assumed that the derivatives have been precomputed into
   the temporary arrays. */
__global__ void doYVortTiltAvg(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;
    float dvdx, dwdy, dvdz, dudy;

    // We do the average for each array at a given point
    // and then finish the computation for the zvort tilt
    if ((i < NX) && (j < NY) && (k < NZ) && (k > 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            dum0 = data->tem1_4d_chunk;
            dvdx = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );

            dum0 = data->tem2_4d_chunk;
            dwdy = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem3_4d_chunk;
            dvdz = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem4_4d_chunk;
            dudy = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j+1, k, tidx) + TEM4D(i+1, j+1, k, tidx) );

            buf0 = data->yvtilt_4d_chunk;
            BUF4D(i, j, k, tidx) = -1.0*((dvdx*dwdy) - (dvdz*dudy));
        }
    }
}

/* Average the derivatives within the temporary arrays used to compute
   the tilting rate and then combine the terms into the final zvtilt
   array. It is assumed that the derivatives have been precomputed into
   the temporary arrays. */
__global__ void doZVortTiltAvg(datagrid *grid, integration_data *data, int tStart, int tEnd) {
    // get our grid indices based on our block and thread info
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int NX = grid->NX;
    int NY = grid->NY;
    int NZ = grid->NZ;
    float *buf0, *dum0;
    float dwdx, dvdz, dwdy, dudz;

    // We do the average for each array at a given point
    // and then finish the computation for the zvort tilt
    if ((i < NX) && (j < NY) && (k < NZ) && (k > 0)) {
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            dum0 = data->tem1_4d_chunk;
            dwdx = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            dum0 = data->tem2_4d_chunk;
            dvdz = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem3_4d_chunk;
            dwdy = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i, j+1, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i, j+1, k+1, tidx) );

            dum0 = data->tem4_4d_chunk;
            dudz = 0.25 * ( TEM4D(i, j, k, tidx) + TEM4D(i+1, j, k, tidx) + \
                            TEM4D(i, j, k+1, tidx) + TEM4D(i+1, j, k+1, tidx) );

            buf0 = data->zvtilt_4d_chunk;
            BUF4D(i, j, k, tidx) = (dwdy*dudz)-(dwdx*dvdz);
        }
    }
}


/*  Execute all of the required kernels on the GPU that are necessary for computing the 3
    components of vorticity. The idea here is that we're building wrappers on wrappers to
    simplify the process for the end user that just wants to calculate vorticity. This is
    also a necessary adjustment because the tendency calculations will require multiple
    steps, so transitioning this block of code as a proof of concept for how the programming
    model should work. */
void doCalcVort(datagrid *grid, integration_data *data, int tStart, int tEnd, dim3 numBlocks, dim3 threadsPerBlock) {
    // calculate the three compionents of vorticity
    calcvort<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );

    // apply the lower boundary condition to the horizontal
    // components of vorticity
    applyVortBC<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );

    // Average the vorticity to the scalar grid using the temporary
    // arrays we allocated. After doing the averaging, we have to 
    // set the pointers to the temporary arrays as the new xvort,
    // yvort, and zvort, and set the old x/y/zvort arrays as the new
    // temporary arrays. Note: may have to zero those out in the future...
    doVortAvg<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );
    zeroTemArrays<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );
} 

void doCalcVortTend(datagrid *grid, integration_data *data, int tStart, int tEnd, dim3 numBlocks, dim3 threadsPerBlock) {

    // Compute the vorticity tendency due to stretching. These conveniently
    // end up on the scalar grid, and no extra steps are required. This will
    // compute the tendency for all 3 components of vorticity. 
    calcvortstretch<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );
    applyVortTendBC<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );

    // Compute the vertical vorticity tendency due to tilting. We have to do 
    // each component individually because we have to average the arrays back
    // to the scalar grid. It's a mess. 
    calcxvorttilt<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );
    applyVortTendBC<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );
    doXVortTiltAvg<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );
    zeroTemArrays<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );

    calcyvorttilt<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );
    applyVortTendBC<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );
    doYVortTiltAvg<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );
    zeroTemArrays<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );

    calczvorttilt<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );
    applyVortTendBC<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );
    doZVortTiltAvg<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );
    zeroTemArrays<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk( hipPeekAtLastError() );


    calczvortsolenoid<<<numBlocks, threadsPerBlock>>>(grid, data, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
}

__global__ void integrate(datagrid *grid, parcel_pos *parcels, integration_data *data, \
                          int tStart, int tEnd, int totTime, int direct) {

	int parcel_id = blockIdx.x;
    // safety check to make sure our thread index doesn't
    // go out of our array bounds
    if (parcel_id < parcels->nParcels) {
        bool is_ugrd = false;
        bool is_vgrd = false;
        bool is_wgrd = false;

        float pcl_u, pcl_v, pcl_w;
        float point[3];

        // loop over the number of time steps we are
        // integrating over
        for (int tidx = tStart; tidx < tEnd; ++tidx) {
            point[0] = parcels->xpos[PCL(tidx, parcel_id, totTime)];
            point[1] = parcels->ypos[PCL(tidx, parcel_id, totTime)];
            point[2] = parcels->zpos[PCL(tidx, parcel_id, totTime)];
            //printf("My Point Is: X = %f Y = %f Z = %f t = %d nParcels = %d\n", point[0], point[1], point[2], tidx, parcels->nParcels);

            is_ugrd = true;
            is_vgrd = false;
            is_wgrd = false;
            pcl_u = interp3D(grid, data->u_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);

            is_ugrd = false;
            is_vgrd = true;
            is_wgrd = false;
            pcl_v = interp3D(grid, data->v_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);

            is_ugrd = false;
            is_vgrd = false;
            is_wgrd = true;
            pcl_w = interp3D(grid, data->w_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            //printf("pcl u: %f pcl v: %f pcl w: %f\n", pcl_u, pcl_v, pcl_w);

            // interpolate scalar values to the parcel point
            is_ugrd = false;
            is_vgrd = false;
            is_wgrd = false;
            float pclxvort = interp3D(grid, data->xvort_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            float pclyvort = interp3D(grid, data->yvort_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            float pclzvort = interp3D(grid, data->zvort_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            float pclxvorttilt = interp3D(grid, data->xvtilt_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            float pclyvorttilt = interp3D(grid, data->yvtilt_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            float pclzvorttilt = interp3D(grid, data->zvtilt_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            float pclxvortstretch = interp3D(grid, data->xvstretch_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            float pclyvortstretch = interp3D(grid, data->yvstretch_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            float pclzvortstretch = interp3D(grid, data->zvstretch_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            float pclzvortsolenoid = interp3D(grid, data->zvort_solenoid_4d_chunk, point, is_ugrd, is_vgrd, is_wgrd, tidx);
            
            // integrate X position forward by the U wind
            point[0] += pcl_u * (0.5) * direct;
            // integrate Y position forward by the V wind
            point[1] += pcl_v * (0.5) * direct;
            // integrate Z position forward by the W wind
            point[2] += pcl_w * (0.5) * direct;
            if ((pcl_u == -999.0) || (pcl_v == -999.0) || (pcl_w == -999.0)) {
                printf("Warning: missing values detected at x: %f y:%f z:%f with ground bounds X0: %f Y0: %f Z0: %f X1: %f Y1: %f Z1: %f\n", \
                    point[0], point[1], point[2], grid->xh[0], grid->yh[0], grid->zh[0], grid->xh[grid->NX-1], grid->yh[grid->NY-1], grid->zh[grid->NZ-1]);
                return;
            }


            parcels->xpos[PCL(tidx+1, parcel_id, totTime)] = point[0]; 
            parcels->ypos[PCL(tidx+1, parcel_id, totTime)] = point[1];
            parcels->zpos[PCL(tidx+1, parcel_id, totTime)] = point[2];
            parcels->pclu[PCL(tidx,   parcel_id, totTime)] = pcl_u;
            parcels->pclv[PCL(tidx,   parcel_id, totTime)] = pcl_v;
            parcels->pclw[PCL(tidx,   parcel_id, totTime)] = pcl_w;

            // Store the vorticity in the parcel
            parcels->pclxvort[PCL(tidx, parcel_id, totTime)] = pclxvort;
            parcels->pclyvort[PCL(tidx, parcel_id, totTime)] = pclyvort;
            parcels->pclzvort[PCL(tidx, parcel_id, totTime)] = pclzvort;
            parcels->pclxvorttilt[PCL(tidx, parcel_id, totTime)] = pclxvorttilt;
            parcels->pclyvorttilt[PCL(tidx, parcel_id, totTime)] = pclyvorttilt;
            parcels->pclzvorttilt[PCL(tidx, parcel_id, totTime)] = pclzvorttilt;
            parcels->pclxvortstretch[PCL(tidx, parcel_id, totTime)] = pclxvortstretch;
            parcels->pclyvortstretch[PCL(tidx, parcel_id, totTime)] = pclyvortstretch;
            parcels->pclzvortstretch[PCL(tidx, parcel_id, totTime)] = pclzvortstretch;
            parcels->pclzvortsolenoid[PCL(tidx, parcel_id, totTime)] = pclzvortsolenoid;
        }
    }
}

/*This function handles allocating memory on the GPU, transferring the CPU
arrays to GPU global memory, calling the integrate GPU kernel, and then
updating the position vectors with the new stuff*/
void cudaIntegrateParcels(datagrid *grid, integration_data *data, parcel_pos *parcels, int nT, int totTime, int direct) {

    int tStart, tEnd;
    tStart = 0;
    tEnd = nT;
    int NX, NY, NZ;
    // set the NX, NY, NZ
    // variables for calculations
    NX = grid->NX;
    NY = grid->NY;
    NZ = grid->NZ;


    // set the thread/block execution strategy for the kernels

    // Okay, so I think the last remaining issue might lie here. For some reason, some blocks 
    // must not be executing or something, seemingly related to the threadsPerBlock size. 
    // Changing to 4x4x4 fixed for xvort, but not yvort. I think we need to dynamically set
    // threadsPerBloc(x, y, z) based on the size of our grid at a given time step. 
    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((int)ceil(NX/threadsPerBlock.x)+1, (int)ceil(NY/threadsPerBlock.y)+1, (int)ceil(NZ/threadsPerBlock.z)+1); 

    // we synchronize the device before doing anything to make sure all
    // array memory transfers have safely completed. This is probably 
    // unnecessary but I'm doing it anyways because overcaution never
    // goes wrong. Ever.
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );

    // Calculate the three compionents of vorticity
    // and do the necessary averaging. This is a wrapper that
    // calls the necessary kernels and assigns the pointers
    // appropriately such that the "user" only has to call this method.
    doCalcVort(grid, data, tStart, tEnd, numBlocks, threadsPerBlock);
    

    // Calculate the vorticity forcing terms for each of the 3 components.
    // This is a wrapper that calls the necessary kernels to compute the
    // derivatives and average them back to the scalar grid where necessary. 
    doCalcVortTend(grid, data, tStart, tEnd, numBlocks, threadsPerBlock);
    // Before integrating the trajectories, George Bryan sets some below-grid/surface conditions 
    // that we need to consider. This handles applying those boundary conditions. 
    applyMomentumBC<<<numBlocks, threadsPerBlock>>>(data->u_4d_chunk, data->v_4d_chunk, data->w_4d_chunk, NX, NY, NZ, tStart, tEnd);
    gpuErrchk(hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );

    // integrate the parcels forward in time and interpolate
    // calculations to trajectories. 
    integrate<<<parcels->nParcels, 1>>>(grid, parcels, data, tStart, tEnd, totTime, direct);
    gpuErrchk(hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );

}

#endif
